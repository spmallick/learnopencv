#include "hip/hip_runtime.h"
#include "darknet.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipblas.h>
#include <float.h>

#include "activations.h"
#include "dark_cuda.h"

__device__ float lhtan_activate_kernel(float x)
{
    if(x < 0) return .001*x;
    if(x > 1) return .001*(x-1) + 1;
    return x;
}
__device__ float lhtan_gradient_kernel(float x)
{
    if(x > 0 && x < 1) return 1;
    return .001;
}

__device__ float hardtan_activate_kernel(float x)
{
    if (x < -1) return -1;
    if (x > 1) return 1;
    return x;
}
__device__ float linear_activate_kernel(float x){return x;}
__device__ float logistic_activate_kernel(float x){return 1.f/(1.f + expf(-x));}
__device__ float loggy_activate_kernel(float x){return 2.f/(1.f + expf(-x)) - 1;}
__device__ float relu_activate_kernel(float x){return x*(x>0);}
__device__ float relu6_activate_kernel(float x) { return min_val_cmp(max_val_cmp(x, 0), 6); }
__device__ float elu_activate_kernel(float x){return (x >= 0)*x + (x < 0)*(expf(x)-1);}
__device__ float selu_activate_kernel(float x) { return (x >= 0)*1.0507f*x + (x < 0)*1.0507f*1.6732f*(expf(x) - 1); }
__device__ float relie_activate_kernel(float x){return (x>0) ? x : .01f*x;}
__device__ float ramp_activate_kernel(float x){return x*(x>0)+.1f*x;}
__device__ float leaky_activate_kernel(float x){return (x>0) ? x : .1f*x;}
__device__ float tanh_activate_kernel(float x){return (2/(1 + expf(-2*x)) - 1);}
__device__ float gelu_activate_kernel(float x){return (0.5*x*(1 + tanhf(0.797885*x + 0.035677*powf(x, 3))));}
__device__ float softplus_kernel(float x, float threshold = 20) {
    if (x > threshold) return x;                // too large
    else if (x < -threshold) return expf(x);    // too small
    return log1pf(expf(x));
    //return logf(expf(x) + 1);
}
__device__ float plse_activate_kernel(float x)
{
    if(x < -4) return .01f * (x + 4);
    if(x > 4)  return .01f * (x - 4) + 1;
    return .125f*x + .5f;
}
__device__ float stair_activate_kernel(float x)
{
    int n = floorf(x);
    if (n%2 == 0) return floorf(x/2.f);
    else return (x - n) + floorf(x/2.f);
}


__device__ float hardtan_gradient_kernel(float x)
{
    if (x > -1 && x < 1) return 1;
    return 0;
}
__device__ float linear_gradient_kernel(float x){return 1;}
__device__ float logistic_gradient_kernel(float x){return (1-x)*x;}
__device__ float loggy_gradient_kernel(float x)
{
    float y = (x+1.F)/2.F;
    return 2*(1-y)*y;
}
__device__ float relu_gradient_kernel(float x){return (x>0);}
__device__ float relu6_gradient_kernel(float x) { return (x > 0 && x < 6); }
__device__ float elu_gradient_kernel(float x){return (x >= 0) + (x < 0)*(x + 1);}
__device__ float selu_gradient_kernel(float x) { return (x >= 0)*1.0507f + (x < 0)*(x + 1.0507f*1.6732f); }
__device__ float relie_gradient_kernel(float x){return (x>0) ? 1 : .01f;}
__device__ float ramp_gradient_kernel(float x){return (x>0)+.1f;}
__device__ float leaky_gradient_kernel(float x){return (x>0) ? 1 : .1f;}
__device__ float tanh_gradient_kernel(float x){return 1-x*x;}
__device__ float sech_gpu(float x) { return 2 / (expf(x) + expf(-x)); }
__device__ float gelu_gradient_kernel(float x) {
    const float x3 = powf(x, 3);
    return 0.5*tanhf(0.0356774*x3 + 0.797885*x) + (0.0535161*x3 + 0.398942*x) * powf(sech_gpu(0.0356774*x3 + 0.797885*x), 2) + 0.5;
}
__device__ float plse_gradient_kernel(float x){return (x < 0 || x > 1) ? .01f : .125f;}
__device__ float stair_gradient_kernel(float x)
{
    if (floorf(x) == x) return 0;
    return 1;
}

__device__ float activate_kernel(float x, ACTIVATION a)
{
    switch(a){
        case LINEAR:
            return linear_activate_kernel(x);
        case LOGISTIC:
            return logistic_activate_kernel(x);
        case LOGGY:
            return loggy_activate_kernel(x);
        case RELU:
            return relu_activate_kernel(x);
        case RELU6:
            return relu6_activate_kernel(x);
        case ELU:
            return elu_activate_kernel(x);
        case SELU:
            return selu_activate_kernel(x);
        case GELU:
            return gelu_activate_kernel(x);
        case RELIE:
            return relie_activate_kernel(x);
        case RAMP:
            return ramp_activate_kernel(x);
        case LEAKY:
            return leaky_activate_kernel(x);
        case TANH:
            return tanh_activate_kernel(x);
        case PLSE:
            return plse_activate_kernel(x);
        case STAIR:
            return stair_activate_kernel(x);
        case HARDTAN:
            return hardtan_activate_kernel(x);
        case LHTAN:
            return lhtan_activate_kernel(x);
    }
    return 0;
}

__device__ float gradient_kernel(float x, ACTIVATION a)
{
    switch (a) {
    case LINEAR:
        return linear_gradient_kernel(x);
    case LOGISTIC:
        return logistic_gradient_kernel(x);
    case LOGGY:
        return loggy_gradient_kernel(x);
    case RELU:
        return relu_gradient_kernel(x);
    case RELU6:
        return relu6_gradient_kernel(x);
    case NORM_CHAN:
        return relu_gradient_kernel(x);
    case ELU:
        return elu_gradient_kernel(x);
    case SELU:
        return selu_gradient_kernel(x);
    case GELU:
        return gelu_gradient_kernel(x);
    case RELIE:
        return relie_gradient_kernel(x);
    case RAMP:
        return ramp_gradient_kernel(x);
    case LEAKY:
        return leaky_gradient_kernel(x);
    case TANH:
        return tanh_gradient_kernel(x);
    case PLSE:
        return plse_gradient_kernel(x);
    case STAIR:
        return stair_gradient_kernel(x);
    case HARDTAN:
        return hardtan_gradient_kernel(x);
    case LHTAN:
        return lhtan_gradient_kernel(x);
    }
    return 0;
}

__global__ void binary_gradient_array_kernel(float *x, float *dy, int n, int s, BINARY_ACTIVATION a, float *dx)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    int i = id % s;
    int b = id / s;
    float x1 = x[b*s + i];
    float x2 = x[b*s + s / 2 + i];
    if (id < n) {
        float de = dy[id];
        dx[b*s + i] = x2*de;
        dx[b*s + s / 2 + i] = x1*de;
    }
}

extern "C" void binary_gradient_array_gpu(float *x, float *dx, int n, int size, BINARY_ACTIVATION a, float *y)
{
    binary_gradient_array_kernel << <cuda_gridsize(n / 2), BLOCK, 0, get_cuda_stream() >> >(x, dx, n / 2, size, a, y);
    CHECK_CUDA(hipPeekAtLastError());
}
__global__ void binary_activate_array_kernel(float *x, int n, int s, BINARY_ACTIVATION a, float *y)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    int i = id % s;
    int b = id / s;
    float x1 = x[b*s + i];
    float x2 = x[b*s + s / 2 + i];
    if (id < n) y[id] = x1*x2;
}

extern "C" void binary_activate_array_gpu(float *x, int n, int size, BINARY_ACTIVATION a, float *y)
{
    binary_activate_array_kernel << <cuda_gridsize(n / 2), BLOCK, 0, get_cuda_stream() >> >(x, n / 2, size, a, y);
    CHECK_CUDA(hipPeekAtLastError());
}

__global__ void activate_array_kernel(float *x, int n, ACTIVATION a)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < n) x[i] = activate_kernel(x[i], a);
}



__global__ void activate_array_swish_kernel(float *x, int n, float *output_sigmoid_gpu, float *output_gpu)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i < n) {
        float x_val = x[i];
        float sigmoid = logistic_activate_kernel(x_val);
        if (output_sigmoid_gpu) output_sigmoid_gpu[i] = sigmoid;
        output_gpu[i] = x_val * sigmoid;
    }
}

__device__ float mish_njuffa(float x)
{
    float r;
    float e = expf(x);
    r = 1.0f / fmaf(fmaf(-0.5f, e, -1.0f), e, -1.0f);
    r = fmaf(r, x, x);
    return r;
}

__device__ float mish_yashas(float x)
{
    float e = __expf(x);
    if (x <= -18.0f)
        return x * e;

    float n = e * e + 2 * e;
    if (x <= -5.0f)
        return x * __fdividef(n, n + 2);

    return x - 2 * __fdividef(x, n + 2);
}

__device__ float mish_yashas2(float x)
{
    float e = __expf(x);
    float n = e * e + 2 * e;
    if (x <= -0.6f)
        return x * __fdividef(n, n + 2);

    return x - 2 * __fdividef(x, n + 2);
}

// https://github.com/digantamisra98/Mish
__global__ void activate_array_mish_kernel(float *x, int n, float *activation_input, float *output_gpu)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i < n) {
        const float MISH_THRESHOLD = 20;
        float x_val = x[i];
        if (activation_input) activation_input[i] = x_val;    // store value before activation
        //output_gpu[i] = x_val * tanh_activate_kernel(logf(1 + expf(x_val)));

        // Pytorch: https://github.com/thomasbrandon/mish-cuda/blob/master/csrc/mish.h#L17-L20
        // TF: https://github.com/tensorflow/addons/blob/093cdfa85d334cbe19a37624c33198f3140109ed/tensorflow_addons/custom_ops/activations/cc/kernels/mish_op.h#L40-L49
        // log1p(x) == log(x + 1)
        //output_gpu[i] = x_val * tanh_activate_kernel( softplus_kernel(x_val, MISH_THRESHOLD) );
        output_gpu[i] = mish_yashas2(x_val);
        //output_gpu[i] = mish_njuffa(x_val);
    }
}

__device__ float hard_mish_yashas(float x)
{
    if (x > 0)
        return x;
    if (x > -2)
        return x * x / 2 + x;
    return 0;
}

__global__ void activate_array_hard_mish_kernel(float *x, int n, float *activation_input, float *output_gpu)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i < n) {

        float x_val = x[i];
        if (activation_input) activation_input[i] = x_val;    // store value before activation
        output_gpu[i] = hard_mish_yashas(x_val);
    }
}
__global__ void activate_array_leaky_kernel(float *x, int n)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < n) {
        x[index] = leaky_activate_kernel(x[index]);
    }
}

__global__ void activate_array_selu_kernel(float *x, int n)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < n) {
        x[index] = selu_activate_kernel(x[index]);
    }
}

__global__ void activate_array_gelu_kernel(float *x, int n)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < n) {
        x[index] = gelu_activate_kernel(x[index]);
    }
}

__global__ void activate_array_logistic_kernel(float *x, int n)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < n) {
        x[index] = logistic_activate_kernel(x[index]);
    }
}

__global__ void activate_array_tanh_kernel(float *x, int n)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < n) {
        x[index] = tanh_activate_kernel(x[index]);
    }
}

__global__ void activate_array_hardtan_kernel(float *x, int n)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < n) {
        x[index] = hardtan_activate_kernel(x[index]);
    }
}

__global__ void activate_array_relu_kernel(float *x, int n)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < n) {
        x[index] = relu_activate_kernel(x[index]);
    }
}

__global__ void activate_array_relu6_kernel(float *x, int n)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < n) {
        x[index] = relu6_activate_kernel(x[index]);
    }
}

__global__ void gradient_array_kernel(float *x, int n, ACTIVATION a, float *delta)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < n) delta[i] *= gradient_kernel(x[i], a);
}

// https://github.com/BVLC/caffe/blob/04ab089db018a292ae48d51732dd6c66766b36b6/src/caffe/layers/swish_layer.cu#L28-L30
__global__ void gradient_array_swish_kernel(float *x, int n, float *sigmoid_gpu, float *delta)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i < n) {
        float swish = x[i];
        delta[i] *= swish + sigmoid_gpu[i] * (1 - swish); // gradient_kernel(x[i], a);
    }
}

// https://github.com/digantamisra98/Mish
__global__ void gradient_array_mish_kernel(int n, float *activation_input_gpu, float *delta)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i < n) {
        const float MISH_THRESHOLD = 20.0f;

        // implementation from TensorFlow: https://github.com/tensorflow/addons/blob/093cdfa85d334cbe19a37624c33198f3140109ed/tensorflow_addons/custom_ops/activations/cc/kernels/mish_op.h#L66-L80
        // implementation from Pytorch: https://github.com/thomasbrandon/mish-cuda/blob/master/csrc/mish.h#L26-L31
        // log1p(x) == log(x + 1)
        const float inp = activation_input_gpu[i];
        const float sp = softplus_kernel(inp, MISH_THRESHOLD);
        const float grad_sp = -expm1f(-sp);
        //const float grad_sp = 1 - expf(-sp);
        const float tsp = tanh(sp);
        const float grad_tsp = (1 - tsp*tsp) * grad_sp;
        const float grad = inp * grad_tsp + tsp;
        delta[i] *= grad;

        //float x = activation_input[i];
        //float d = 2 * expf(x) + expf(2 * x) + 2;
        //float w = 4 * (x + 1) + 4 * expf(2 * x) + expf(3 * x) + expf(x)*(4 * x + 6);
        //float derivative = expf(x) * w / (d * d);
        //delta[i] *= derivative;
    }
}

__device__ float hard_mish_yashas_grad(float x)
{
    if (x > 0)
        return 1;
    if (x > -2)
        return x + 1;
    return 0;
}

__global__ void gradient_array_hard_mish_kernel(int n, float *activation_input_gpu, float *delta)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i < n) {

        const float x = activation_input_gpu[i];
        delta[i] *= hard_mish_yashas_grad(x);
    }
}

__global__ void gradient_array_leaky_kernel(float *x, int n, float *delta)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < n) {
        delta[index] *= leaky_gradient_kernel(x[index]);
    }
}

__global__ void gradient_array_revleaky_kernel(float *x, int n, float *delta)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < n) {
        delta[index] /= leaky_gradient_kernel(x[index]);
    }
}

__global__ void gradient_array_selu_kernel(float *x, int n, float *delta)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < n) {
        delta[index] *= selu_gradient_kernel(x[index]);
    }
}

__global__ void gradient_array_gelu_kernel(float *x, int n, float *delta)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < n) {
        delta[index] *= gelu_gradient_kernel(x[index]);
    }
}

__global__ void gradient_array_logistic_kernel(float *x, int n, float *delta)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < n) {
        delta[index] *= logistic_gradient_kernel(x[index]);
    }
}

__global__ void gradient_array_tanh_kernel(float *x, int n, float *delta)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < n) {
        delta[index] *= tanh_gradient_kernel(x[index]);
    }
}

__global__ void gradient_array_hardtan_kernel(float *x, int n, float *delta)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < n) {
        delta[index] *= hardtan_gradient_kernel(x[index]);
    }
}

__global__ void gradient_array_relu_kernel(float *x, int n, float *delta)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < n) {
        delta[index] *= relu_gradient_kernel(x[index]);
    }
}

__global__ void gradient_array_relu6_kernel(float *x, int n, float *delta)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < n) {
        delta[index] *= relu6_gradient_kernel(x[index]);
    }
}

extern "C" void activate_array_ongpu(float *x, int n, ACTIVATION a)
{
    const int num_blocks = get_number_of_blocks(n, BLOCK);
    if (a == LINEAR) return;
    else if (a == LEAKY || a == REVLEAKY) activate_array_leaky_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> >(x, n);
    else if (a == LOGISTIC) activate_array_logistic_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> >(x, n);
    else if (a == TANH) activate_array_tanh_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> >(x, n);
    else if (a == HARDTAN) activate_array_hardtan_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> >(x, n);
    else if (a == RELU) activate_array_relu_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> >(x, n);
    else if (a == RELU6) activate_array_relu6_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> >(x, n);
    else if (a == SELU) activate_array_selu_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> >(x, n);
    else if (a == GELU) activate_array_gelu_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> >(x, n);
    else
        activate_array_kernel<<<cuda_gridsize(n), BLOCK, 0, get_cuda_stream()>>>(x, n, a);
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void activate_array_swish_ongpu(float *x, int n, float *output_sigmoid_gpu, float *output_gpu)
{
    const int num_blocks = get_number_of_blocks(n, BLOCK);
    activate_array_swish_kernel << <cuda_gridsize(n), BLOCK, 0, get_cuda_stream() >> >(x, n, output_sigmoid_gpu, output_gpu);
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void activate_array_mish_ongpu(float *x, int n, float *activation_input_gpu, float *output_gpu)
{
    const int num_blocks = get_number_of_blocks(n, BLOCK);
    activate_array_mish_kernel << <cuda_gridsize(n), BLOCK, 0, get_cuda_stream() >> >(x, n, activation_input_gpu, output_gpu);
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void activate_array_hard_mish_ongpu(float *x, int n, float *activation_input_gpu, float *output_gpu)
{
    const int num_blocks = get_number_of_blocks(n, BLOCK);
    activate_array_hard_mish_kernel << <cuda_gridsize(n), BLOCK, 0, get_cuda_stream() >> >(x, n, activation_input_gpu, output_gpu);
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void gradient_array_ongpu(float *x, int n, ACTIVATION a, float *delta)
{
    const int num_blocks = get_number_of_blocks(n, BLOCK);
    if (a == LINEAR) return;
    else if (a == LEAKY) gradient_array_leaky_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> >(x, n, delta);
    else if (a == REVLEAKY) gradient_array_revleaky_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> >(x, n, delta);
    else if (a == LOGISTIC) gradient_array_logistic_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> >(x, n, delta);
    else if (a == TANH) gradient_array_tanh_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> >(x, n, delta);
    else if (a == HARDTAN) gradient_array_hardtan_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> >(x, n, delta);
    else if (a == RELU) gradient_array_relu_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> >(x, n, delta);
    else if (a == RELU6) gradient_array_relu6_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> >(x, n, delta);
    //else if (a == NORM_CHAN) gradient_array_relu_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> >(x, n, delta);
    else if (a == NORM_CHAN_SOFTMAX || a == NORM_CHAN) {
        printf(" Error: should be used custom NORM_CHAN_SOFTMAX-function for gradient \n");
        exit(0);
    }
    else if (a == SELU) gradient_array_selu_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> >(x, n, delta);
    else if (a == GELU) gradient_array_gelu_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> >(x, n, delta);
    else
        gradient_array_kernel << <cuda_gridsize(n), BLOCK, 0, get_cuda_stream() >> > (x, n, a, delta);
    CHECK_CUDA(hipPeekAtLastError());
}


extern "C" void gradient_array_swish_ongpu(float *x, int n, float *sigmoid_gpu, float *delta)
{
    const int num_blocks = get_number_of_blocks(n, BLOCK);
    gradient_array_swish_kernel << <cuda_gridsize(n), BLOCK, 0, get_cuda_stream() >> > (x, n, sigmoid_gpu, delta);
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void gradient_array_mish_ongpu(int n, float *activation_input_gpu, float *delta)
{
    const int num_blocks = get_number_of_blocks(n, BLOCK);
    gradient_array_mish_kernel << <cuda_gridsize(n), BLOCK, 0, get_cuda_stream() >> > (n, activation_input_gpu, delta);
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void gradient_array_hard_mish_ongpu(int n, float *activation_input_gpu, float *delta)
{
    const int num_blocks = get_number_of_blocks(n, BLOCK);
    gradient_array_hard_mish_kernel << <cuda_gridsize(n), BLOCK, 0, get_cuda_stream() >> > (n, activation_input_gpu, delta);
    CHECK_CUDA(hipPeekAtLastError());
}


__global__ void activate_array_normalize_channels_kernel(float *x, int size, int batch, int channels, int wh_step, float *output_gpu)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    int wh_i = i % wh_step;
    int b = i / wh_step;

    const float eps = 0.0001;
    if (i < size) {
        float sum = eps;
        int k;
        for (k = 0; k < channels; ++k) {
            float val = x[wh_i + k * wh_step + b*wh_step*channels];
            if (val > 0) sum += val;
        }
        for (k = 0; k < channels; ++k) {
            float val = x[wh_i + k * wh_step + b*wh_step*channels];
            if (val > 0) val = val / sum;
            else val = 0;
            output_gpu[wh_i + k * wh_step + b*wh_step*channels] = val;
        }
    }
}

extern "C" void activate_array_normalize_channels_ongpu(float *x, int n, int batch, int channels, int wh_step, float *output_gpu)
{
    // n = w*h*c*batch
    // size = w*h*batch
    int size = n / channels;

    const int num_blocks = get_number_of_blocks(size, BLOCK);

    activate_array_normalize_channels_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> > (x, size, batch, channels, wh_step, output_gpu);
    CHECK_CUDA(hipPeekAtLastError());
}



__global__ void activate_array_normalize_channels_softmax_kernel(float *x, int size, int batch, int channels, int wh_step, float *output_gpu, int use_max_val)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    int wh_i = i % wh_step;
    int b = i / wh_step;

    const float eps = 0.0001;
    if (i < size) {
        float sum = eps;
        float max_val = -FLT_MAX;
        int k;
        if (use_max_val) {
            for (k = 0; k < channels; ++k) {
                float val = x[wh_i + k * wh_step + b*wh_step*channels];
                if (val > max_val || k == 0) max_val = val;
            }
        }
        else
            max_val = 0;

        for (k = 0; k < channels; ++k) {
            float val = x[wh_i + k * wh_step + b*wh_step*channels];
            sum += expf(val - max_val);
        }
        for (k = 0; k < channels; ++k) {
            float val = x[wh_i + k * wh_step + b*wh_step*channels];
            val = expf(val - max_val) / sum;
            if (isnan(val) || isinf(val)) val = 0;
            output_gpu[wh_i + k * wh_step + b*wh_step*channels] = val;
        }
    }
}

extern "C" void activate_array_normalize_channels_softmax_ongpu(float *x, int n, int batch, int channels, int wh_step, float *output_gpu, int use_max_val)
{
    // n = w*h*c*batch
    // size = w*h*batch
    int size = n / channels;

    const int num_blocks = get_number_of_blocks(size, BLOCK);

    activate_array_normalize_channels_softmax_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> > (x, size, batch, channels, wh_step, output_gpu, use_max_val);
    CHECK_CUDA(hipPeekAtLastError());
}



__global__ void gradient_array_normalize_channels_softmax_kernel(float *x, int size, int batch, int channels, int wh_step, float *delta_gpu)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    int wh_i = i % wh_step;
    int b = i / wh_step;

    if (i < size) {
        int k;
        /*
        float grad = 0;
        for (k = 0; k < channels; ++k) {
            const int index = wh_i + k * wh_step + b*wh_step*channels;
            float out = x[index];
            float delta = delta_gpu[index];
            grad += out*fabs(delta);
        }
        */
        for (k = 0; k < channels; ++k) {
            const int index = wh_i + k * wh_step + b*wh_step*channels;
            float delta = delta_gpu[index];
            float grad = x[index] * (1 - x[index]);
            delta = delta * grad;
            if (isnan(delta) || isinf(delta)) delta = 0;
            delta_gpu[index] = delta;
        }
    }
}

extern "C" void gradient_array_normalize_channels_softmax_ongpu(float *output_gpu, int n, int batch, int channels, int wh_step, float *delta_gpu)
{
    // n = w*h*c*batch
    // size = w*h*batch
    int size = n / channels;

    const int num_blocks = get_number_of_blocks(size, BLOCK);

    gradient_array_normalize_channels_softmax_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> > (output_gpu, size, batch, channels, wh_step, delta_gpu);
    CHECK_CUDA(hipPeekAtLastError());
}


__global__ void gradient_array_normalize_channels_kernel(float *x, int size, int batch, int channels, int wh_step, float *delta_gpu)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    int wh_i = i % wh_step;
    int b = i / wh_step;

    if (i < size) {
        int k;
        /*
        float grad = 0;
        for (k = 0; k < channels; ++k) {
            const int index = wh_i + k * wh_step + b*wh_step*channels;
            float out = x[index];
            float delta = delta_gpu[index];
            grad += out*fabs(delta);
        }
        */
        for (k = 0; k < channels; ++k) {
            const int index = wh_i + k * wh_step + b*wh_step*channels;
            if (x[index] > 0) {
                float delta = delta_gpu[index];
                float grad = x[index];
                delta = delta * grad;
                delta_gpu[index] = delta;
            }
        }
    }
}

extern "C" void gradient_array_normalize_channels_ongpu(float *output_gpu, int n, int batch, int channels, int wh_step, float *delta_gpu)
{
    // n = w*h*c*batch
    // size = w*h*batch
    int size = n / channels;

    const int num_blocks = get_number_of_blocks(size, BLOCK);

    gradient_array_normalize_channels_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> > (output_gpu, size, batch, channels, wh_step, delta_gpu);
    CHECK_CUDA(hipPeekAtLastError());
}