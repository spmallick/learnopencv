#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipblas.h>

#include "convolutional_layer.h"
#include "deconvolutional_layer.h"
#include "gemm.h"
#include "blas.h"
#include "im2col.h"
#include "col2im.h"
#include "utils.h"
#include "dark_cuda.h"

extern "C" void forward_deconvolutional_layer_gpu(deconvolutional_layer layer, network_state state)
{
    int i;
    int out_h = deconvolutional_out_height(layer);
    int out_w = deconvolutional_out_width(layer);
    int size = out_h*out_w;

    int m = layer.size*layer.size*layer.n;
    int n = layer.h*layer.w;
    int k = layer.c;

    fill_ongpu(layer.outputs*layer.batch, 0, layer.output_gpu, 1);

    for(i = 0; i < layer.batch; ++i){
        float *a = layer.weights_gpu;
        float *b = state.input + i*layer.c*layer.h*layer.w;
        float *c = layer.col_image_gpu;

        gemm_ongpu(1,0,m,n,k,1,a,m,b,n,0,c,n);

        col2im_ongpu(c, layer.n, out_h, out_w, layer.size, layer.stride, 0, layer.output_gpu+i*layer.n*size);
    }
    add_bias_gpu(layer.output_gpu, layer.biases_gpu, layer.batch, layer.n, size);
    activate_array(layer.output_gpu, layer.batch*layer.n*size, layer.activation);
}

extern "C" void backward_deconvolutional_layer_gpu(deconvolutional_layer layer, network_state state)
{
    float alpha = 1./layer.batch;
    int out_h = deconvolutional_out_height(layer);
    int out_w = deconvolutional_out_width(layer);
    int size = out_h*out_w;
    int i;

    gradient_array(layer.output_gpu, size*layer.n*layer.batch, layer.activation, layer.delta_gpu);
    backward_bias(layer.bias_updates_gpu, layer.delta, layer.batch, layer.n, size);

    if(state.delta) memset(state.delta, 0, layer.batch*layer.h*layer.w*layer.c*sizeof(float));

    for(i = 0; i < layer.batch; ++i){
        int m = layer.c;
        int n = layer.size*layer.size*layer.n;
        int k = layer.h*layer.w;

        float *a = state.input + i*m*n;
        float *b = layer.col_image_gpu;
        float *c = layer.weight_updates_gpu;

        im2col_ongpu(layer.delta_gpu + i*layer.n*size, layer.n, out_h, out_w,
                layer.size, layer.stride, 0, b);
        gemm_ongpu(0,1,m,n,k,alpha,a,k,b,k,1,c,n);

        if(state.delta){
            int m = layer.c;
            int n = layer.h*layer.w;
            int k = layer.size*layer.size*layer.n;

            float *a = layer.weights_gpu;
            float *b = layer.col_image_gpu;
            float *c = state.delta + i*n*m;

            gemm(0,0,m,n,k,1,a,k,b,n,1,c,n);
        }
    }
}

extern "C" void pull_deconvolutional_layer(deconvolutional_layer layer)
{
    cuda_pull_array(layer.weights_gpu, layer.weights, layer.c*layer.n*layer.size*layer.size);
    cuda_pull_array(layer.biases_gpu, layer.biases, layer.n);
    cuda_pull_array(layer.weight_updates_gpu, layer.weight_updates, layer.c*layer.n*layer.size*layer.size);
    cuda_pull_array(layer.bias_updates_gpu, layer.bias_updates, layer.n);
}

extern "C" void push_deconvolutional_layer(deconvolutional_layer layer)
{
    cuda_push_array(layer.weights_gpu, layer.weights, layer.c*layer.n*layer.size*layer.size);
    cuda_push_array(layer.biases_gpu, layer.biases, layer.n);
    cuda_push_array(layer.weight_updates_gpu, layer.weight_updates, layer.c*layer.n*layer.size*layer.size);
    cuda_push_array(layer.bias_updates_gpu, layer.bias_updates, layer.n);
}

extern "C" void update_deconvolutional_layer_gpu(deconvolutional_layer layer, int skip, float learning_rate, float momentum, float decay)
{
    int size = layer.size*layer.size*layer.c*layer.n;

    axpy_ongpu(layer.n, learning_rate, layer.bias_updates_gpu, 1, layer.biases_gpu, 1);
    scal_ongpu(layer.n, momentum, layer.bias_updates_gpu, 1);

    axpy_ongpu(size, -decay, layer.weights_gpu, 1, layer.weight_updates_gpu, 1);
    axpy_ongpu(size, learning_rate, layer.weight_updates_gpu, 1, layer.weights_gpu, 1);
    scal_ongpu(size, momentum, layer.weight_updates_gpu, 1);
}
