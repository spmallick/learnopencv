#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipblas.h>

#include "convolutional_layer.h"
#include "batchnorm_layer.h"
#include "gemm.h"
#include "blas.h"
#include "im2col.h"
#include "col2im.h"
#include "utils.h"
#include "dark_cuda.h"
#include "box.h"


__global__ void binarize_kernel(float *x, int n, float *binary)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= n) return;
    binary[i] = (x[i] >= 0) ? 1 : -1;
}

void binarize_gpu(float *x, int n, float *binary)
{
    binarize_kernel<<<cuda_gridsize(n), BLOCK, 0, get_cuda_stream() >>>(x, n, binary);
    CHECK_CUDA(hipPeekAtLastError());
}

__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    int s = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (s >= size) return;
    int i = 0;
    float mean = 0;
    for(i = 0; i < n; ++i){
        mean += fabs(input[i*size + s]);
    }
    mean = mean / n;
    for(i = 0; i < n; ++i){
        binary[i*size + s] = (input[i*size + s] > 0) ? mean : -mean;
    }
}

void binarize_input_gpu(float *input, int n, int size, float *binary)
{
    binarize_input_kernel<<<cuda_gridsize(size), BLOCK, 0, get_cuda_stream() >>>(input, n, size, binary);
    CHECK_CUDA(hipPeekAtLastError());
}

__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    int f = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (f >= n) return;
    int i = 0;
    float mean = 0;
    for (i = 0; i < size; ++i) {
        mean += fabs(weights[f*size + i]);
    }
    mean = mean / size;
    for (i = 0; i < size; ++i) {
        binary[f*size + i] = (weights[f*size + i] > 0) ? mean : -mean;
        //binary[f*size + i] = weights[f*size + i];
    }
}

void binarize_weights_gpu(float *weights, int n, int size, float *binary)
{
    binarize_weights_kernel << <cuda_gridsize(n), BLOCK, 0, get_cuda_stream() >> >(weights, n, size, binary);
    CHECK_CUDA(hipPeekAtLastError());
}


__global__ void set_zero_kernel(float *src, int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) src[i] = 0;
}

__inline__ __device__
float warpAllReduceSum(float val) {
    for (int mask = WARP_SIZE / 2; mask > 0; mask /= 2)
#if CUDART_VERSION >= 9000
        val += __shfl_xor_sync(0xffffffff, val, mask);
#else
        val += __shfl_xor(val, mask);
#endif
    return val;
}

// only if (size % 32 == 0)
__global__ void reduce_kernel(float *weights, int n, int size, float *mean_arr_gpu)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int f = i / size;
    if (f >= n) return;
    float warp_mean = warpAllReduceSum(fabs(weights[i]));
    if(i % 32 == 0)
        atomicAdd(&mean_arr_gpu[f], warp_mean / size);
}

__global__ void binarize_weights_mean_kernel(float *weights, int n, int size, float *binary, float *mean_arr_gpu)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int f = i / size;
    if (f >= n) return;
    float mean = mean_arr_gpu[f];
    binary[i] = (weights[i] > 0) ? mean : -mean;
}

void fast_binarize_weights_gpu(float *weights, int n, int size, float *binary, float *mean_arr_gpu)
{
    if (size % 32 == 0) {
        size_t gridsize = n * size;
        const int num_blocks = get_number_of_blocks(gridsize, BLOCK);// gridsize / BLOCK + 1;

        set_zero_kernel << <(n/BLOCK + 1), BLOCK, 0, get_cuda_stream() >> > (mean_arr_gpu, n);
        reduce_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> > (weights, n, size, mean_arr_gpu);
        binarize_weights_mean_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> > (weights, n, size, binary, mean_arr_gpu);
        CHECK_CUDA(hipPeekAtLastError());
    }
    else {
        binarize_weights_gpu(weights, n, size, binary);
    }
}


__global__ void cuda_f32_to_f16(float* input_f32, size_t size, half *output_f16)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) output_f16[idx] = __float2half(input_f32[idx]);
    //if (idx < size) output_f16[idx] = __float2half_rn(input_f32[idx]); // can't be compiled on Linux without casting
    // __float2half_ru, __float2half_rd, __float2half_rz, __float2half_rn
    //if (idx < size) *((unsigned short *)output_f16 + idx) = __float2half(input_f32[idx]);
}

void cuda_convert_f32_to_f16(float* input_f32, size_t size, float *output_f16) {
    cuda_f32_to_f16 <<< get_number_of_blocks(size, BLOCK), BLOCK, 0, get_cuda_stream() >>> (input_f32, size, (half *)output_f16);
    CHECK_CUDA(hipPeekAtLastError());
}

__global__ void cuda_f16_to_f32(half* input_f16, size_t size, float *output_f32)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) output_f32[idx] = __half2float(input_f16[idx]);
    //if (idx < size) output_f32[idx] = __half2float(*((unsigned short *)input_f16 + idx));
}

void cuda_convert_f16_to_f32(float* input_f16, size_t size, float *output_f32) {
    cuda_f16_to_f32 <<< get_number_of_blocks(size, BLOCK), BLOCK, 0, get_cuda_stream() >>> ((half *)input_f16, size, output_f32);
    CHECK_CUDA(hipPeekAtLastError());
}

half *cuda_make_f16_from_f32_array(float *src, size_t n)
{
    half *dst16;
    size_t size = sizeof(half)*n;
    CHECK_CUDA(hipMalloc((void **)&dst16, size));
    if (src) {
        assert(n > 0);
        cuda_convert_f32_to_f16(src, n, (float *)dst16);
    }
    if (!dst16) error("Cuda malloc failed", DARKNET_LOC);
    return dst16;
}

void forward_convolutional_layer_gpu(convolutional_layer l, network_state state)
{
    if (l.train == 0) state.train = 0;

    if (l.stream >= 0) {
        switch_stream(l.stream);
    }

    if (l.wait_stream_id >= 0) {
        wait_stream(l.wait_stream_id);
    }

    //fill_ongpu(l.outputs*l.batch, 0, l.output_gpu, 1);
    if(l.binary){
        binarize_weights_gpu(l.weights_gpu, l.n, (l.c / l.groups)*l.size*l.size, l.binary_weights_gpu);
        swap_binary(&l);
    }

    if(l.xnor){
        if (!l.align_bit_weights_gpu || state.train) {
            //binarize_weights_gpu(l.weights_gpu, l.n, (l.c / l.groups)*l.size*l.size, l.binary_weights_gpu);

            fast_binarize_weights_gpu(l.weights_gpu, l.n, (l.c / l.groups)*l.size*l.size, l.binary_weights_gpu, l.mean_arr_gpu);
        }

        if (l.align_bit_weights_gpu && !state.train && l.c >= 32 && l.stride_x == l.stride_y)
        {
            //return;
            //hipError_t status = hipSuccess;
            //int input_size = l.c*l.h*l.w*l.batch;

            int m = l.n / l.groups;
            int k = l.size*l.size*l.c / l.groups;
            int n = l.out_w*l.out_h;
            //float * a = l.weights_gpu;

            // int i, j;
            // for(i = 0; i < l.batch; ++i){
            // for (j = 0; j < l.groups; ++j) {

            int ldb_align = l.lda_align;
            size_t new_ldb = k + (ldb_align - k%ldb_align); // (k / 8 + 1) * 8;
            //size_t t_intput_size = new_ldb * n;
            //size_t t_bit_input_size = t_intput_size / 8;// +1;

            if (l.c % 32 == 0)
            {
                //printf("\n\n l.index = %d, l.w = %d, l.c = %d, l.n = %d, l.stride = %d, l.pad = %d - new XNOR \n", l.index, l.w, l.c, l.n, l.stride, l.pad);
                //printf("l.align_workspace_size = %d, (l.c * l.w * l.h)  = %d \n", l.align_workspace_size, (l.c * l.w * l.h));

                //float *intput_cpu = (float *)calloc(l.inputs, sizeof(float));
                // state.input
                //hipMemcpy(intput_cpu, state.input, l.inputs * sizeof(float), hipMemcpyDefault);

                int ldb_align = l.lda_align;
                size_t new_ldb = k + (ldb_align - k%ldb_align); // (k / 8 + 1) * 8;
                //size_t t_intput_size = new_ldb * l.bit_align;// n;
                //size_t t_bit_input_size = t_intput_size / 8;// +1;

                const int new_c = l.c / 32;

                //float *re_packed_input = (float *)calloc(l.c * l.w * l.h, sizeof(float));
                //uint32_t *bin_re_packed_input = (uint32_t *)calloc(new_c * l.w * l.h + 1, sizeof(uint32_t));

                // float32x4 by channel (as in cuDNN)
                //repack_input(intput_cpu, re_packed_input, l.w, l.h, l.c);


                // 32 x floats -> 1 x uint32_t
                //float_to_bit(re_packed_input, (uint8_t *)bin_re_packed_input, l.c * l.w * l.h);

                //hipDeviceSynchronize();
                //start_timer();

                repack_input_gpu_bin(state.input, (uint32_t *)l.align_workspace_gpu, l.w, l.h, l.c);

                //repack_input_gpu(state.input, state.workspace, l.w, l.h, l.c);

                // 32 x floats -> 1 x uint32_t
                //float_to_bit_gpu(state.workspace, (unsigned char *)l.align_workspace_gpu, l.c * l.w * l.h);// l.align_workspace_size);

                //hipDeviceSynchronize();
                //stop_timer_and_show_name("repack_input_gpu + float_to_bit_gpu");

                //free(re_packed_input);

                // slow - convolution the packed inputs and weights: float x 32 by channel (as in cuDNN)
                //convolution_repacked((uint32_t *)bin_re_packed_input, (uint32_t *)l.align_bit_weights, l.output,
                //    l.w, l.h, l.c, l.n, l.size, l.pad, l.new_lda, l.mean_arr);

                // // then exit from if()

                //float *b = state.workspace;
                //float *b = (float *)calloc(100 * 1024 * 1024, sizeof(float));
                //float *c = l.output;
                //memset(c, 0, l.outputs * sizeof(float));


                //im2col_cpu_custom((float *)bin_re_packed_input, new_c, l.h, l.w, l.size, l.stride, l.pad, b);

                //hipMemcpy(l.align_workspace_gpu, bin_re_packed_input, (new_c * l.w * l.h + 1) * sizeof(uint32_t), hipMemcpyDefault);

                //start_timer();
                im2col_ongpu(l.align_workspace_gpu, new_c, l.h, l.w, l.size, l.stride, l.pad, state.workspace);
                //hipDeviceSynchronize();
                //stop_timer_and_show_name("im2col_ongpu");

                //free(bin_re_packed_input);

                int new_k = l.size*l.size*l.c / 32;

                // good for (l.c == 64)
                //gemm_nn_bin_32bit_packed(m, n, new_k, 1,
                //    l.align_bit_weights, l.new_lda/32,
                //    b, n,
                //    c, n, l.mean_arr);

                // // then exit from if()


                //size_t new_ldb = k + (ldb_align - k%ldb_align); // (k / 8 + 1) * 8;
                //size_t t_intput_size = new_ldb * l.bit_align;// n;
                //size_t t_bit_input_size = t_intput_size / 8;// +1;

                //char *t_bit_input = (char *)calloc(t_bit_input_size, sizeof(char));
                //transpose_uint32((uint32_t *)b, (uint32_t *)t_bit_input, new_k, n, n, new_ldb);
                //hipMemcpy(l.transposed_align_workspace_gpu, t_bit_input, t_bit_input_size * sizeof(char), hipMemcpyDefault);

                //hipMemcpy(state.workspace, b, t_bit_input_size * sizeof(char), hipMemcpyDefault);
                //printf("\n n = %d, n % 32 = %d, new_ldb = %d, new_ldb % 32 = %d \n", n, n % 32, new_ldb, new_ldb % 32);

                //start_timer();
                transpose_uint32_gpu((uint32_t *)state.workspace, (uint32_t *)l.transposed_align_workspace_gpu, new_k, n, n, new_ldb);
                //hipDeviceSynchronize();
                //stop_timer_and_show_name("transpose_uint32_gpu");

                //hipDeviceSynchronize();
                //stop_timer_and_show_name("repack_input_gpu_bin + im2col_ongpu + transpose_uint32_gpu_2");

                //start_timer();
                gemm_nn_custom_bin_mean_transposed_gpu(m, n, k,
                    (unsigned char *)l.align_bit_weights_gpu, new_ldb, (unsigned char *)l.transposed_align_workspace_gpu,
                    new_ldb, l.output_gpu, n, l.mean_arr_gpu, l.biases_gpu, l.activation == LEAKY,
                    l.bin_conv_shortcut_in_gpu, l.bin_conv_shortcut_out_gpu);
                //hipDeviceSynchronize();
                //stop_timer_and_show_name("gemm_nn_custom_bin_mean_transposed_gpu");


                // the main GEMM function
                //gemm_nn_custom_bin_mean_transposed(m, n, k, 1, (uint8_t *)l.align_bit_weights, new_ldb, (uint8_t *)t_bit_input, new_ldb, c, n, l.mean_arr);

                //add_bias(l.output, l.biases, l.batch, l.n, l.out_h*l.out_w);

                //hipMemcpy(l.output_gpu, l.output, l.outputs * sizeof(float), hipMemcpyDefault);


                // // alternative GEMM
                //gemm_nn_bin_transposed_32bit_packed(m, n, new_k, 1,
                //    l.align_bit_weights, l.new_lda/32,
                //    t_bit_input, new_ldb / 32,
                //    c, n, l.mean_arr);

                //free(t_bit_input);

                //free(b);
            }
            else
            {
                //printf("\n\n l.index = %d, l.w = %d, l.c = %d, l.n = %d, l.stride = %d, l.pad = %d - old XNOR \n", l.index, l.w, l.c, l.n, l.stride, l.pad);
                //hipDeviceSynchronize();

                int i = 0;
                /*
                // if (l.stride == 1 && l.c >= 256 && l.size > 1)
                if (l.stride == 1 && l.c >= 1024 && l.size > 1 && 0)// && l.w >= 13) // disabled
                {
                    // stride=1 only
                    //start_timer();
                    im2col_align_bin_ongpu(state.input + i*l.c*l.h*l.w, l.c, l.h, l.w, l.size, l.stride, l.pad, state.workspace, l.bit_align);
                    //hipDeviceSynchronize();
                    //stop_timer_and_show_name("im2col_align_bin_ongpu");
                }
                else*/
                {
                    //start_timer();
                    im2col_align_ongpu(state.input + i*l.c*l.h*l.w, l.c, l.h, l.w, l.size, l.stride, l.pad, l.align_workspace_gpu, l.bit_align);
                    //hipDeviceSynchronize();
                    //stop_timer_and_show_name("im2col_align_ongpu");
                    //getchar();

                    // should be optimized
                    //start_timer();
                    float_to_bit_gpu(l.align_workspace_gpu, (unsigned char *)state.workspace, l.align_workspace_size);
                    //hipDeviceSynchronize();
                    //stop_timer_and_show_name("float_to_bit_gpu");
                }
                //start_timer();
                transpose_bin_gpu((unsigned char *)state.workspace, (unsigned char *)l.transposed_align_workspace_gpu, k, n, l.bit_align, new_ldb, 8);
                //hipDeviceSynchronize();
                //stop_timer_and_show_name("transpose_bin_gpu");

                //hipDeviceSynchronize();
                //stop_timer_and_show_name("im2col_align_ongpu + float_to_bit_gpu + transpose_bin_gpu");

                // should be optimized
                //if(0) {//if (k > 1000) {    // sequentially input-shared - BAD
                //    gemm_nn_custom_bin_mean_transposed_sequentially_gpu(m, n, k,
                //        (unsigned char *)l.align_bit_weights_gpu, new_ldb, (unsigned char *)l.transposed_align_workspace_gpu, new_ldb, l.output_gpu, n, l.mean_arr_gpu);
                //}
                //else {  // coalescing & weights-shared-memory - GOOD
                    //start_timer();
                    gemm_nn_custom_bin_mean_transposed_gpu(m, n, k,
                        (unsigned char *)l.align_bit_weights_gpu, new_ldb, (unsigned char *)l.transposed_align_workspace_gpu,
                        new_ldb, l.output_gpu, n, l.mean_arr_gpu, l.biases_gpu, l.activation == LEAKY,
                        l.bin_conv_shortcut_in_gpu, l.bin_conv_shortcut_out_gpu);
                    //hipDeviceSynchronize();
                    //stop_timer_and_show_name("gemm_nn_custom_bin_mean_transposed_gpu");
                //}
                //hipDeviceSynchronize();
                //check_error(status);
                //getchar();
            }


            /*
            {
                float_to_bit_gpu(state.input, (unsigned char *)l.align_workspace_gpu, input_size);
                convolve_bin_gpu(l.align_workspace_gpu, (float *)l.align_bit_weights_gpu, l.output_gpu, l.w, l.h, l.c, l.n, l.size, l.pad, l.new_lda, l.mean_arr_gpu);

                //convolve_gpu(state.input, l.weights_gpu, l.output_gpu, l.w, l.h, l.c, l.n, l.size, l.pad);

                //hipDeviceSynchronize();
                //check_error(status);

                add_bias_gpu(l.output_gpu, l.biases_gpu, l.batch, l.n, l.out_w*l.out_h);
            }
            */

            //add_bias_gpu(l.output_gpu, l.biases_gpu, l.batch, l.n, l.out_w*l.out_h);
            if (l.activation == SWISH) activate_array_swish_ongpu(l.output_gpu, l.outputs*l.batch, l.activation_input_gpu, l.output_gpu);
            else if (l.activation == MISH) activate_array_mish_ongpu(l.output_gpu, l.outputs*l.batch, l.activation_input_gpu, l.output_gpu);
            else if (l.activation == HARD_MISH) activate_array_hard_mish_ongpu(l.output_gpu, l.outputs*l.batch, l.activation_input_gpu, l.output_gpu);
            else if (l.activation == NORM_CHAN) activate_array_normalize_channels_ongpu(l.output_gpu, l.outputs*l.batch, l.batch, l.out_c, l.out_w*l.out_h, l.output_gpu);
            else if (l.activation == NORM_CHAN_SOFTMAX) activate_array_normalize_channels_softmax_ongpu(l.output_gpu, l.outputs*l.batch, l.batch, l.out_c, l.out_w*l.out_h, l.output_gpu, 0);
            else if (l.activation == NORM_CHAN_SOFTMAX_MAXVAL) activate_array_normalize_channels_softmax_ongpu(l.output_gpu, l.outputs*l.batch, l.batch, l.out_c, l.out_w*l.out_h, l.output_gpu, 1);
            else if (l.activation != LINEAR && l.activation != LEAKY) activate_array_ongpu(l.output_gpu, l.outputs*l.batch, l.activation);
            //if(l.activation != LINEAR && l.activation != LEAKY) activate_array_ongpu(l.output_gpu, l.outputs*l.batch, l.activation);
            //if (l.binary || l.xnor) swap_binary(&l);
            //hipDeviceSynchronize();
            return;
        }
    }

    if (l.xnor) {
        swap_binary(&l);
        binarize_gpu(state.input, l.c*l.h*l.w*l.batch, l.binary_input_gpu);
        state.input = l.binary_input_gpu;
    }

    //fill_ongpu(l.outputs*l.batch, 0, l.output_gpu, 1);

#ifdef CUDNN
    //float one = 1;    // alpha[0], beta[0] is float for HALF and FLOAT
    float alpha = 1, beta = 0;

//#ifdef CUDNN_HALF
    //if (state.use_mixed_precision) {
    int iteration_num = get_current_iteration(state.net); // (*state.net.seen) / (state.net.batch*state.net.subdivisions);
    if (state.index != 0 && state.net.cudnn_half && !l.xnor && (!state.train || (iteration_num > 3 * state.net.burn_in) && state.net.loss_scale != 1) &&
        (l.c / l.groups) % 8 == 0 && l.n % 8 == 0 && l.groups <= 1 && l.size > 1)
    {
        //printf("\n CUDNN_HALF!!! state.index = %d \n", state.index);

        // Note: For improved performance it is advised to use beta[0] = 0.0.
        // For Tensor Core: hipdnnSetConvolutionMathType() where hipdnnMathType_t mathType = HIPDNN_TENSOR_OP_MATH;
        // 1. or HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM and use HIPDNN_DATA_HALF
        // 2. or HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED
        // More: http://docs.nvidia.com/deeplearning/sdk/cudnn-developer-guide/index.html#tensor_ops

        const size_t input16_size = l.batch*l.c*l.w*l.h;
        const size_t output16_size = l.batch*l.out_c*l.out_h*l.out_w;

        if (*state.net.max_input16_size < input16_size) {
            //printf("\n input16_size: cur = %zu \t max = %zu \n", input16_size, *state.net.max_input16_size);
            *state.net.max_input16_size = input16_size;
            if (*state.net.input16_gpu) cuda_free(*state.net.input16_gpu);
            assert(*state.net.max_input16_size > 0);
            *state.net.input16_gpu = (float *)cuda_make_f16_from_f32_array(NULL, *state.net.max_input16_size);
        }
        float *input16 = *state.net.input16_gpu;

        if (*state.net.max_output16_size < output16_size) {
            *state.net.max_output16_size = output16_size;
            if (*state.net.output16_gpu) cuda_free(*state.net.output16_gpu);
            assert(*state.net.max_output16_size > 0);
            *state.net.output16_gpu = (float *)cuda_make_f16_from_f32_array(NULL, *state.net.max_output16_size);
        }
        float *output16 = *state.net.output16_gpu;

        assert(input16_size > 0);
        cuda_convert_f32_to_f16(state.input, input16_size, input16);

        //fill_ongpu(output16_size / 2, 0, (float *)output16, 1);
        CHECK_CUDNN(hipdnnConvolutionForward(cudnn_handle(),
            &alpha,
            l.srcTensorDesc16,
            input16,
            l.weightDesc16,
            l.weights_gpu16,
            l.convDesc,
            l.fw_algo16,
            state.workspace,
            l.workspace_size,
            &beta,
            l.dstTensorDesc16,
            output16));


        if (l.batch_normalize)
        {
            if (state.train && !state.net.adversarial) // Training
            {
                simple_copy_ongpu(l.outputs*l.batch / 2, output16, l.x_gpu);
                //copy_ongpu(l.outputs*l.batch / 2, output16, 1, l.x_gpu, 1);
                //hipMemcpyAsync(l.x_gpu, output16, l.outputs*l.batch*sizeof(half), hipMemcpyDefault, get_cuda_stream());
                float one = 1.0f;
                float zero = 0.0f;
                // Batch-normalization can still take FP16 inputs and outputs, saving half the bandwidth
                // compared to FP32, it's just that the statistics and value adjustment should be done in FP32.
                CHECK_CUDNN(hipdnnBatchNormalizationForwardTraining(cudnn_handle(),
                    HIPDNN_BATCHNORM_SPATIAL,
                    &one,
                    &zero,
                    l.normDstTensorDescF16,
                    l.x_gpu,            // input
                    l.normDstTensorDescF16,
                    output16,            // output
                    l.normTensorDesc,
                    l.scales_gpu,       // input
                    l.biases_gpu,       // input
                    .01,
                    l.rolling_mean_gpu,        // input/output (should be FP32)
                    l.rolling_variance_gpu,    // input/output (should be FP32)
                    .00001,
                    l.mean_gpu,            // output (should be FP32) - optional cache to speedup hipdnnBatchNormalizationBackward()
                    l.variance_gpu));    // output (should be FP32) - optional cache to speedup hipdnnBatchNormalizationBackward()

                cuda_convert_f16_to_f32(output16, output16_size, l.output_gpu);
                //forward_batchnorm_layer_gpu(l, state);
            }
            else // Detection
            {
                cuda_convert_f16_to_f32(output16, output16_size, l.output_gpu);
                normalize_gpu(l.output_gpu, l.rolling_mean_gpu, l.rolling_variance_gpu, l.batch, l.out_c, l.out_h*l.out_w);
                scale_bias_gpu(l.output_gpu, l.scales_gpu, l.batch, l.out_c, l.out_h*l.out_w);
                add_bias_gpu(l.output_gpu, l.biases_gpu, l.batch, l.out_c, l.out_w*l.out_h);
            }
        }
        else // BIAS only
        {
            cuda_convert_f16_to_f32(output16, output16_size, l.output_gpu);
            add_bias_gpu(l.output_gpu, l.biases_gpu, l.batch, l.n, l.out_w*l.out_h);
        }
    }
    else {

        //#else
        /*
        int input_nan_inf = is_nan_or_inf(state.input, l.inputs * l.batch);
        printf("\n is_nan_or_inf(state.input) = %d \n", input_nan_inf);
        if (input_nan_inf) getchar();

        int weights_nan_inf = is_nan_or_inf(l.weights_gpu, l.nweights);
        printf("\n is_nan_or_inf(l.weights_gpu) = %d \n", weights_nan_inf);
        if (weights_nan_inf) getchar();
        */

        CHECK_CUDNN(hipdnnConvolutionForward(cudnn_handle(),
            &alpha, //&one,
            l.srcTensorDesc,
            state.input,
            l.weightDesc,
            l.weights_gpu,
            l.convDesc,
            l.fw_algo,
            state.workspace,
            l.workspace_size,
            &beta,  //&one,
            l.dstTensorDesc,
            l.output_gpu));

        //hipDeviceSynchronize();
        if (l.batch_normalize) {
            forward_batchnorm_layer_gpu(l, state);
        }
        else {
            add_bias_gpu(l.output_gpu, l.biases_gpu, l.batch, l.n, l.out_w*l.out_h);
        }
    //#endif    // CUDNN_HALF
    }


#else
    fill_ongpu(l.outputs*l.batch, 0, l.output_gpu, 1);

    int i, j;
    int m = l.n / l.groups;
    int k = l.size*l.size*l.c / l.groups;
    int n = l.out_w*l.out_h;
    for(i = 0; i < l.batch; ++i){
        for (j = 0; j < l.groups; ++j) {
            //float *im = state.input + i*l.c*l.h*l.w;
            float *im = state.input + (i*l.groups + j)*l.c / l.groups*l.h*l.w;
            float *a = l.weights_gpu + j*l.nweights / l.groups;
            float *b = state.workspace;
            float *c = l.output_gpu + (i*l.groups + j)*n*m;
            if (l.size == 1 && l.stride == 1 && l.dilation == 1) {
                b = im;
            }
            else {
                //im2col_ongpu(im, l.c / l.groups, l.h, l.w, l.size, l.stride, l.pad, state.workspace);

                im2col_gpu_ext(im,          // input
                    l.c / l.groups,         // input channels
                    l.h, l.w,               // input size (h, w)
                    l.size, l.size,         // kernel size (h, w)
                    l.pad * l.dilation, l.pad * l.dilation,   // padding (h, w)
                    l.stride_y, l.stride_x,     // stride (h, w)
                    l.dilation, l.dilation, // dilation (h, w)
                    state.workspace);       // output

            }
            //gemm_ongpu(0, 0, m, n, k, 1., a, k, b, n, 1., c + i*m*n, n);
            gemm_ongpu(0, 0, m, n, k, 1, a, k, b, n, 1, c, n);
        }
    }

    if (l.batch_normalize) {
        forward_batchnorm_layer_gpu(l, state);
    }
    else {
        add_bias_gpu(l.output_gpu, l.biases_gpu, l.batch, l.n, l.out_w*l.out_h);
    }
#endif

//#ifndef CUDNN_HALF
//#endif // no CUDNN_HALF

    if (l.activation == SWISH) activate_array_swish_ongpu(l.output_gpu, l.outputs*l.batch, l.activation_input_gpu, l.output_gpu);
    else if (l.activation == MISH) activate_array_mish_ongpu(l.output_gpu, l.outputs*l.batch, l.activation_input_gpu, l.output_gpu);
    else if (l.activation == HARD_MISH) activate_array_hard_mish_ongpu(l.output_gpu, l.outputs*l.batch, l.activation_input_gpu, l.output_gpu);
    else if (l.activation == NORM_CHAN) activate_array_normalize_channels_ongpu(l.output_gpu, l.outputs*l.batch, l.batch, l.out_c, l.out_w*l.out_h, l.output_gpu);
    else if (l.activation == NORM_CHAN_SOFTMAX) activate_array_normalize_channels_softmax_ongpu(l.output_gpu, l.outputs*l.batch, l.batch, l.out_c, l.out_w*l.out_h, l.output_gpu, 0);
    else if (l.activation == NORM_CHAN_SOFTMAX_MAXVAL) activate_array_normalize_channels_softmax_ongpu(l.output_gpu, l.outputs*l.batch, l.batch, l.out_c, l.out_w*l.out_h, l.output_gpu, 1);
    else if (l.activation != LINEAR) activate_array_ongpu(l.output_gpu, l.outputs*l.batch, l.activation);
    //if(l.dot > 0) dot_error_gpu(l);
    if(l.binary || l.xnor) swap_binary(&l);
    //hipDeviceSynchronize();    // for correct profiling of performance

    if (state.net.try_fix_nan) {
        fix_nan_and_inf(l.output_gpu, l.outputs*l.batch);
    }

    if(l.assisted_excitation && state.train) assisted_excitation_forward_gpu(l, state);

    if (l.antialiasing) {
        network_state s = { 0 };
        s.train = state.train;
        s.workspace = state.workspace;
        s.net = state.net;
        if (!state.train) s.index = state.index;  // don't use TC for training (especially without cuda_convert_f32_to_f16() )
        s.input = l.output_gpu;
        forward_convolutional_layer_gpu(*(l.input_layer), s);
        simple_copy_ongpu(l.outputs*l.batch, l.output_gpu, l.input_antialiasing_gpu);
        simple_copy_ongpu(l.input_layer->outputs*l.input_layer->batch, l.input_layer->output_gpu, l.output_gpu);
    }

    if (l.coordconv) {
        coord_conv_gpu(l.output_gpu, l.outputs*l.batch, l.out_w, l.out_h, l.out_c, l.batch, 0);
    }
}

void backward_convolutional_layer_gpu(convolutional_layer l, network_state state)
{
    if (l.coordconv) {
        coord_conv_gpu(l.delta_gpu, l.outputs*l.batch, l.out_w, l.out_h, l.out_c, l.batch, 1);
    }

    if (l.antialiasing) {
        network_state s = { 0 };
        s.train = state.train;
        s.workspace = state.workspace;
        s.net = state.net;
        s.delta = l.delta_gpu;  // s.delta will be returned to l.delta_gpu
        s.input = l.input_antialiasing_gpu;
        //if (!state.train) s.index = state.index;  // don't use TC for training (especially without cuda_convert_f32_to_f16() )
        simple_copy_ongpu(l.input_layer->outputs*l.input_layer->batch, l.delta_gpu, l.input_layer->delta_gpu);
        backward_convolutional_layer_gpu(*(l.input_layer), s);

        simple_copy_ongpu(l.outputs*l.batch, l.input_antialiasing_gpu, l.output_gpu);
    }

    if(state.net.try_fix_nan) constrain_ongpu(l.outputs*l.batch, 1, l.delta_gpu, 1);

    if (l.activation == SWISH) gradient_array_swish_ongpu(l.output_gpu, l.outputs*l.batch, l.activation_input_gpu, l.delta_gpu);
    else if (l.activation == MISH) gradient_array_mish_ongpu(l.outputs*l.batch, l.activation_input_gpu, l.delta_gpu);
    else if (l.activation == HARD_MISH) gradient_array_hard_mish_ongpu(l.outputs*l.batch, l.activation_input_gpu, l.delta_gpu);
    else if (l.activation == NORM_CHAN_SOFTMAX || l.activation == NORM_CHAN_SOFTMAX_MAXVAL) gradient_array_normalize_channels_softmax_ongpu(l.output_gpu, l.outputs*l.batch, l.batch, l.out_c, l.out_w*l.out_h, l.delta_gpu);
    else if (l.activation == NORM_CHAN) gradient_array_normalize_channels_ongpu(l.output_gpu, l.outputs*l.batch, l.batch, l.out_c, l.out_w*l.out_h, l.delta_gpu);
    else gradient_array_ongpu(l.output_gpu, l.outputs*l.batch, l.activation, l.delta_gpu);

    if (!l.batch_normalize)
        backward_bias_gpu(l.bias_updates_gpu, l.delta_gpu, l.batch, l.n, l.out_w*l.out_h);

//#ifndef CUDNN_HALF
    //if(l.batch_normalize){
    //    backward_batchnorm_layer_gpu(l, state);
    //} else {
    //    //backward_bias_gpu(l.bias_updates_gpu, l.delta_gpu, l.batch, l.n, l.out_w*l.out_h);
    //}
//#endif // no CUDNN_HALF
    float *original_input = state.input;

    if(l.xnor) state.input = l.binary_input_gpu;
#ifdef CUDNN
    float one = 1.f;
    float alpha = 1, beta = 0;

//#ifdef CUDNN_HALF
    int iteration_num = get_current_iteration(state.net); //(*state.net.seen) / (state.net.batch*state.net.subdivisions);
    if (state.index != 0 && state.net.cudnn_half && !l.xnor && (!state.train || (iteration_num > 3 * state.net.burn_in) && state.net.loss_scale != 1) &&
        (l.c / l.groups) % 8 == 0 && l.n % 8 == 0  && l.groups <= 1 && l.size > 1)
    {
        const size_t input16_size = l.batch*l.c*l.w*l.h;
        const size_t delta16_size = l.batch*l.n*l.out_w*l.out_h;

        if (*state.net.max_input16_size < input16_size) {
            *state.net.max_input16_size = input16_size;
            if (*state.net.input16_gpu) cuda_free(*state.net.input16_gpu);
            assert(*state.net.max_input16_size > 0);
            *state.net.input16_gpu = (float *)cuda_make_f16_from_f32_array(NULL, *state.net.max_input16_size);
        }
        float *input16 = *state.net.input16_gpu;

        if (*state.net.max_output16_size < delta16_size) {
            *state.net.max_output16_size = delta16_size;
            if (*state.net.output16_gpu) cuda_free(*state.net.output16_gpu);
            assert(*state.net.max_output16_size > 0);
            *state.net.output16_gpu = (float *)cuda_make_f16_from_f32_array(NULL, *state.net.max_output16_size);
        }
        float *delta16 = *state.net.output16_gpu;

        assert(input16_size > 0);
        assert(delta16_size > 0);
        cuda_convert_f32_to_f16(state.input, input16_size, input16);
        cuda_convert_f32_to_f16(l.delta_gpu, delta16_size, delta16);

        if (l.batch_normalize) {
            //if (!state.train) {
            //    l.mean_gpu = l.rolling_mean_gpu;
            //    l.variance_gpu = l.rolling_variance_gpu;
            //}
            float one = 1.0f;
            float zero = 0.0f;
            CHECK_CUDNN(hipdnnBatchNormalizationBackward(cudnn_handle(),
                HIPDNN_BATCHNORM_SPATIAL,
                &one,
                &zero,
                &one,
                &one,
                l.normDstTensorDescF16,
                l.x_gpu,                // input (input in BN-forward-inference)
                l.normDstTensorDescF16,
                delta16,                // input
                l.normDstTensorDescF16,
                l.output_gpu, //l.x_norm_gpu,            // output (new delta)
                l.normTensorDesc,
                l.scales_gpu,            // input (should be FP32)
                l.scale_updates_gpu,    // output (should be FP32)
                l.bias_updates_gpu,        // output (should be FP32)
                .00001,
                l.mean_gpu,                // input (should be FP32)
                l.variance_gpu));        // input (should be FP32)

            simple_copy_ongpu(l.outputs*l.batch / 2, l.output_gpu, delta16);
            //copy_ongpu(l.outputs*l.batch / 2, l.x_norm_gpu, 1, delta16, 1);
            //hipMemcpyAsync(delta16, l.x_norm_gpu, l.outputs*l.batch * sizeof(half), hipMemcpyDefault, get_cuda_stream());
        }
        else
        {
            //backward_bias_gpu(l.bias_updates_gpu, l.delta_gpu, l.batch, l.n, l.out_w*l.out_h);
        }

        // convert input: state.input (x), l.delta_gpu (y) from fp32 to fp16
        // get output: l.weight_updates_gpu (dw) and convert it to fp32 (ONLY if it is fp16)

        // calculate conv weight updates
        // Already: l.weight_updates_gpu = (l.weight_updates_gpu - l.weight*decay*batch*subdivision)*momentum
        //   so we should copy f32 to f16, or compute: f16=(w_up - w*d*b*s)*m
        assert((l.nweights) > 0);
        cuda_convert_f32_to_f16(l.weight_updates_gpu, l.nweights, l.weight_updates_gpu16);

        if (!state.net.adversarial && !l.train_only_bn) {
            CHECK_CUDNN(hipdnnConvolutionBackwardFilter(cudnn_handle(),
                &one,
                l.srcTensorDesc16,
                input16, //state.input,
                l.ddstTensorDesc16,
                delta16, //l.delta_gpu,
                l.convDesc,
                l.bf_algo16,
                state.workspace,
                l.workspace_size,
                &one,
                l.dweightDesc16,
                l.weight_updates_gpu16));    // l.weight_updates_gpu);

            cuda_convert_f16_to_f32(l.weight_updates_gpu16, l.nweights, l.weight_updates_gpu);
        }

        if (state.delta) {
            if (l.binary || l.xnor) swap_binary(&l);

            // http://docs.nvidia.com/deeplearning/sdk/cudnn-developer-guide/index.html#hipdnnConvolutionBackwardData
            // calculate delta for the next layer
            // convert input: l.weights_gpu (w), l.delta_gpu (dy) from fp32 to fp16
            // get output: state.delta (dx) and convert it to fp32 (ONLY if it is fp16)
            CHECK_CUDNN(hipdnnConvolutionBackwardData(cudnn_handle(),
                &alpha,
                l.weightDesc16,
                l.weights_gpu16, //l.weights_gpu,
                l.ddstTensorDesc16,
                delta16, //l.delta_gpu,
                l.convDesc,
                l.bd_algo16,
                state.workspace,
                l.workspace_size,
                &beta,
                l.dsrcTensorDesc16,
                input16));    // state.delta);

            cuda_convert_f16_to_f32(input16, input16_size, state.delta);

            if (l.binary || l.xnor) swap_binary(&l);
            if (l.xnor) gradient_array_ongpu(original_input, l.batch*l.c*l.h*l.w, HARDTAN, state.delta);
        }
    }
    else {
        //#else    // CUDNN_HALF

        if(l.batch_normalize){
            backward_batchnorm_layer_gpu(l, state);
        }

        if (!state.net.adversarial && !l.train_only_bn) {

            float *old_input = state.input;

            /*
            if (l.reverse) {
                if (*state.net.max_output16_size < l.inputs*l.batch) {
                    *state.net.max_output16_size = l.inputs*l.batch;
                    if (*state.net.output16_gpu) cuda_free(*state.net.output16_gpu);
                    assert(*state.net.max_output16_size > 0);
                    *state.net.output16_gpu = cuda_make_array(NULL, *state.net.max_output16_size);
                }
                float clip = 0.0;
                float divider = 1.0;
                float abs_add = 1.0;
                mult_inverse_array_gpu(state.input, *state.net.output16_gpu, l.inputs*l.batch, l.reverse, divider, clip, abs_add);
                state.input = *state.net.output16_gpu;
            }
            */

            // calculate conv weight updates
            // if used: beta=1 then loss decreases faster
            CHECK_CUDNN(hipdnnConvolutionBackwardFilter(cudnn_handle(),
                &one,
                l.srcTensorDesc,
                state.input,
                l.ddstTensorDesc,
                l.delta_gpu,
                l.convDesc,
                l.bf_algo,
                state.workspace,
                l.workspace_size,
                &one,
                l.dweightDesc,
                l.weight_updates_gpu));

            state.input = old_input;
        }


        if (state.delta) {
            if (l.binary || l.xnor) swap_binary(&l);

            float *old_weights = l.weights_gpu;

            /*
            if (l.reverse) {
                if (*state.net.max_output16_size < l.nweights) {
                    *state.net.max_output16_size = l.nweights;
                    if (*state.net.output16_gpu && *state.net.max_output16_size > 0) cuda_free(*state.net.output16_gpu);
                    assert(*state.net.max_output16_size > 0);
                    *state.net.output16_gpu = cuda_make_array(NULL, l.nweights);
                }
                float clip = 0.0;
                float divider = 1.0;
                float abs_add = 1.0;
                mult_inverse_array_gpu(l.weights_gpu, *state.net.output16_gpu, l.nweights, l.reverse, divider, clip, abs_add);
                l.weights_gpu = *state.net.output16_gpu;
            }
            */

            // http://docs.nvidia.com/deeplearning/sdk/cudnn-developer-guide/index.html#hipdnnConvolutionBackwardData
            // calculate delta for the next layer
            CHECK_CUDNN(hipdnnConvolutionBackwardData(cudnn_handle(),
                &one,
                l.weightDesc,
                l.weights_gpu,
                l.ddstTensorDesc,
                l.delta_gpu,
                l.convDesc,
                l.bd_algo,
                state.workspace,
                l.workspace_size,
                &one,
                l.dsrcTensorDesc,
                state.delta));

            l.weights_gpu = old_weights;

            if (l.binary || l.xnor) swap_binary(&l);
            if (l.xnor) gradient_array_ongpu(original_input, l.batch*l.c*l.h*l.w, HARDTAN, state.delta);
        }
    }

//#endif    // CUDNN_HALF

#else    // CUDNN
    if (l.batch_normalize) {
        backward_batchnorm_layer_gpu(l, state);
    }

    int m = l.n / l.groups;
    int n = l.size*l.size*l.c / l.groups;
    int k = l.out_w*l.out_h;

    int i, j;
    for(i = 0; i < l.batch; ++i){
        for (j = 0; j < l.groups; ++j) {
            float * a = l.delta_gpu + (i*l.groups + j)*m*k;
            float * b = state.workspace;
            float * c = l.weight_updates_gpu + j*l.nweights / l.groups;

            float *im = state.input + (i*l.groups + j)*l.c / l.groups*l.h*l.w;

            if (!state.net.adversarial && !l.train_only_bn) {
                //im2col_ongpu(im, l.c / l.groups, l.h, l.w, l.size, l.stride, l.pad, state.workspace);
                im2col_gpu_ext(im,          // input
                    l.c / l.groups,         // input channels
                    l.h, l.w,               // input size (h, w)
                    l.size, l.size,         // kernel size (h, w)
                    l.pad * l.dilation, l.pad * l.dilation,   // padding (h, w)
                    l.stride_y, l.stride_x,     // stride (h, w)
                    l.dilation, l.dilation, // dilation (h, w)
                    state.workspace);       // output
                //gemm_ongpu(0, 1, m, n, k, 1, a + i*m*k, k, b, k, 1, c, n);
                gemm_ongpu(0, 1, m, n, k, 1, a, k, b, k, 1, c, n);
            }

            if (state.delta) {
                if (l.binary || l.xnor) swap_binary(&l);
                float * a = l.weights_gpu + j*l.nweights / l.groups;
                float * b = l.delta_gpu + (i*l.groups + j)*m*k;
                float * c = state.workspace;

                //gemm_ongpu(1, 0, n, k, m, 1, a, n, b + i*k*m, k, 0, c, k);
                gemm_ongpu(1, 0, n, k, m, 1, a, n, b, k, 0, c, k);


                float *delta = state.delta + (i*l.groups + j)*l.c / l.groups*l.h*l.w;

                //col2im_ongpu(state.workspace, l.c / l.groups, l.h, l.w, l.size, l.stride, l.pad, delta);
                col2im_gpu_ext(
                    state.workspace,        // input
                    l.c / l.groups,         // input channels
                    l.h, l.w,               // input size (h, w)
                    l.size, l.size,         // kernel size (h, w)
                    l.pad * l.dilation, l.pad * l.dilation,   // padding size (h, w)
                    l.stride_y, l.stride_x,     // stride size (h, w)
                    l.dilation, l.dilation, // dilation size (h, w)
                    delta);                 // output (delta)

                if (l.binary || l.xnor) {
                    swap_binary(&l);
                }
                if (l.xnor) gradient_array_ongpu(original_input + i*l.c*l.h*l.w, l.c*l.h*l.w, HARDTAN, state.delta + i*l.c*l.h*l.w);
            }
        }
    }
#endif
    if (state.net.try_fix_nan) {
        if (state.delta) {
            reset_nan_and_inf(state.delta, l.inputs * l.batch);
        }
        int size = l.nweights;
        reset_nan_and_inf(l.weight_updates_gpu, size);
        fix_nan_and_inf(l.weights_gpu, size);
    }


}

__global__ void calc_avg_activation_kernel(float *src, float *dst, int size, int channels, int batches)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int xy = i % size;
    int b = i / size;

    if (i < size*batches) {
        dst[i] = 0;
        for (int c = 0; c < channels; ++c) {
            dst[i] += src[xy + size*(c + channels*b)];
        }
        dst[i] = dst[i] / channels;
    }
}

void calc_avg_activation_gpu(float *src, float *dst, int size, int channels, int batches)
{
    const int num_blocks = get_number_of_blocks(size*batches, BLOCK);

    calc_avg_activation_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> > (src, dst, size, channels, batches);
}


__global__ void assisted_activation_kernel(float alpha, float *output, float *gt_gpu, float *a_avg_gpu, int size, int channels, int batches)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int xy = i % size;
    int b = i / size;

    if (b < batches) {
        for (int c = 0; c < channels; ++c) {
            output[xy + size*(c + channels*b)] += alpha * gt_gpu[i] * a_avg_gpu[i];
            //output[xy + size*(c + channels*b)] += gt_gpu[i] * a_avg_gpu[i];
            //output[xy + size*(c + channels*b)] += gt_gpu[i] * output[xy + size*(c + channels*b)];
            //output[xy + size*(c + channels*b)] = a_avg_gpu[i];
        }
    }
}

void assisted_activation_gpu(float alpha, float *output, float *gt_gpu, float *a_avg_gpu, int size, int channels, int batches)
{
    const int num_blocks = get_number_of_blocks(size*batches, BLOCK);

    assisted_activation_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> > (alpha, output, gt_gpu, a_avg_gpu, size, channels, batches);
}


__global__ void assisted_activation2_kernel(float alpha, float *output, float *gt_gpu, float *a_avg_gpu, int size, int channels, int batches)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int xy = i % size;
    int b = i / size;
    float beta = 1 - alpha;

    if (b < batches) {
        for (int c = 0; c < channels; ++c) {
            if(gt_gpu[i] == 0)
                output[xy + size*(c + channels*b)] *= beta;

        }
    }
}

void assisted_activation2_gpu(float alpha, float *output, float *gt_gpu, float *a_avg_gpu, int size, int channels, int batches)
{
    const int num_blocks = get_number_of_blocks(size*batches, BLOCK);

    assisted_activation2_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> > (alpha, output, gt_gpu, a_avg_gpu, size, channels, batches);
}

void assisted_excitation_forward_gpu(convolutional_layer l, network_state state)
{
    const int iteration_num = get_current_iteration(state.net); //(*state.net.seen) / (state.net.batch*state.net.subdivisions);

    // epoch
    //const float epoch = (float)(*state.net.seen) / state.net.train_images_num;

    // calculate alpha
    //const float alpha = (1 + cos(3.141592 * iteration_num)) / (2 * state.net.max_batches);
    //const float alpha = (1 + cos(3.141592 * epoch)) / (2 * state.net.max_batches);
    float alpha = (1 + cos(3.141592 * iteration_num / state.net.max_batches)) / 2;
    //float alpha = (1 + cos(3.141592 * iteration_num / state.net.max_batches));

    if (l.assisted_excitation == 1) {
        if (iteration_num > state.net.max_batches / 2) return;
    }
    else {
        if (iteration_num < state.net.burn_in) return;
        else
            if (iteration_num > l.assisted_excitation) return;
        else
            alpha = (1 + cos(3.141592 * iteration_num / (state.net.burn_in + l.assisted_excitation))) / 2; // from 1 to 0
    }

    //printf("\n epoch = %f, alpha = %f, seen = %d, max_batches = %d, train_images_num = %d \n",
    //    epoch, alpha, (*state.net.seen), state.net.max_batches, state.net.train_images_num);

    //const int size = l.outputs * l.batch;

    float *a_avg = (float *)calloc(l.out_w * l.out_h * l.batch, sizeof(float));
    float *gt = (float *)calloc(l.out_w * l.out_h * l.batch, sizeof(float));

    int b;
    int w, h;

    l.max_boxes = state.net.num_boxes;
    l.truths = l.max_boxes*(4 + 1);

    int num_truth = l.batch*l.truths;
    float *truth_cpu = (float *)calloc(num_truth, sizeof(float));
    cuda_pull_array(state.truth, truth_cpu, num_truth);
    //hipStreamSynchronize(get_cuda_stream());
    //CHECK_CUDA(hipPeekAtLastError());

    for (b = 0; b < l.batch; ++b)
    {
        // calculate G
        int t;
        for (t = 0; t < state.net.num_boxes; ++t) {
            box truth = float_to_box_stride(truth_cpu + t*(4 + 1) + b*l.truths, 1);
            if (!truth.x) break;  // continue;
            float beta = 0;
            //float beta = 1 - alpha; // from 0 to 1
            float dw = (1 - truth.w) * beta;
            float dh = (1 - truth.h) * beta;
            //printf(" alpha = %f, beta = %f, truth.w = %f, dw = %f, tw+dw = %f, l.out_w = %d \n", alpha, beta, truth.w, dw, truth.w+dw, l.out_w);

            int left = floorf((truth.x - (dw + truth.w) / 2) * l.out_w);
            int right = ceilf((truth.x + (dw + truth.w) / 2) * l.out_w);
            int top = floorf((truth.y - (dh + truth.h) / 2) * l.out_h);
            int bottom = ceilf((truth.y + (dh + truth.h) / 2) * l.out_h);
            if (left < 0) left = 0;
            if (top < 0) top = 0;
            if (right > l.out_w) right = l.out_w;
            if (bottom > l.out_h) bottom = l.out_h;

            for (w = left; w <= right; w++) {
                for (h = top; h < bottom; h++) {
                    gt[w + l.out_w * h + l.out_w*l.out_h*b] = 1;
                }
            }
        }
    }

    cuda_push_array(l.gt_gpu, gt, l.out_w * l.out_h * l.batch);
    //hipStreamSynchronize(get_cuda_stream());
    //CHECK_CUDA(hipPeekAtLastError());

    // calc avg_output on GPU - for whole batch
    calc_avg_activation_gpu(l.output_gpu, l.a_avg_gpu, l.out_w * l.out_h, l.out_c, l.batch);
    //hipStreamSynchronize(get_cuda_stream());
    //CHECK_CUDA(hipPeekAtLastError());

    // calc new output
    //assisted_activation2_gpu(1, l.output_gpu, l.gt_gpu, l.a_avg_gpu, l.out_w * l.out_h, l.out_c, l.batch);  // AE3: gt increases (beta = 1 - alpha = 0)
    //assisted_activation2_gpu(alpha, l.output_gpu, l.gt_gpu, l.a_avg_gpu, l.out_w * l.out_h, l.out_c, l.batch);
    assisted_activation_gpu(alpha, l.output_gpu, l.gt_gpu, l.a_avg_gpu, l.out_w * l.out_h, l.out_c, l.batch);
    //hipStreamSynchronize(get_cuda_stream());
    //CHECK_CUDA(hipPeekAtLastError());



    /*
    for (b = 0; b < l.batch; ++b)
    {
        // calculate average A
        for (w = 0; w < l.out_w; w++) {
            for (h = 0; h < l.out_h; h++) {
                for (c = 0; c < l.out_c; c++) {
                    a_avg[w + l.out_w*(h + l.out_h*b)] += l.output[w + l.out_w*(h + l.out_h*(c + l.out_c*b))];
                }
                a_avg[w + l.out_w*(h + l.out_h*b)] /= l.out_c;  // a_avg / d
            }
        }
    }

    // change activation
    for (b = 0; b < l.batch; ++b)
    {
        for (w = 0; w < l.out_w; w++) {
            for (h = 0; h < l.out_h; h++) {
                for (c = 0; c < l.out_c; c++)
                {
                    // a = a + alpha(t) + e(c,i,j) = a + alpha(t) + g(i,j) * avg_a(i,j) / channels
                    l.output[w + l.out_w*(h + l.out_h*(c + l.out_c*b))] +=
                        alpha *
                        g[w + l.out_w*(h + l.out_h*b)] *
                        a_avg[w + l.out_w*(h + l.out_h*b)];

                    //l.output[w + l.out_w*(h + l.out_h*(c + l.out_c*b))] =
                    //    alpha * g[w + l.out_w*(h + l.out_h*b)] * a_avg[w + l.out_w*(h + l.out_h*b)];
                }
            }
        }
    }
    */

    if (0)   // visualize ground truth
    {
#ifdef OPENCV
        cuda_pull_array(l.output_gpu, l.output, l.outputs * l.batch);
        hipStreamSynchronize(get_cuda_stream());
        CHECK_CUDA(hipPeekAtLastError());

        for (b = 0; b < l.batch; ++b)
        {
            printf(" Assisted Excitation alpha = %f \n", alpha);
            image img = float_to_image(l.out_w, l.out_h, 1, &gt[l.out_w*l.out_h*b]);
            char buff[100];
            sprintf(buff, "a_excitation_gt_%d", b);
            show_image_cv(img, buff);

            //image img2 = float_to_image(l.out_w, l.out_h, 1, &l.output[l.out_w*l.out_h*l.out_c*b]);
            image img2 = float_to_image_scaled(l.out_w, l.out_h, 1, &l.output[l.out_w*l.out_h*l.out_c*b]);
            char buff2[100];
            sprintf(buff2, "a_excitation_output_%d", b);
            show_image_cv(img2, buff2);

            /*
            int c = l.out_c;
            if (c > 4) c = 4;
            image img3 = float_to_image(l.out_w, l.out_h, c, &l.output[l.out_w*l.out_h*l.out_c*b]);
            image dc = collapse_image_layers(img3, 1);
            char buff3[100];
            sprintf(buff3, "a_excitation_act_collapsed_%d", b);
            show_image_cv(dc, buff3);
            */

            wait_key_cv(5);
        }
        wait_until_press_key_cv();
#endif // OPENCV
    }

    free(truth_cpu);
    free(gt);
    free(a_avg);
}

void pull_convolutional_layer(convolutional_layer l)
{
    cuda_pull_array_async(l.weights_gpu, l.weights, l.nweights);
    cuda_pull_array_async(l.biases_gpu, l.biases, l.n);
    if (l.weight_updates_gpu) cuda_pull_array_async(l.weight_updates_gpu, l.weight_updates, l.nweights);
    if (l.bias_updates_gpu) cuda_pull_array_async(l.bias_updates_gpu, l.bias_updates, l.n);
    if (l.batch_normalize){
        cuda_pull_array_async(l.scales_gpu, l.scales, l.n);
        cuda_pull_array_async(l.rolling_mean_gpu, l.rolling_mean, l.n);
        cuda_pull_array_async(l.rolling_variance_gpu, l.rolling_variance, l.n);
    }
    if (l.adam){
        cuda_pull_array_async(l.m_gpu, l.m, l.nweights);
        cuda_pull_array_async(l.v_gpu, l.v, l.nweights);
    }
    CHECK_CUDA(hipPeekAtLastError());
    hipStreamSynchronize(get_cuda_stream());
}

void push_convolutional_layer(convolutional_layer l)
{
    cuda_push_array(l.weights_gpu, l.weights, l.nweights);
#ifdef CUDNN_HALF
    assert(l.nweights > 0);
    cuda_convert_f32_to_f16(l.weights_gpu, l.nweights, l.weights_gpu16);
#endif
    cuda_push_array(l.biases_gpu, l.biases, l.n);
    if (l.train) {
        cuda_push_array(l.weight_updates_gpu, l.weight_updates, l.nweights);
        cuda_push_array(l.bias_updates_gpu, l.bias_updates, l.n);
    }
    if (l.batch_normalize){
        cuda_push_array(l.scales_gpu, l.scales, l.n);
        cuda_push_array(l.rolling_mean_gpu, l.rolling_mean, l.n);
        cuda_push_array(l.rolling_variance_gpu, l.rolling_variance, l.n);
    }
    if (l.adam){
        cuda_push_array(l.m_gpu, l.m, l.nweights);
        cuda_push_array(l.v_gpu, l.v, l.nweights);
    }
    CHECK_CUDA(hipPeekAtLastError());
}

void update_convolutional_layer_gpu(layer l, int batch, float learning_rate_init, float momentum, float decay, float loss_scale)
{

        /*
        for (int angle = 0; angle < 360; angle++) {
            printf(" angle = %d \n", angle);
            smooth_rotate_weights_kernel(l.weights_gpu, l.weight_deform_gpu, l.nweights, l.n, l.size, angle, 0);

            cuda_pull_array(l.weight_deform_gpu, l.weights, l.nweights);
            visualize_convolutional_layer(l, "weights", NULL);
            wait_key_cv(10);
        }
        */

    if (l.deform) {

        //for (l.angle = 0; l.angle < 360; l.angle += 1)
        //{
            //stretch_weights_gpu(l.weight_updates_gpu, l.weight_deform_gpu, l.nweights, l.n, l.size, l.angle/180, 1);
            //else simple_copy_ongpu(l.nweights, l.weight_updates_gpu, l.weight_deform_gpu);

            if (l.rotate) rotate_weights_gpu(l.weight_updates_gpu, l.weight_deform_gpu, l.nweights, l.n, l.size, 1);
            else if (l.sway) sway_and_flip_weights_gpu(l.weight_updates_gpu, l.weight_deform_gpu, l.nweights, l.n, l.size, l.angle, 1);
            else if (l.stretch) stretch_weights_gpu(l.weight_updates_gpu, l.weight_deform_gpu, l.nweights, l.n, l.size, 0, 1);
            else if (l.stretch_sway) stretch_sway_flip_weights_gpu(l.weight_updates_gpu, l.weight_deform_gpu, l.nweights, l.n, l.size, l.angle, 1);

            //simple_copy_ongpu(l.nweights, l.weight_updates_gpu, l.weight_deform_gpu);

            reduce_and_expand_array_gpu(l.weight_deform_gpu, l.weight_updates_gpu, l.nweights, 4);

            //printf(" angle = %f \n", l.angle);
            //cuda_pull_array(l.weight_deform_gpu, l.weights, l.nweights);
            //visualize_convolutional_layer(l, "weights", NULL);
            //wait_key_cv(10);
        //}

    }

    // Loss scale for Mixed-Precision on Tensor-Cores
    float learning_rate = learning_rate_init*l.learning_rate_scale / loss_scale;
    //float momentum = a.momentum;
    //float decay = a.decay;
    //int batch = a.batch;


    reset_nan_and_inf(l.weight_updates_gpu, l.nweights);
    fix_nan_and_inf(l.weights_gpu, l.nweights);

    // Gradient Centralization
    if (l.grad_centr && l.batch_normalize) {
        // weights[filters][channels][height][width]
        // for(filters) w[f] = w[f] - mean(w[c][h][w])
        gradient_centralization_gpu(l.size, l.size, l.c / l.groups, l.n, l.weight_updates_gpu);
    }


    if (l.adam) {
        //adam_update_gpu(l.weights_gpu, l.weight_updates_gpu, l.m_gpu, l.v_gpu, a.B1, a.B2, a.eps, decay, learning_rate, l.nweights, batch, a.t);
        adam_update_gpu(l.weights_gpu, l.weight_updates_gpu, l.m_gpu, l.v_gpu, l.B1, l.B2, l.eps, decay, learning_rate, l.nweights, batch, l.t);

        adam_update_gpu(l.biases_gpu, l.bias_updates_gpu, l.bias_m_gpu, l.bias_v_gpu, l.B1, l.B2, l.eps, decay, learning_rate, l.n, batch, l.t);
        if (l.scales_gpu) {
            adam_update_gpu(l.scales_gpu, l.scale_updates_gpu, l.scale_m_gpu, l.scale_v_gpu, l.B1, l.B2, l.eps, decay, learning_rate, l.n, batch, l.t);
        }
    }
    else {
        //axpy_ongpu(l.nweights, -decay*batch, l.weights_gpu, 1, l.weight_updates_gpu, 1);
        //axpy_ongpu(l.nweights, learning_rate / batch, l.weight_updates_gpu, 1, l.weights_gpu, 1);
        //scal_ongpu(l.nweights, momentum, l.weight_updates_gpu, 1);

        float *old_weight_updates_gpu = l.weight_updates_gpu;


        if (l.reverse) {
            float clip = 0.0;
            float divider = 1.0;
            float abs_add = 1.0;
            mult_inverse_array_gpu(l.weight_updates_gpu, l.output_gpu, l.inputs*l.batch, l.reverse, divider, clip, abs_add);
            l.weight_updates_gpu = l.output_gpu;
        }


        axpy_ongpu(l.nweights, -decay*batch*loss_scale, l.weights_gpu, 1, l.weight_updates_gpu, 1);
        axpy_ongpu(l.nweights, learning_rate / batch, l.weight_updates_gpu, 1, l.weights_gpu, 1);

        l.weight_updates_gpu = old_weight_updates_gpu;

        scal_ongpu(l.nweights, momentum, l.weight_updates_gpu, 1);

        axpy_ongpu(l.n, learning_rate / batch, l.bias_updates_gpu, 1, l.biases_gpu, 1);
        scal_ongpu(l.n, momentum, l.bias_updates_gpu, 1);

        if (l.scales_gpu) {
            axpy_ongpu(l.n, learning_rate / batch, l.scale_updates_gpu, 1, l.scales_gpu, 1);
            scal_ongpu(l.n, momentum, l.scale_updates_gpu, 1);
        }
    }

    if (l.deform) {
        //for (l.angle = 0; l.angle < 360; l.angle += 4)
        //{
            expand_array_gpu(l.weights_gpu, l.weight_deform_gpu, l.nweights, 4);

            //simple_copy_ongpu(l.nweights, l.weight_deform_gpu, l.weights_gpu);

            if (l.rotate) rotate_weights_gpu(l.weight_deform_gpu, l.weights_gpu, l.nweights, l.n, l.size, 0);
            else if (l.sway) sway_and_flip_weights_gpu(l.weight_deform_gpu, l.weights_gpu, l.nweights, l.n, l.size, l.angle, 0);
            else if (l.stretch) stretch_weights_gpu(l.weight_deform_gpu, l.weights_gpu, l.nweights, l.n, l.size, 0, 0);
            else if (l.stretch_sway) stretch_sway_flip_weights_gpu(l.weight_deform_gpu, l.weights_gpu, l.nweights, l.n, l.size, l.angle, 0);

            //printf(" angle = %f, reverse = %d \n", l.angle, 0);
            //cuda_pull_array(l.weights_gpu, l.weights, l.nweights);
            //visualize_convolutional_layer(l, "weights", NULL);
            //wait_key_cv(10);
        //}
    }

    if (l.clip) {
        constrain_ongpu(l.nweights, l.clip, l.weights_gpu, 1);
    }
}



/*
void update_convolutional_layer_gpu(convolutional_layer layer, int batch, float learning_rate, float momentum, float decay)
{
    int size = layer.size*layer.size*layer.c*layer.n;
    axpy_ongpu(layer.n, learning_rate/batch, layer.bias_updates_gpu, 1, layer.biases_gpu, 1);
    scal_ongpu(layer.n, momentum, layer.bias_updates_gpu, 1);

    if(layer.scales_gpu){
        axpy_ongpu(layer.n, learning_rate/batch, layer.scale_updates_gpu, 1, layer.scales_gpu, 1);
        scal_ongpu(layer.n, momentum, layer.scale_updates_gpu, 1);
    }

    if(layer.adam){
        scal_ongpu(size, layer.B1, layer.m_gpu, 1);
        scal_ongpu(size, layer.B2, layer.v_gpu, 1);

        axpy_ongpu(size, -decay*batch, layer.weights_gpu, 1, layer.weight_updates_gpu, 1);

        axpy_ongpu(size, -(1-layer.B1), layer.weight_updates_gpu, 1, layer.m_gpu, 1);
        mul_ongpu(size, layer.weight_updates_gpu, 1, layer.weight_updates_gpu, 1);
        axpy_ongpu(size, (1-layer.B2), layer.weight_updates_gpu, 1, layer.v_gpu, 1);

        adam_gpu(size, layer.weights_gpu, layer.m_gpu, layer.v_gpu, layer.B1, layer.B2, learning_rate/batch, layer.eps, layer.t+1);
        fill_ongpu(size, 0, layer.weight_updates_gpu, 1);
    }else{
        axpy_ongpu(size, -decay*batch, layer.weights_gpu, 1, layer.weight_updates_gpu, 1);  // wu = wu - w*decay*batch
        axpy_ongpu(size, learning_rate/batch, layer.weight_updates_gpu, 1, layer.weights_gpu, 1); // w = w + wu*lr/batch
        scal_ongpu(size, momentum, layer.weight_updates_gpu, 1);    // wu = wu*momentum // wu = (wu - w*decay*batch)*momentum
        // w = w + (wu - w*decay*batch)*lr/batch = w + wu*lr/batch - w*decay*lr = w*(1-decay*lr) + wu*lr/batch
        //wu_prev = (wu_old - w_old*decay*batch)*momentum


        //weights_update = weights_update_new + (weights_update_old - weights_old*decay*batch)*momentum - weights_new*decay*batch =
        // = weights_update_new + weights_update_old*momentum - weights_old*decay*batch*momentum - weights_new*decay*batch
        // = weights_update_new + weights_update_old*momentum - (weights_old*momentum + weights_new)*decay*batch

        //------------- RESULT --------------
        // weights_update = weights_update_new + weights_update_old*momentum - (weights_old*momentum + weights_new)*decay*batch
        //-----------------------------------

        // weights_newest = weights_new + (weights_update_new + weights_update_old*momentum - (weights_old*momentum + weights_new)*decay*batch)*lr/batch
        // = weights_new + weights_update_new*lr/batch + weights_update_old*momentum*lr/batch - weights_old*momentum*decay*batch*lr/batch - weights_new*decay*batch*lr/batch
        // = weights_new + weights_update_new*lr/batch + weights_update_old*momentum*lr/batch - weights_old*momentum*decay*lr - weights_new*decay*lr
        // = weights_new*(1 - decay*lr) - weights_old*momentum*decay*lr + (weights_update_new + weights_update_old*momentum)*lr/batch

        //------------- RESULT --------------
        // weights_newest = weights_new*(1 - decay*lr) - weights_old*momentum*(decay*lr) + (weights_update_new + weights_update_old*momentum)*lr/batch =
        // = weights_new - (weights_new + weights_old*momentum)*decay*lr + (weights_update_new + weights_update_old*momentum)*lr / batch
        //-----------------------------------
    }
}
*/
