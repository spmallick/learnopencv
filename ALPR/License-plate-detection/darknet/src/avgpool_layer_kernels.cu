#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipblas.h>

#include "avgpool_layer.h"
#include "dark_cuda.h"

__global__ void forward_avgpool_layer_kernel(int n, int w, int h, int c, float *input, float *output)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int k = id % c;
    id /= c;
    int b = id;

    int i;
    int out_index = (k + c*b);
    output[out_index] = 0;
    for(i = 0; i < w*h; ++i){
        int in_index = i + h*w*(k + b*c);
        output[out_index] += input[in_index];
    }
    output[out_index] /= w*h;
}

__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c, float *in_delta, float *out_delta)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int k = id % c;
    id /= c;
    int b = id;

    int i;
    int out_index = (k + c*b);
    for(i = 0; i < w*h; ++i){
        int in_index = i + h*w*(k + b*c);
        in_delta[in_index] += out_delta[out_index] / (w*h);
    }
}

extern "C" void forward_avgpool_layer_gpu(avgpool_layer layer, network_state state)
{
    size_t n = layer.c*layer.batch;

    forward_avgpool_layer_kernel<<<cuda_gridsize(n), BLOCK, 0, get_cuda_stream() >>>(n, layer.w, layer.h, layer.c, state.input, layer.output_gpu);
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void backward_avgpool_layer_gpu(avgpool_layer layer, network_state state)
{
    size_t n = layer.c*layer.batch;

    backward_avgpool_layer_kernel<<<cuda_gridsize(n), BLOCK, 0, get_cuda_stream() >>>(n, layer.w, layer.h, layer.c, state.delta, layer.delta_gpu);
    CHECK_CUDA(hipPeekAtLastError());
}
