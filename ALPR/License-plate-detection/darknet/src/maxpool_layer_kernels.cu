#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipblas.h>

#include "maxpool_layer.h"
#include "convolutional_layer.h"
#include "blas.h"
#include "dark_cuda.h"

__global__ void forward_maxpool_depth_layer_kernel(int n, int w, int h, int c, int out_c, int batch, float *input, float *output, int *indexes)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= n) return;

    int j = id % w;
    id = id / w;
    int i = id % h;
    id = id / h;
    //int g = id % out_c;
    //id = id / out_c;
    int b = id % batch;

    int k;
    for (int g = 0; g < out_c; ++g)
    {
        int out_index = j + w*(i + h*(g + out_c*b));
        float max = -FLT_MAX;
        int max_i = -1;

        for (k = g; k < c; k += out_c)
        {
            int in_index = j + w*(i + h*(k + c*b));
            float val = input[in_index];

            max_i = (val > max) ? in_index : max_i;
            max = (val > max) ? val : max;
        }
        output[out_index] = max;
        if (indexes) indexes[out_index] = max_i;
    }
}


__global__ void backward_maxpool_depth_layer_kernel(int n, int w, int h, int c, int batch, float *delta, float *prev_delta, int *indexes)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= n) return;

    int index = indexes[id];
    prev_delta[index] += delta[id];
}


__global__ void forward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride_x, int stride_y, int size, int pad, float *input, float *output, int *indexes)
{
    int h = (in_h + pad - size) / stride_y + 1;
    int w = (in_w + pad - size) / stride_x + 1;
    int c = in_c;

    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int j = id % w;
    id /= w;
    int i = id % h;
    id /= h;
    int k = id % c;
    id /= c;
    int b = id;

    int w_offset = -pad / 2;
    int h_offset = -pad / 2;

    int out_index = j + w*(i + h*(k + c*b));
    float max = -INFINITY;
    int max_i = -1;
    int l, m;
    for(l = 0; l < size; ++l){
        for(m = 0; m < size; ++m){
            int cur_h = h_offset + i*stride_y + l;
            int cur_w = w_offset + j*stride_x + m;
            int index = cur_w + in_w*(cur_h + in_h*(k + b*in_c));
            int valid = (cur_h >= 0 && cur_h < in_h &&
                    cur_w >= 0 && cur_w < in_w);
            float val = (valid != 0) ? input[index] : -INFINITY;
            max_i = (val > max) ? index : max_i;
            max   = (val > max) ? val   : max;
        }
    }
    output[out_index] = max;
    if (indexes) indexes[out_index] = max_i;
}

__global__ void forward_zero_nonmax_kernel(int n, float *input, float *output)
{

    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= n) return;

    if (input[id] != output[id]) output[id] = 0;
}

__global__ void backward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride_x, int stride_y, int size, int pad, float *delta, float *prev_delta, int *indexes)
{
    int h = (in_h + pad - size) / stride_y + 1;
    int w = (in_w + pad - size) / stride_x + 1;
    int c = in_c;
    int area_x = (size - 1) / stride_x;
    int area_y = (size - 1) / stride_y;

    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int index = id;
    int j = id % in_w;
    id /= in_w;
    int i = id % in_h;
    id /= in_h;
    int k = id % in_c;
    id /= in_c;
    int b = id;

    int w_offset = -pad / 2;
    int h_offset = -pad / 2;

    float d = 0;
    int l, m;
    for(l = -area_y; l < area_y+1; ++l){
        for(m = -area_x; m < area_x+1; ++m){
            int out_w = (j-w_offset)/stride_x + m;
            int out_h = (i-h_offset)/stride_y + l;
            int out_index = out_w + w*(out_h + h*(k + c*b));
            int valid = (out_w >= 0 && out_w < w &&
                     out_h >= 0 && out_h < h);
            d += (valid && indexes[out_index] == index) ? delta[out_index] : 0;
        }
    }
    prev_delta[index] += d;
}

__global__ void backward_zero_nonmax_kernel(int n, int *indexes, float *prev_delta)
{

    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= n) return;

    if (indexes[id] != id) prev_delta[id] = 0;
}
extern "C" void forward_maxpool_layer_gpu(maxpool_layer layer, network_state state)
{
    if (layer.maxpool_depth) {
        int h = layer.out_h;
        int w = layer.out_w;
        int c = 1;// layer.out_c;

        size_t n = h*w*c*layer.batch;

        forward_maxpool_depth_layer_kernel << <cuda_gridsize(n), BLOCK, 0, get_cuda_stream() >> >(
            n, layer.w, layer.h, layer.c, layer.out_c, layer.batch, state.input, layer.output_gpu, layer.indexes_gpu);
        CHECK_CUDA(hipPeekAtLastError());

        return;
    }

#ifdef CUDNN_DISABLED
    if (!state.train && layer.stride == layer.size) {
        // hipdnnPoolingBackward
        hipdnnStatus_t maxpool_status;

        float alpha = 1, beta = 0;
        maxpool_status = hipdnnPoolingForward(
            cudnn_handle(),
            layer.poolingDesc,
            &alpha,
            layer.srcTensorDesc,
            state.input,
            &beta,
            layer.dstTensorDesc,
            layer.output_gpu);

        //maxpool_status = hipdnnDestroyPoolingDescriptor(poolingDesc);
        //hipdnnDestroyTensorDescriptor(layer.srcTensorDesc);
        //hipdnnDestroyTensorDescriptor(layer.dstTensorDesc);

    }
    else
#endif
    {
        int h = layer.out_h;
        int w = layer.out_w;
        int c = layer.out_c;

        size_t n = h*w*c*layer.batch;

        forward_maxpool_layer_kernel << <cuda_gridsize(n), BLOCK, 0, get_cuda_stream() >> > (n, layer.h, layer.w, layer.c, layer.stride_x, layer.stride_y, layer.size, layer.pad, state.input, layer.output_gpu, layer.indexes_gpu);
        CHECK_CUDA(hipPeekAtLastError());

        if (layer.maxpool_zero_nonmax) {
            forward_zero_nonmax_kernel << <cuda_gridsize(n), BLOCK, 0, get_cuda_stream() >> > (n, state.input, layer.output_gpu);
            CHECK_CUDA(hipPeekAtLastError());
        }
    }

    if (layer.antialiasing) {
        network_state s = { 0 };
        s.train = state.train;
        s.workspace = state.workspace;
        s.net = state.net;
        if (!state.train) s.index = state.index;  // don't use TC for training (especially without cuda_convert_f32_to_f16() )
        s.input = layer.output_gpu;
        forward_convolutional_layer_gpu(*(layer.input_layer), s);
        simple_copy_ongpu(layer.outputs*layer.batch, layer.output_gpu, layer.input_antialiasing_gpu);
        simple_copy_ongpu(layer.input_layer->outputs*layer.input_layer->batch, layer.input_layer->output_gpu, layer.output_gpu);
    }
}

extern "C" void backward_maxpool_layer_gpu(maxpool_layer layer, network_state state)
{
    if (layer.antialiasing) {
        network_state s = { 0 };
        s.train = state.train;
        s.workspace = state.workspace;
        s.net = state.net;
        s.delta = layer.delta_gpu;  // s.delta will be returned to l.delta_gpu
        s.input = layer.input_antialiasing_gpu;
        //if (!state.train) s.index = state.index;  // don't use TC for training (especially without cuda_convert_f32_to_f16() )
        simple_copy_ongpu(layer.input_layer->outputs*layer.input_layer->batch, layer.delta_gpu, layer.input_layer->delta_gpu);
        backward_convolutional_layer_gpu(*(layer.input_layer), s);

        //simple_copy_ongpu(layer.outputs*layer.batch, layer.input_antialiasing_gpu, layer.output_gpu);
    }

    if (layer.maxpool_depth) {
        int h = layer.out_h;
        int w = layer.out_w;
        int c = layer.out_c;

        size_t n = h * w * c * layer.batch;

        backward_maxpool_depth_layer_kernel << <cuda_gridsize(n), BLOCK, 0, get_cuda_stream() >> >(n, layer.w, layer.h, layer.c, layer.batch, layer.delta_gpu, state.delta, layer.indexes_gpu);
        CHECK_CUDA(hipPeekAtLastError());
        return;
    }

    size_t n = layer.h*layer.w*layer.c*layer.batch;

    backward_maxpool_layer_kernel<<<cuda_gridsize(n), BLOCK, 0, get_cuda_stream() >>>(n, layer.h, layer.w, layer.c, layer.stride_x, layer.stride_y, layer.size, layer.pad, layer.delta_gpu, state.delta, layer.indexes_gpu);
    CHECK_CUDA(hipPeekAtLastError());

    if (layer.maxpool_zero_nonmax) {
        backward_zero_nonmax_kernel << <cuda_gridsize(n), BLOCK, 0, get_cuda_stream() >> > (n, layer.indexes_gpu, state.delta);
        CHECK_CUDA(hipPeekAtLastError());
    }
}




__global__ void forward_local_avgpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride_x, int stride_y, int size, int pad, float *input, float *output)
{
    int h = (in_h + pad - size) / stride_y + 1;
    int w = (in_w + pad - size) / stride_x + 1;
    int c = in_c;

    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= n) return;

    int j = id % w;
    id /= w;
    int i = id % h;
    id /= h;
    int k = id % c;
    id /= c;
    int b = id;

    int w_offset = -pad / 2;
    int h_offset = -pad / 2;

    int out_index = j + w*(i + h*(k + c*b));
    float avg = 0;
    int counter = 0;
    int l, m;
    for (l = 0; l < size; ++l) {
        for (m = 0; m < size; ++m) {
            int cur_h = h_offset + i*stride_y + l;
            int cur_w = w_offset + j*stride_x + m;
            int index = cur_w + in_w*(cur_h + in_h*(k + b*in_c));
            int valid = (cur_h >= 0 && cur_h < in_h &&
                cur_w >= 0 && cur_w < in_w);
            if (valid) {
                counter++;
                avg += input[index];
            }
        }
    }
    output[out_index] = avg / counter;  // as HIPDNN_POOLING_AVERAGE_COUNT_EXCLUDE_PADDING
}


__global__ void backward_local_avgpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride_x, int stride_y, int size, int pad, float *delta, float *prev_delta)
{
    int h = (in_h + pad - size) / stride_y + 1;
    int w = (in_w + pad - size) / stride_x + 1;
    int c = in_c;
    int area_x = (size - 1) / stride_x;
    int area_y = (size - 1) / stride_y;

    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= n) return;

    int index = id;
    int j = id % in_w;
    id /= in_w;
    int i = id % in_h;
    id /= in_h;
    int k = id % in_c;
    id /= in_c;
    int b = id;

    int w_offset = -pad / 2;
    int h_offset = -pad / 2;

    int counter = 0;
    float d = 0;
    int l, m;
    for (l = -area_y; l < area_y + 1; ++l) {
        for (m = -area_x; m < area_x + 1; ++m) {
            int out_w = (j - w_offset) / stride_x + m;
            int out_h = (i - h_offset) / stride_y + l;
            int out_index = out_w + w*(out_h + h*(k + c*b));
            int valid = (out_w >= 0 && out_w < w && out_h >= 0 && out_h < h);
            if (valid) {
                counter++;
                d += delta[out_index];
            }
        }
    }
    if(counter > 0) prev_delta[index] += d / counter;
}



extern "C" void forward_local_avgpool_layer_gpu(maxpool_layer layer, network_state state)
{

#ifdef CUDNN_DISABLED
    if (!state.train && layer.stride == layer.size) {
        // hipdnnPoolingBackward
        hipdnnStatus_t maxpool_status;

        float alpha = 1, beta = 0;
        maxpool_status = hipdnnPoolingForward(
            cudnn_handle(),
            layer.poolingDesc,
            &alpha,
            layer.srcTensorDesc,
            state.input,
            &beta,
            layer.dstTensorDesc,
            layer.output_gpu);

        //maxpool_status = hipdnnDestroyPoolingDescriptor(poolingDesc);
        //hipdnnDestroyTensorDescriptor(layer.srcTensorDesc);
        //hipdnnDestroyTensorDescriptor(layer.dstTensorDesc);

    }
    else
#endif
    {
        int h = layer.out_h;
        int w = layer.out_w;
        int c = layer.out_c;

        size_t n = h*w*c*layer.batch;

        forward_local_avgpool_layer_kernel << <cuda_gridsize(n), BLOCK, 0, get_cuda_stream() >> > (n, layer.h, layer.w, layer.c, layer.stride_x, layer.stride_y, layer.size, layer.pad, state.input, layer.output_gpu);
        CHECK_CUDA(hipPeekAtLastError());
    }
}

extern "C" void backward_local_avgpool_layer_gpu(maxpool_layer layer, network_state state)
{
    size_t n = layer.h*layer.w*layer.c*layer.batch;

    backward_local_avgpool_layer_kernel << <cuda_gridsize(n), BLOCK, 0, get_cuda_stream() >> >(n, layer.h, layer.w, layer.c, layer.stride_x, layer.stride_y, layer.size, layer.pad, layer.delta_gpu, state.delta);
    CHECK_CUDA(hipPeekAtLastError());
}
