#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipblas.h>
#include <assert.h>
#include <float.h>

#include "blas.h"
#include "dark_cuda.h"
#include "utils.h"
#include "tree.h"

__inline__ __device__
float warpAllReduceSum(float val) {
    for (int mask = WARP_SIZE / 2; mask > 0; mask /= 2)
#if CUDART_VERSION >= 9000
        val += __shfl_xor_sync(0xffffffff, val, mask);
#else
        val += __shfl_xor(val, mask);
#endif
    return val;
}

__global__ void compare_2_arrays_kernel(float *one, float *two, int size)
{
    const int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index >= size) return;

    const float diff = 100 * fabs(one[index] - two[index]) / fabs(one[index]);

    if (diff > 10) printf(" i: %d - one = %f, two = %f, diff = %f %% \n", index, one[index], two[index], diff);
}

void compare_2_arrays_gpu(float *one, float *two, int size)
{
    const int num_blocks = get_number_of_blocks(size, BLOCK);

    compare_2_arrays_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> >(one, two, size);
    CHECK_CUDA(hipPeekAtLastError());
    CHECK_CUDA(hipDeviceSynchronize());
}

__global__ void mean_array_kernel(float *src, int size, float alpha, float *avg)
{
    const int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i >= size) return;

    avg[i] = avg[i] * (1 - alpha) + src[i] * alpha;
    src[i] = avg[i];
}


void mean_array_gpu(float *src, int size, float alpha, float *avg)
{
    const int num_blocks = get_number_of_blocks(size, BLOCK);

    mean_array_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> >(src, size, alpha, avg);
    CHECK_CUDA(hipPeekAtLastError());
}


__global__ void scale_bias_kernel(float *output, float *scale, int batch, int filters, int spatial, int current_size)
{
    const int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index >= current_size) return;

    int f = (index / spatial) % filters;
    output[index] *= scale[f];
}

void scale_bias_gpu(float *output, float *scale, int batch, int filters, int spatial)
{
    const int current_size = batch * filters * spatial;
    const int num_blocks = get_number_of_blocks(current_size, BLOCK);

    scale_bias_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> >(output, scale, batch, filters, spatial, current_size);
    CHECK_CUDA(hipPeekAtLastError());
}


__global__ void backward_scale_kernel(float *x_norm, float *delta, int batch, int n, int size, float *scale_updates)
{
    __shared__ float part[BLOCK];
    int i,b;
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;
    for(b = 0; b < batch; ++b){
        for(i = 0; i < size; i += BLOCK){
            int index = p + i + size*(filter + n*b);
            sum += (p+i < size) ? delta[index]*x_norm[index] : 0;
        }
    }
    part[p] = sum;
    __syncthreads();
    if (p == 0) {
        for(i = 0; i < BLOCK; ++i) scale_updates[filter] += part[i];
    }
}

void backward_scale_gpu(float *x_norm, float *delta, int batch, int n, int size, float *scale_updates)
{
    backward_scale_kernel<<<n, BLOCK, 0, get_cuda_stream() >>>(x_norm, delta, batch, n, size, scale_updates);
    CHECK_CUDA(hipPeekAtLastError());
}

__global__ void add_bias_kernel(float *output, float *biases, int batch, int filters, int spatial, int current_size)
{
    const int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index >= current_size) return;

    int f = (index / spatial) % filters;
    output[index] += biases[f];
}

void add_bias_gpu(float *output, float *biases, int batch, int filters, int spatial)
{
    const int current_size = batch * filters * spatial;
    const int num_blocks = get_number_of_blocks(current_size, BLOCK);

    add_bias_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> >(output, biases, batch, filters, spatial, current_size);
    CHECK_CUDA(hipPeekAtLastError());
}

__global__ void backward_bias_kernel(float *bias_updates, float *delta, int batch, int n, int size)
{
    __shared__ float part[BLOCK];
    int i,b;
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;
    for(b = 0; b < batch; ++b){
        for(i = 0; i < size; i += BLOCK){
            int index = p + i + size*(filter + n*b);
            sum += (p+i < size) ? delta[index] : 0;
        }
    }
    part[p] = sum;
    __syncthreads();
    if (p == 0) {
        for(i = 0; i < BLOCK; ++i) bias_updates[filter] += part[i];
    }
}

/*
__global__ void dot_kernel(float *output, float scale, int batch, int n, int size, float *delta)
{
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    int f1 = index / n;
    int f2 = index % n;
    if (f2 <= f1) return;

    float sum = 0;
    float norm1 = 0;
    float norm2 = 0;
    int b, i;
    for(b = 0; b <  batch; ++b){
        for(i = 0; i < size; ++i){
            int i1 = b * size * n + f1 * size + i;
            int i2 = b * size * n + f2 * size + i;
            sum += output[i1] * output[i2];
            norm1 += output[i1] * output[i1];
            norm2 += output[i2] * output[i2];
        }
    }
    norm1 = sqrt(norm1);
    norm2 = sqrt(norm2);
    float norm = norm1 * norm2;
    sum = sum / norm;
    for(b = 0; b <  batch; ++b){
        for(i = 0; i < size; ++i){
            int i1 = b * size * n + f1 * size + i;
            int i2 = b * size * n + f2 * size + i;
            delta[i1] += - scale * sum * output[i2] / norm;
            delta[i2] += - scale * sum * output[i1] / norm;
        }
    }
}

void dot_error_gpu(layer l)
{
    dot_kernel<<<cuda_gridsize(l.n*l.n), BLOCK, 0, get_cuda_stream()>>>(l.output_gpu, l.dot, l.batch, l.n, l.out_w * l.out_h, l.delta_gpu);
    CHECK_CUDA(hipPeekAtLastError());
}
*/

void backward_bias_gpu(float *bias_updates, float *delta, int batch, int n, int size)
{
    backward_bias_kernel<<<n, BLOCK, 0, get_cuda_stream() >>>(bias_updates, delta, batch, n, size);
    CHECK_CUDA(hipPeekAtLastError());
}

__global__ void adam_kernel(int N, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= N) return;

    float mhat = m[index] / (1.f - powf(B1, t));
    float vhat = v[index] / (1.f - powf(B2, t));

    x[index] = x[index] + rate * mhat / (sqrtf(vhat) + eps);
}

extern "C" void adam_gpu(int n, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
    adam_kernel << <cuda_gridsize(n), BLOCK, 0, get_cuda_stream() >> >(n, x, m, v, B1, B2, rate, eps, t);
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void adam_update_gpu(float *w, float *d, float *m, float *v, float B1, float B2, float eps, float decay, float rate, int n, int batch, int t)
{
    scal_ongpu(n, B1, m, 1);
    scal_ongpu(n, B2, v, 1);
    axpy_ongpu(n, -decay*batch, w, 1, d, 1);

    axpy_ongpu(n, (1 - B1), d, 1, m, 1);
    mul_ongpu(n, d, 1, d, 1);
    axpy_ongpu(n, (1 - B2), d, 1, v, 1);

    adam_gpu(n, w, m, v, B1, B2, rate, eps, t);
    fill_ongpu(n, 0, d, 1);
    CHECK_CUDA(hipPeekAtLastError());
}

__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    const int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index >= N) return;
    int f = (index / spatial) % filters;

    x[index] = (x[index] - mean[f]) / (sqrtf(variance[f] + .00001f));
}

extern "C" void normalize_gpu(float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    const int current_size = batch * filters * spatial;
    const int num_blocks = get_number_of_blocks(current_size, BLOCK);

    normalize_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> >(current_size, x, mean, variance, batch, filters, spatial);
    CHECK_CUDA(hipPeekAtLastError());
}

__global__ void normalize_delta_kernel(int N, float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= N) return;
    int f = (index/spatial)%filters;

    delta[index] = delta[index] * 1.F/(sqrtf(variance[f]) + .000001f) + variance_delta[f] * 2. * (x[index] - mean[f]) / (spatial * batch) + mean_delta[f]/(spatial*batch);
}

extern "C" void normalize_delta_gpu(float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    size_t N = batch*filters*spatial;
    normalize_delta_kernel<<<cuda_gridsize(N), BLOCK, 0, get_cuda_stream() >>>(N, x, mean, variance, mean_delta, variance_delta, batch, filters, spatial, delta);
    CHECK_CUDA(hipPeekAtLastError());
}

__global__ void  variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= filters) return;
    int j,k;
    variance_delta[i] = 0;
    for(j = 0; j < batch; ++j){
        for(k = 0; k < spatial; ++k){
            int index = j*filters*spatial + i*spatial + k;
            variance_delta[i] += delta[index]*(x[index] - mean[i]);
        }
    }
    variance_delta[i] *= -.5 * powf(variance[i] + .000001f, (float)(-3./2.));
}

__global__ void accumulate_kernel(float *x, int n, int groups, float *sum)
{
    int k;
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= groups) return;
    sum[i] = 0;
    for(k = 0; k < n; ++k){
        sum[i] += x[k*groups + i];
    }
}

__global__ void fast_mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    int i, j;
    for(j = 0; j < batch; ++j){
        for(i = 0; i < spatial; i += threads){
            int index = j*spatial*filters + filter*spatial + i + id;
            local[id] += (i+id < spatial) ? delta[index] : 0;
        }
    }
    __syncthreads();

    if(id == 0){
        mean_delta[filter] = 0;
        for(i = 0; i < threads; ++i){
            mean_delta[filter] += local[i];
        }
        mean_delta[filter] *= (-1.F/sqrtf(variance[filter] + .000001f));
    }
}

__global__ void  fast_variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    int i, j;
    for(j = 0; j < batch; ++j){
        for(i = 0; i < spatial; i += threads){
            int index = j*spatial*filters + filter*spatial + i + id;

            local[id] += (i+id < spatial) ? delta[index]*(x[index] - mean[filter]) : 0;
        }
    }
    __syncthreads();

    if(id == 0){
        variance_delta[filter] = 0;
        for(i = 0; i < threads; ++i){
            variance_delta[filter] += local[i];
        }
        variance_delta[filter] *= -.5 * powf(variance[filter] + .000001f, (float)(-3./2.));
    }
}


__global__ void mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= filters) return;
    int j,k;
    mean_delta[i] = 0;
    for (j = 0; j < batch; ++j) {
        for (k = 0; k < spatial; ++k) {
            int index = j*filters*spatial + i*spatial + k;
            mean_delta[i] += delta[index];
        }
    }
    mean_delta[i] *= (-1.F/sqrtf(variance[i] + .000001f));
}

extern "C" void mean_delta_gpu(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    mean_delta_kernel<<<cuda_gridsize(filters), BLOCK, 0, get_cuda_stream() >>>(delta, variance, batch, filters, spatial, mean_delta);
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void fast_mean_delta_gpu(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    fast_mean_delta_kernel<<<filters, BLOCK, 0, get_cuda_stream() >>>(delta, variance, batch, filters, spatial, mean_delta);
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void fast_variance_delta_gpu(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    fast_variance_delta_kernel<<<filters, BLOCK, 0, get_cuda_stream() >>>(x, delta, mean, variance, batch, filters, spatial, variance_delta);
    CHECK_CUDA(hipPeekAtLastError());
}

__global__ void  mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    float scale = 1.F/(batch * spatial);
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= filters) return;
    int j,k;
    mean[i] = 0;
    for(j = 0; j < batch; ++j){
        for(k = 0; k < spatial; ++k){
            int index = j*filters*spatial + i*spatial + k;
            mean[i] += x[index];
        }
    }
    mean[i] *= scale;
}

__global__ void variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    float scale = 1.F/(batch * spatial - 1);
    int j,k;
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= filters) return;
    variance[i] = 0;
    for(j = 0; j < batch; ++j){
        for(k = 0; k < spatial; ++k){
            int index = j*filters*spatial + i*spatial + k;
            variance[i] += powf((x[index] - mean[i]), 2);
        }
    }
    variance[i] *= scale;
}

__global__ void reorg_kernel(int N, float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i >= N) return;
    int in_index = i;
    int in_w = i%w;
    i = i/w;
    int in_h = i%h;
    i = i/h;
    int in_c = i%c;
    i = i/c;
    int b = i%batch;

    int out_c = c/(stride*stride);

    int c2 = in_c % out_c;
    int offset = in_c / out_c;
    int w2 = in_w*stride + offset % stride;
    int h2 = in_h*stride + offset / stride;
    //printf("%d\n", offset);
    int out_index = w2 + w*stride*(h2 + h*stride*(c2 + out_c*b));

   // printf("%d %d %d\n", w2, h2, c2);
    //printf("%d %d\n", in_index, out_index);
    //if(out_index >= N || out_index < 0) printf("bad bad bad \n");

    if(forward) out[out_index] = x[in_index];
    else out[in_index] = x[out_index];
    //if(forward) out[1] = x[1];
    //else out[0] = x[0];
}

__global__ void constrain_weight_updates_kernel(int N, float coef, float *weights_gpu, float *weight_updates_gpu)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i < N) {
        const float w = weights_gpu[i];
        const float wu = weight_updates_gpu[i];
        const float wu_sign = (wu == 0) ? 0 : (fabs(wu) / wu);
        const float abs_limit = fabs(w * coef);
        if (fabs(wu) > abs_limit) weight_updates_gpu[i] = abs_limit * wu_sign;
    }
}

extern "C" void constrain_weight_updates_ongpu(int N, float coef, float *weights_gpu, float *weight_updates_gpu)
{
    constrain_weight_updates_kernel << <cuda_gridsize(N), BLOCK, 0, get_cuda_stream() >> >(N, coef, weights_gpu, weight_updates_gpu);
    CHECK_CUDA(hipPeekAtLastError());
}

__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX,  float *Y, int OFFY, int INCY)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) Y[OFFY+i*INCY] += ALPHA*X[OFFX+i*INCX];
}

__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) Y[i*INCY] = powf(X[i*INCX], ALPHA);
}

__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) X[i*INCX] = ALPHA;
}

__global__ void constrain_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) X[i*INCX] = fminf(ALPHA, fmaxf(-ALPHA, X[i*INCX]));
}
__global__ void constrain_min_max_kernel(int N, float MIN, float MAX, float *X, int INCX)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i < N) X[i*INCX] = fminf(MAX, fmaxf(MIN, X[i*INCX]));
}

__global__ void supp_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) {
        if((X[i*INCX] * X[i*INCX]) < (ALPHA * ALPHA)) X[i*INCX] = 0;
    }
}

__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) X[i*INCX] *= ALPHA;
}

__global__ void scal_add_kernel(int N, float ALPHA, float BETA, float *X, int INCX)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i < N) X[i*INCX] = X[i*INCX] * ALPHA + BETA;
}

__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    const int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index >= N) return;
    X[index*INCX] = ALPHA;
}

__global__ void mask_kernel_new_api(int n, float *x, float mask_num, float *mask, float val)
{
	int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if (i < n && mask[i] == mask_num) x[i] = val;
}

__global__ void mask_kernel(int n, float *x, float mask_num, float *mask)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < n && mask[i] == mask_num) x[i] = mask_num;
}

__global__ void copy_kernel(int N,  float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) Y[i*INCY + OFFY] = X[i*INCX + OFFX];
}

__global__ void simple_copy_kernel(int size, float *src, float *dst)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < size)
        dst[index] = src[index];
}

__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) Y[i*INCY] *= X[i*INCX];
}


__global__ void  fast_mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    int i, j;
    for(j = 0; j < batch; ++j){
        for(i = 0; i < spatial; i += threads){
            int index = j*spatial*filters + filter*spatial + i + id;
            local[id] += (i+id < spatial) ? x[index] : 0;
        }
    }
    __syncthreads();

    if(id == 0){
        float mean_tmp = 0;
        for(i = 0; i < threads; ++i){
            mean_tmp += local[i];
        }
        mean_tmp /= spatial * batch;
        mean[filter] = mean_tmp;
    }
}

extern "C" void fast_mean_gpu(float *x, int batch, int filters, int spatial, float *mean)
{
    fast_mean_kernel << <filters, BLOCK, 0, get_cuda_stream() >> >(x, batch, filters, spatial, mean);
    CHECK_CUDA(hipPeekAtLastError());
}

__global__ void  fast_variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    int i, j;
    for(j = 0; j < batch; ++j){
        for(i = 0; i < spatial; i += threads){
            int index = j*spatial*filters + filter*spatial + i + id;

            local[id] += (i+id < spatial) ? powf((x[index] - mean[filter]), 2) : 0;
        }
    }
    __syncthreads();

    if(id == 0){
        float variance_tmp = 0;
        for(i = 0; i < threads; ++i){
            variance_tmp += local[i];
        }
        variance_tmp /= (spatial * batch);// -1);
        variance[filter] = variance_tmp;
    }
}

extern "C" void fast_variance_gpu(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    fast_variance_kernel<<<filters, BLOCK, 0, get_cuda_stream() >>>(x, mean, batch, filters, spatial, variance);
    CHECK_CUDA(hipPeekAtLastError());
}


__global__ void  fast_v_cbn_kernel(const float *x, float *mean, int batch, int filters, int spatial, int minibatch_index, int max_minibatch_index, float *m_avg, float *v_avg, float *variance,
    const float alpha, float *rolling_mean_gpu, float *rolling_variance_gpu, int inverse_variance, float epsilon)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    int i, j;
    for (j = 0; j < batch; ++j) {
        for (i = 0; i < spatial; i += threads) {
            int index = j*spatial*filters + filter*spatial + i + id;

            local[id] += (i + id < spatial) ? powf(x[index], 2) : 0;
        }
    }
    __syncthreads();

    if (id == 0) {
        float v_tmp = 0;
        v_tmp = 0;
        for (i = 0; i < threads; ++i) {
            v_tmp += local[i];
        }
        v_tmp /= (spatial * batch - 1);

        v_tmp = fmax(v_tmp, powf(mean[filter], 2));


        const float alpha_cbn = 1.0f / minibatch_index;

        m_avg[filter] = alpha_cbn * mean[filter] + (1 - alpha_cbn) * m_avg[filter];
        mean[filter] = m_avg[filter];

        v_avg[filter] = alpha_cbn * v_tmp + (1 - alpha_cbn) * v_avg[filter];

        float variance_tmp = fmax(0.0f, v_avg[filter] - powf(m_avg[filter], 2));
        if (inverse_variance) variance[filter] = 1.0f / sqrtf(variance_tmp + epsilon);
        else variance[filter] = variance_tmp;

        //if (max_minibatch_index == minibatch_index)
        {
            if(rolling_mean_gpu) rolling_mean_gpu[filter] = alpha * mean[filter] + (1 - alpha) * rolling_mean_gpu[filter];

            if(rolling_variance_gpu) rolling_variance_gpu[filter] = alpha * variance_tmp + (1 - alpha) * rolling_variance_gpu[filter];
        }
    }
}

extern "C" void fast_v_cbn_gpu(const float *x, float *mean, int batch, int filters, int spatial, int minibatch_index, int max_minibatch_index, float *m_avg, float *v_avg, float *variance,
    const float alpha, float *rolling_mean_gpu, float *rolling_variance_gpu, int inverse_variance, float epsilon)
{
    fast_v_cbn_kernel << <filters, BLOCK, 0, get_cuda_stream() >> >(x, mean, batch, filters, spatial, minibatch_index, max_minibatch_index, m_avg, v_avg, variance, alpha, rolling_mean_gpu, rolling_variance_gpu, inverse_variance, epsilon);
    CHECK_CUDA(hipPeekAtLastError());
}

__global__ void inverse_variance_kernel(int size, float *src, float *dst, float epsilon)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < size)
        dst[index] = 1.0f / sqrtf(src[index] + epsilon);
}

extern "C" void inverse_variance_ongpu(int size, float *src, float *dst, float epsilon)
{
    const int num_blocks = size / BLOCK + 1;
    inverse_variance_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> >(size, src, dst, epsilon);
    CHECK_CUDA(hipPeekAtLastError());
}

__global__ void normalize_scale_bias_kernel(int N, float *x, float *mean, float *variance, float *scales, float *biases, int batch, int filters, int spatial, int inverse_variance, float epsilon)
{
    const int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index >= N) return;
    int f = (index / spatial) % filters;

    float val = 0;
    if(inverse_variance) val = (x[index] - mean[f]) * variance[f];
    else val = (x[index] - mean[f]) / (sqrtf(variance[f] + epsilon));
    val *= scales[f];
    val += biases[f];

    if (!isnan(val) && !isinf(val))
        x[index] = val;
}

extern "C" void normalize_scale_bias_gpu(float *x, float *mean, float *variance, float *scales, float *biases, int batch, int filters, int spatial, int inverse_variance, float epsilon)
{
    const int current_size = batch * filters * spatial;
    const int num_blocks = get_number_of_blocks(current_size, BLOCK);

    normalize_scale_bias_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> >(current_size, x, mean, variance, scales, biases, batch, filters, spatial, inverse_variance, epsilon);
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void mean_gpu(float *x, int batch, int filters, int spatial, float *mean)
{
    mean_kernel<<<cuda_gridsize(filters), BLOCK, 0, get_cuda_stream() >>>(x, batch, filters, spatial, mean);
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void variance_gpu(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    variance_kernel<<<cuda_gridsize(filters), BLOCK, 0, get_cuda_stream() >>>(x, mean, batch, filters, spatial, variance);
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void axpy_ongpu(int N, float ALPHA, float * X, int INCX, float * Y, int INCY)
{
    axpy_ongpu_offset(N, ALPHA, X, 0, INCX, Y, 0, INCY);
}

extern "C" void pow_ongpu(int N, float ALPHA, float * X, int INCX, float * Y, int INCY)
{
    pow_kernel<<<cuda_gridsize(N), BLOCK, 0, get_cuda_stream() >>>(N, ALPHA, X, INCX, Y, INCY);
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void axpy_ongpu_offset(int N, float ALPHA, float * X, int OFFX, int INCX, float * Y, int OFFY, int INCY)
{
    axpy_kernel<<<cuda_gridsize(N), BLOCK, 0, get_cuda_stream()>>>(N, ALPHA, X, OFFX, INCX, Y, OFFY, INCY);
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void copy_ongpu(int N, float * X, int INCX, float * Y, int INCY)
{
    copy_ongpu_offset(N, X, 0, INCX, Y, 0, INCY);
}

extern "C" void simple_copy_ongpu(int size, float *src, float *dst)
{
    const int num_blocks = size / BLOCK + 1;
    simple_copy_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> >(size, src, dst);
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void memcpy_ongpu(void *dst, void *src, int size_bytes)
{
    CHECK_CUDA(hipMemcpyAsync(dst, src, size_bytes, hipMemcpyDefault, get_cuda_stream()));
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void mul_ongpu(int N, float * X, int INCX, float * Y, int INCY)
{
    mul_kernel<<<cuda_gridsize(N), BLOCK, 0, get_cuda_stream() >>>(N, X, INCX, Y, INCY);
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void copy_ongpu_offset(int N, float * X, int OFFX, int INCX, float * Y, int OFFY, int INCY)
{
    copy_kernel<<<cuda_gridsize(N), BLOCK, 0, get_cuda_stream()>>>(N, X, OFFX, INCX, Y, OFFY, INCY);
    CHECK_CUDA(hipPeekAtLastError());
}

__global__ void flatten_kernel(int N, float *x, int spatial, int layers, int batch, int forward, float *out)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i >= N) return;
    int in_s = i%spatial;
    i = i/spatial;
    int in_c = i%layers;
    i = i/layers;
    int b = i;

    int i1 = b*layers*spatial + in_c*spatial + in_s;
    int i2 = b*layers*spatial + in_s*layers +  in_c;

    if (forward) out[i2] = x[i1];
    else out[i1] = x[i2];
}

extern "C" void flatten_ongpu(float *x, int spatial, int layers, int batch, int forward, float *out)
{
    int size = spatial*batch*layers;
    flatten_kernel<<<cuda_gridsize(size), BLOCK, 0, get_cuda_stream()>>>(size, x, spatial, layers, batch, forward, out);
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void reorg_ongpu(float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
    int size = w*h*c*batch;
    reorg_kernel<<<cuda_gridsize(size), BLOCK, 0, get_cuda_stream()>>>(size, x, w, h, c, batch, stride, forward, out);
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void mask_gpu_new_api(int N, float * X, float mask_num, float * mask, float val)
{
	mask_kernel_new_api <<<cuda_gridsize(N), BLOCK, 0, get_cuda_stream() >>>(N, X, mask_num, mask, val);
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void mask_ongpu(int N, float * X, float mask_num, float * mask)
{
    mask_kernel<<<cuda_gridsize(N), BLOCK, 0, get_cuda_stream() >>>(N, X, mask_num, mask);
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void const_ongpu(int N, float ALPHA, float * X, int INCX)
{
    const_kernel<<<cuda_gridsize(N), BLOCK, 0, get_cuda_stream() >>>(N, ALPHA, X, INCX);
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void constrain_ongpu(int N, float ALPHA, float * X, int INCX)
{
    constrain_kernel<<<cuda_gridsize(N), BLOCK, 0, get_cuda_stream() >>>(N, ALPHA, X, INCX);
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void constrain_min_max_ongpu(int N, float MIN, float MAX, float * X, int INCX)
{
    constrain_min_max_kernel << <cuda_gridsize(N), BLOCK, 0, get_cuda_stream() >> >(N, MIN, MAX, X, INCX);
    CHECK_CUDA(hipPeekAtLastError());
}


extern "C" void scal_ongpu(int N, float ALPHA, float * X, int INCX)
{
    scal_kernel<<<cuda_gridsize(N), BLOCK, 0, get_cuda_stream()>>>(N, ALPHA, X, INCX);
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void scal_add_ongpu(int N, float ALPHA, float BETA, float * X, int INCX)
{
    scal_add_kernel << <cuda_gridsize(N), BLOCK, 0, get_cuda_stream() >> >(N, ALPHA, BETA, X, INCX);
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void supp_ongpu(int N, float ALPHA, float * X, int INCX)
{
    supp_kernel<<<cuda_gridsize(N), BLOCK, 0, get_cuda_stream() >>>(N, ALPHA, X, INCX);
    CHECK_CUDA(hipPeekAtLastError());
}

extern "C" void fill_ongpu(int N, float ALPHA, float * X, int INCX)
{
    //fill_kernel<<<cuda_gridsize(N), BLOCK, 0, get_cuda_stream()>>>(N, ALPHA, X, INCX);
    //CHECK_CUDA(hipPeekAtLastError());
    fill_kernel << <get_number_of_blocks(N, BLOCK), BLOCK, 0, get_cuda_stream() >> >(N, ALPHA, X, INCX);
    CHECK_CUDA(hipPeekAtLastError());
}

__global__ void gradient_centralization_kernel(int filters, int f_size, float *in)
{
    const int index = blockIdx.x*blockDim.x + threadIdx.x;
    const int tid = index % WARP_SIZE;
    const int f = index / WARP_SIZE;

    if (f >= filters) return;

    float mean = 0;
    for (int i = 0; i < f_size; i += WARP_SIZE) {
        mean += warpAllReduceSum(in[f*f_size + i + tid]);
    }
    mean = mean / f_size;
    for (int i = 0; i < f_size; i += WARP_SIZE) {
        in[f*f_size + i + tid] -= mean;
    }

}

extern "C" void gradient_centralization_gpu(int w, int h, int c, int f, float *in)
{
    const int size = f * WARP_SIZE;
    const int f_size = c * h * w;
    if (f_size % WARP_SIZE == 0) {

        gradient_centralization_kernel << <get_number_of_blocks(size, BLOCK), BLOCK, 0, get_cuda_stream() >> > (f, f_size, in);
        CHECK_CUDA(hipPeekAtLastError());
    }
}

__device__ float relu(float src) {
    if (src > 0) return src;
    return 0;
}

__device__ float lrelu(float src) {
    const float eps = 0.001;
    if (src > eps) return src;
    return eps;
}

__device__ float grad_relu(float src) {
    return (src > 0);
}

__device__ float grad_lrelu(float src) {
    const float eps = 0.001;
    return (src > eps);
}

__global__ void shortcut_singlelayer_simple_kernel(int size, int src_outputs, int batch, int n, int *outputs_of_layers_gpu, float **layers_output_gpu, float *out, float *in, float *weights_gpu, int nweights, WEIGHTS_NORMALIZATION_T weights_normalization)
{
    const int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= size) return;

    int src_id = id;
    const int src_i = src_id % src_outputs;
    src_id /= src_outputs;
    int src_b = src_id;

    float out_val = in[id];

    int add_outputs = outputs_of_layers_gpu[0];
    if (src_i < add_outputs) {
        int add_index = add_outputs*src_b + src_i;

        float *add = layers_output_gpu[0];
        out_val += add[add_index];
    }
    out[id] = out_val;
}

__global__ void shortcut_multilayer_kernel(int size, int src_outputs, int batch, int n, int *outputs_of_layers_gpu, float **layers_output_gpu, float *out, float *in, float *weights_gpu, int nweights, WEIGHTS_NORMALIZATION_T weights_normalization)
{
    const int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= size) return;

    // nweights - l.n or l.n*l.c or (l.n*l.c*l.h*l.w)
    const int layer_step = nweights / (n + 1);    // 1 or l.c or (l.c * l.h * l.w)
    int step = 0;
    if (nweights > 0) step = src_outputs / layer_step; // (l.c * l.h * l.w) or (l.w*l.h) or 1

    int src_id = id;
    const int src_i = src_id % src_outputs;
    src_id /= src_outputs;
    int src_b = src_id;

    float sum = 1, max_val = -FLT_MAX;
    if (weights_gpu && weights_normalization) {
        if (weights_normalization == SOFTMAX_NORMALIZATION) {
            for (int i = 0; i < (n + 1); ++i) {
                const int weights_index = src_i / step + i*layer_step;  // [0 or c or (c, h ,w)]
                const float w = weights_gpu[weights_index];
                if (max_val < w) max_val = w;
            }
        }
        const float eps = 0.0001;
        sum = eps;
        for (int i = 0; i < (n + 1); ++i) {
            const int weights_index = src_i / step + i*layer_step;  // [0 or c or (c, h ,w)]
            const float w = weights_gpu[weights_index];
            if (weights_normalization == RELU_NORMALIZATION) sum += lrelu(w);
            else if (weights_normalization == SOFTMAX_NORMALIZATION) sum += expf(w - max_val);
        }
    }

    float out_val = 0;

    if (weights_gpu) {
        float w = weights_gpu[src_i / step];
        if (weights_normalization == RELU_NORMALIZATION) w = lrelu(w) / sum;
        else if (weights_normalization == SOFTMAX_NORMALIZATION) w = expf(w - max_val) / sum;

        out_val = in[id] * w; // [0 or c or (c, h ,w)]
    }
    else out_val = in[id];

    // layers
    for (int i = 0; i < n; ++i) {
        int add_outputs = outputs_of_layers_gpu[i];
        if (src_i < add_outputs) {
            int add_index = add_outputs*src_b + src_i;

            float *add = layers_output_gpu[i];

            if (weights_gpu) {
                const int weights_index = src_i / step + (i + 1)*layer_step;  // [0 or c or (c, h ,w)]
                float w = weights_gpu[weights_index];
                if (weights_normalization == RELU_NORMALIZATION) w = lrelu(w) / sum;
                else if (weights_normalization == SOFTMAX_NORMALIZATION) w = expf(w - max_val) / sum;

                out_val += add[add_index] * w; // [0 or c or (c, h ,w)]
            }
            else out_val += add[add_index];
        }
    }
    out[id] = out_val;
}

extern "C" void shortcut_multilayer_gpu(int src_outputs, int batch, int n, int *outputs_of_layers_gpu, float **layers_output_gpu, float *out, float *in, float *weights_gpu, int nweights, WEIGHTS_NORMALIZATION_T weights_normalization)
{
    //printf(" src_outputs = %d, batch = %d, n = %d \n", src_outputs, batch, n);
    int size = batch * src_outputs;
    if (nweights == 0 && n == 1) {
        shortcut_singlelayer_simple_kernel << <cuda_gridsize(size), BLOCK, 0, get_cuda_stream() >> > (size, src_outputs, batch, n, outputs_of_layers_gpu, layers_output_gpu, out, in, weights_gpu, nweights, weights_normalization);
    }
    else {
        shortcut_multilayer_kernel << <cuda_gridsize(size), BLOCK, 0, get_cuda_stream() >> > (size, src_outputs, batch, n, outputs_of_layers_gpu, layers_output_gpu, out, in, weights_gpu, nweights, weights_normalization);
    }
    CHECK_CUDA(hipPeekAtLastError());
}


__global__ void backward_shortcut_multilayer_kernel(int size, int src_outputs, int batch, int n, int *outputs_of_layers_gpu,
    float **layers_delta_gpu, float *delta_out, float *delta_in, float *weights_gpu, float *weight_updates_gpu, int nweights, float *in, float **layers_output_gpu, WEIGHTS_NORMALIZATION_T weights_normalization)
{
    const int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= size) return;

    // nweights - l.n or l.n*l.c or (l.n*l.c*l.h*l.w)
    const int layer_step = nweights / (n + 1);    // 1 or l.c or (l.c * l.h * l.w)
    int step = 0;
    if (nweights > 0) step = src_outputs / layer_step; // (l.c * l.h * l.w) or (l.w*l.h) or 1

    int src_id = id;
    const int src_i = src_id % src_outputs;
    src_id /= src_outputs;
    int src_b = src_id;

    float grad = 1, sum = 1, max_val = -FLT_MAX;
    int i;
    if (weights_gpu && weights_normalization) {
        if (weights_normalization == SOFTMAX_NORMALIZATION) {
            for (int i = 0; i < (n + 1); ++i) {
                const int weights_index = src_i / step + i*layer_step;  // [0 or c or (c, h ,w)]
                float w = weights_gpu[weights_index];
                if (max_val < w) max_val = w;
            }
        }
        const float eps = 0.0001;
        sum = eps;
        for (i = 0; i < (n + 1); ++i) {
            const int weights_index = src_i / step + i*layer_step;  // [0 or c or (c, h ,w)]
            const float w = weights_gpu[weights_index];
            if (weights_normalization == RELU_NORMALIZATION) sum += lrelu(w);
            else if (weights_normalization == SOFTMAX_NORMALIZATION) sum += expf(w - max_val);
        }

    }

    if (weights_gpu) {
        float w = weights_gpu[src_i / step];
        if (weights_normalization == RELU_NORMALIZATION) w = lrelu(w) / sum;
        else if (weights_normalization == SOFTMAX_NORMALIZATION) w = expf(w - max_val) / sum;

        if (weights_normalization == RELU_NORMALIZATION) grad = w;
        else if (weights_normalization == SOFTMAX_NORMALIZATION) grad = w*(1-w);

        delta_out[id] += delta_in[id] * w; // [0 or c or (c, h ,w)]
        float weights_update_tmp = delta_in[id] * in[id] * grad;// / step;

        if (layer_step == 1 && (size/32) > (id/32 + 1)) {
            if (isnan(weights_update_tmp) || isinf(weights_update_tmp)) {
                weights_update_tmp = 0;
            }
            float wu = warpAllReduceSum(weights_update_tmp);
            if (threadIdx.x % 32 == 0) {
                if (!isnan(wu) && !isinf(wu))
                    atomicAdd(&weight_updates_gpu[src_i / step], wu);
            }
        }
        else {
            if (!isnan(weights_update_tmp) && !isinf(weights_update_tmp))
                atomicAdd(&weight_updates_gpu[src_i / step], weights_update_tmp);
                //weight_updates_gpu[src_i / step] += weights_update_tmp;
        }
    }
    else delta_out[id] += delta_in[id];

    // layers
    for (int i = 0; i < n; ++i) {
        int add_outputs = outputs_of_layers_gpu[i];
        if (src_i < add_outputs) {
            int add_index = add_outputs*src_b + src_i;
            int out_index = id;

            float *layer_delta = layers_delta_gpu[i];
            if (weights_gpu) {
                float *add = layers_output_gpu[i];

                const int weights_index = src_i / step + (i + 1)*layer_step;  // [0 or c or (c, h ,w)]
                float w = weights_gpu[weights_index];
                if (weights_normalization == RELU_NORMALIZATION) w = lrelu(w) / sum;
                else if (weights_normalization == SOFTMAX_NORMALIZATION) w = expf(w - max_val) / sum;

                if (weights_normalization == RELU_NORMALIZATION) grad = w;
                else if (weights_normalization == SOFTMAX_NORMALIZATION) grad = w*(1 - w);

                layer_delta[add_index] += delta_in[id] * w;
                float weights_update_tmp = delta_in[id] * add[add_index] * grad;// / step;

                if (layer_step == 1 && (size / 32) > (id / 32 + 1)) {
                    if (isnan(weights_update_tmp) || isinf(weights_update_tmp)) {
                        weights_update_tmp = 0;
                    }
                    float wu = warpAllReduceSum(weights_update_tmp);
                    if (threadIdx.x % 32 == 0) {
                        if (!isnan(wu) && !isinf(wu))
                            atomicAdd(&weight_updates_gpu[weights_index], wu);
                        //if(weights_gpu[weights_index] != 1) printf(" wu = %f, weights_update_tmp = %f, w = %f, weights_gpu[weights_index] = %f, grad = %f, weights_normalization = %d ",
                        //    wu, weights_update_tmp, w, weights_gpu[weights_index], grad, weights_normalization);
                    }
                }
                else {
                    if (!isnan(weights_update_tmp) && !isinf(weights_update_tmp))
                        atomicAdd(&weight_updates_gpu[weights_index], weights_update_tmp);
                        //weight_updates_gpu[weights_index] += weights_update_tmp;
                }
            }
            else layer_delta[add_index] += delta_in[id];
        }
    }
}

extern "C" void backward_shortcut_multilayer_gpu(int src_outputs, int batch, int n, int *outputs_of_layers_gpu,
    float **layers_delta_gpu, float *delta_out, float *delta_in, float *weights_gpu, float *weight_updates_gpu, int nweights, float *in, float **layers_output_gpu, WEIGHTS_NORMALIZATION_T weights_normalization)
{
    const int layer_step = nweights / (n + 1);    // 1 or l.c or (l.c * l.h * l.w)
    int step = 0;
    if (nweights > 0) step = src_outputs / layer_step; // (l.c * l.h * l.w) or (l.w*l.h) or 1
    //printf(" nweights = %d, n = %d, layer_step = %d, step = %d \n", nweights, n, layer_step, step);

    //printf(" src_outputs = %d, batch = %d, n = %d \n", src_outputs, batch, n);
    int size = batch * src_outputs;
    backward_shortcut_multilayer_kernel << <cuda_gridsize(size), BLOCK, 0, get_cuda_stream() >> > (size, src_outputs, batch, n, outputs_of_layers_gpu,
        layers_delta_gpu, delta_out, delta_in, weights_gpu, weight_updates_gpu, nweights, in, layers_output_gpu, weights_normalization);
    CHECK_CUDA(hipPeekAtLastError());
}

__global__ void shortcut_kernel(int size, int minw, int minh, int minc, int stride, int sample, int batch, int w1, int h1, int c1, float *add, int w2, int h2, int c2, float *out)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= size) return;
    int i = id % minw;
    id /= minw;
    int j = id % minh;
    id /= minh;
    int k = id % minc;
    id /= minc;
    int b = id % batch;

    int out_index = i*sample + w2*(j*sample + h2*(k + c2*b));
    int add_index = i*stride + w1*(j*stride + h1*(k + c1*b));
    out[out_index] += add[add_index];
}

extern "C" void shortcut_gpu(int batch, int w1, int h1, int c1, float *add, int w2, int h2, int c2, float *out)
{
    int minw = (w1 < w2) ? w1 : w2;
    int minh = (h1 < h2) ? h1 : h2;
    int minc = (c1 < c2) ? c1 : c2;

    int stride = w1/w2;
    int sample = w2/w1;
    assert(stride == h1/h2);
    assert(sample == h2/h1);
    if(stride < 1) stride = 1;
    if(sample < 1) sample = 1;

    int size = batch * minw * minh * minc;
    shortcut_kernel<<<cuda_gridsize(size), BLOCK, 0, get_cuda_stream()>>>(size, minw, minh, minc, stride, sample, batch, w1, h1, c1, add, w2, h2, c2, out);
    CHECK_CUDA(hipPeekAtLastError());
}

__global__ void simple_input_shortcut_kernel(float *in, int size, float *add, float *out)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= size) return;

    out[id] = in[id] + add[id];
}

__global__ void input_shortcut_kernel(float *in, int size, int minw, int minh, int minc, int stride, int sample, int batch, int w1, int h1, int c1, float *add, int w2, int h2, int c2, float *out)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= size) return;
    int i = id % minw;
    id /= minw;
    int j = id % minh;
    id /= minh;
    int k = id % minc;
    id /= minc;
    int b = id % batch;

    int out_index = i*sample + w2*(j*sample + h2*(k + c2*b));
    int add_index = i*stride + w1*(j*stride + h1*(k + c1*b));
    out[out_index] = in[out_index] + add[add_index];
}

extern "C" void input_shortcut_gpu(float *in, int batch, int w1, int h1, int c1, float *add, int w2, int h2, int c2, float *out)
{
    if (w1 == w2 && h1 == h2 && c1 == c2) {
        int size = batch * w1 * h1 * c1;
        simple_input_shortcut_kernel << <cuda_gridsize(size), BLOCK, 0, get_cuda_stream() >> >(in, size, add, out);
        CHECK_CUDA(hipPeekAtLastError());
        return;
    }

    int minw = (w1 < w2) ? w1 : w2;
    int minh = (h1 < h2) ? h1 : h2;
    int minc = (c1 < c2) ? c1 : c2;

    int stride = w1 / w2;
    int sample = w2 / w1;
    assert(stride == h1 / h2);
    assert(sample == h2 / h1);
    if (stride < 1) stride = 1;
    if (sample < 1) sample = 1;

    int size = batch * minw * minh * minc;
    //input_shortcut_kernel << <cuda_gridsize(size), BLOCK, 0, get_cuda_stream() >> >(in, size, minw, minh, minc, stride, sample, batch, w1, h1, c1, add, w2, h2, c2, out);
    simple_copy_ongpu(w2 * h2 * c2 * batch, in, out);
    shortcut_kernel << <cuda_gridsize(size), BLOCK, 0, get_cuda_stream() >> >(size, minw, minh, minc, stride, sample, batch, w1, h1, c1, add, w2, h2, c2, out);
    CHECK_CUDA(hipPeekAtLastError());
}

__global__ void smooth_l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < n){
        float diff = truth[i] - pred[i];
        float abs_val = abs(diff);
        if(abs_val < 1) {
            error[i] = diff * diff;
            delta[i] = diff;
        }
        else {
            error[i] = 2*abs_val - 1;
            delta[i] = (diff < 0) ? -1 : 1;
        }
    }
}

extern "C" void smooth_l1_gpu(int n, float *pred, float *truth, float *delta, float *error)
{
    smooth_l1_kernel<<<cuda_gridsize(n), BLOCK, 0, get_cuda_stream() >>>(n, pred, truth, delta, error);
    CHECK_CUDA(hipPeekAtLastError());
}

__global__ void softmax_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
	int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if (i < n) {
		float t = truth[i];
		float p = pred[i];
		error[i] = (t) ? -log(p) : 0;
		delta[i] = t - p;
	}
}

extern "C" void softmax_x_ent_gpu(int n, float *pred, float *truth, float *delta, float *error)
{
	softmax_x_ent_kernel << <cuda_gridsize(n), BLOCK, 0, get_cuda_stream() >> >(n, pred, truth, delta, error);
    CHECK_CUDA(hipPeekAtLastError());
}

__global__ void l2_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < n){
        float diff = truth[i] - pred[i];
        error[i] = diff * diff; //I know this is technically wrong, deal with it.
        delta[i] = diff;
    }
}

extern "C" void l2_gpu(int n, float *pred, float *truth, float *delta, float *error)
{
    l2_kernel<<<cuda_gridsize(n), BLOCK, 0, get_cuda_stream() >>>(n, pred, truth, delta, error);
    CHECK_CUDA(hipPeekAtLastError());
}



__global__ void weighted_sum_kernel(int n, float *a, float *b, float *s, float *c)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < n){
        c[i] = s[i]*a[i] + (1-s[i])*(b ? b[i] : 0);
    }
}

extern "C" void weighted_sum_gpu(float *a, float *b, float *s, int num, float *c)
{
    weighted_sum_kernel<<<cuda_gridsize(num), BLOCK, 0, get_cuda_stream() >>>(num, a, b, s, c);
    CHECK_CUDA(hipPeekAtLastError());
}

__global__ void weighted_delta_kernel(int n, float *a, float *b, float *s, float *da, float *db, float *ds, float *dc)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < n){
        if(da) da[i] += dc[i] * s[i];
        db[i] += dc[i] * (1-s[i]);
        ds[i] += dc[i] * a[i] + dc[i] * -b[i];
    }
}

extern "C" void weighted_delta_gpu(float *a, float *b, float *s, float *da, float *db, float *ds, int num, float *dc)
{
    weighted_delta_kernel<<<cuda_gridsize(num), BLOCK, 0, get_cuda_stream() >>>(num, a, b, s, da, db, ds, dc);
    CHECK_CUDA(hipPeekAtLastError());
}

__global__ void mult_add_into_kernel(int n, float *a, float *b, float *c)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < n){
        c[i] += a[i]*b[i];
    }
}

extern "C" void mult_add_into_gpu(int num, float *a, float *b, float *c)
{
    mult_add_into_kernel<<<cuda_gridsize(num), BLOCK, 0, get_cuda_stream() >>>(num, a, b, c);
    CHECK_CUDA(hipPeekAtLastError());
}


__device__ void softmax_device(int n, float *input, float temp, float *output)
{
    int i;
    float sum = 0;
    float largest = -INFINITY;
    for(i = 0; i < n; ++i){
        int val = input[i];
        largest = (val>largest) ? val : largest;
    }
    for(i = 0; i < n; ++i){
        float e = exp(input[i]/temp - largest/temp);
        sum += e;
        output[i] = e;
    }
    for(i = 0; i < n; ++i){
        output[i] /= sum;
    }
}

__global__ void softmax_kernel(int n, int offset, int batch, float *input, float temp, float *output)
{
    int b = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(b >= batch) return;
    softmax_device(n, input + b*offset, temp, output + b*offset);
}

extern "C" void softmax_gpu(float *input, int n, int offset, int groups, float temp, float *output)
{
    int inputs = n;
    int batch = groups;
    softmax_kernel<<<cuda_gridsize(batch), BLOCK, 0, get_cuda_stream()>>>(inputs, offset, batch, input, temp, output);
    CHECK_CUDA(hipPeekAtLastError());
}

__device__ void softmax_device_new_api(float *input, int n, float temp, int stride, float *output)
{
	int i;
	float sum = 0;
	float largest = -INFINITY;
	for (i = 0; i < n; ++i) {
		int val = input[i*stride];
		largest = (val>largest) ? val : largest;
	}
	for (i = 0; i < n; ++i) {
		float e = expf(input[i*stride] / temp - largest / temp);
		sum += e;
		output[i*stride] = e;
	}
	for (i = 0; i < n; ++i) {
		output[i*stride] /= sum;
	}
}

__global__ void softmax_kernel_new_api(float *input, int n, int batch, int batch_offset, int groups, int group_offset, int stride, float temp, float *output)
{
	int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if (id >= batch*groups) return;
	int b = id / groups;
	int g = id % groups;
	softmax_device_new_api(input + b*batch_offset + g*group_offset, n, temp, stride, output + b*batch_offset + g*group_offset);
}

extern "C" void softmax_gpu_new_api(float *input, int n, int batch, int batch_offset, int groups, int group_offset, int stride, float temp, float *output)
{
	softmax_kernel_new_api << <cuda_gridsize(batch*groups), BLOCK, 0, get_cuda_stream() >> >(input, n, batch, batch_offset, groups, group_offset, stride, temp, output);
    CHECK_CUDA(hipPeekAtLastError());
}


__global__ void upsample_kernel(size_t N, float *x, int w, int h, int c, int batch, int stride, int forward, float scale, float *out)
{
    size_t i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= N) return;
    int out_index = i;
    int out_w = i % (w*stride);
    i = i / (w*stride);
    int out_h = i % (h*stride);
    i = i / (h*stride);
    int out_c = i%c;
    i = i / c;
    int b = i%batch;

    int in_w = out_w / stride;
    int in_h = out_h / stride;
    int in_c = out_c;

    int in_index = b*w*h*c + in_c*w*h + in_h*w + in_w;


    if (forward) out[out_index] += scale * x[in_index];
    else atomicAdd(x + in_index, scale * out[out_index]);
}

extern "C" void upsample_gpu(float *in, int w, int h, int c, int batch, int stride, int forward, float scale, float *out)
{
    size_t size = w*h*c*batch*stride*stride;
    upsample_kernel << <cuda_gridsize(size), BLOCK, 0, get_cuda_stream() >> >(size, in, w, h, c, batch, stride, forward, scale, out);
    CHECK_CUDA(hipPeekAtLastError());
}

__global__ void softmax_tree_kernel(float *input, int spatial, int batch, int stride, float temp, float *output, int groups, int *group_size, int *group_offset)
{
	int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if (id >= spatial*batch*groups) return;
	int s = id % spatial;
	id = id / spatial;
	int g = id % groups;
	int b = id / groups;
	int goff = group_offset[g] * spatial;
	int boff = b*stride;
	softmax_device_new_api(input + goff + boff + s, group_size[g], temp, spatial, output + goff + boff + s);
}

extern "C" void softmax_tree_gpu(float *input, int spatial, int batch, int stride, float temp, float *output, tree hier)
{
	int *tree_groups_size = cuda_make_int_array_new_api(hier.group_size, hier.groups);
	int *tree_groups_offset = cuda_make_int_array_new_api(hier.group_offset, hier.groups);
	/*
	static int *tree_groups_size = 0;
	static int *tree_groups_offset = 0;
	if(!tree_groups_size){
	tree_groups_size = cuda_make_int_array(hier.group_size, hier.groups);
	tree_groups_offset = cuda_make_int_array(hier.group_offset, hier.groups);
	}
	*/
	int num = spatial*batch*hier.groups;
	softmax_tree_kernel <<<cuda_gridsize(num), BLOCK, 0, get_cuda_stream() >>>(input, spatial, batch, stride, temp, output, hier.groups, tree_groups_size, tree_groups_offset);
    CHECK_CUDA(hipPeekAtLastError());
	cuda_free((float *)tree_groups_size);
	cuda_free((float *)tree_groups_offset);
}


__global__ void fix_nan_and_inf_kernel(float *input, size_t size)
{
    const int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < size) {
        float val = input[index];
        if (isnan(val) || isinf(val)) {
            input[index] = 1.0f / (fabs((float)index) + 1);  // pseudo random value
        }
    }
}

extern "C" void fix_nan_and_inf(float *input, size_t size)
{
    const int block_size = BLOCK;
    const int num_blocks = get_number_of_blocks(size, block_size);
    fix_nan_and_inf_kernel << <num_blocks, block_size, 0, get_cuda_stream() >> >(input, size);
    CHECK_CUDA(hipPeekAtLastError());
    //CHECK_CUDA(hipDeviceSynchronize());
}


__global__ void reset_nan_and_inf_kernel(float *input, size_t size)
{
    const int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < size) {
        float val = input[index];
        if (isnan(val) || isinf(val)) {
            input[index] = 0;
        }
    }
}

extern "C" void reset_nan_and_inf(float *input, size_t size)
{
    const int block_size = BLOCK;
    const int num_blocks = get_number_of_blocks(size, block_size);
    reset_nan_and_inf_kernel << <num_blocks, block_size, 0, get_cuda_stream() >> >(input, size);
    CHECK_CUDA(hipPeekAtLastError());
    //CHECK_CUDA(hipDeviceSynchronize());
}



__global__ void is_nan_or_inf_kernel(float *input, size_t size, int *pinned_return)
{
    const int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < size) {
        float val = input[index];
        if (isnan(val) || isinf(val))
            *pinned_return = 1;
    }
}

extern "C" int is_nan_or_inf(float *input, size_t size)
{
    int *pinned_return;
    CHECK_CUDA(hipHostAlloc(&pinned_return, sizeof(int), hipHostRegisterMapped));
    *pinned_return = 0;

    const int block_size = BLOCK;
    const int num_blocks = get_number_of_blocks(size, block_size);
    is_nan_or_inf_kernel << <num_blocks, block_size, 0, get_cuda_stream() >> >(input, size, pinned_return);
    CHECK_CUDA(hipDeviceSynchronize());
    int ret_val = *pinned_return;

    CHECK_CUDA(hipHostFree(pinned_return));
    return ret_val;
}

__global__ void add_3_arrays_activate_kernel(float *a1, float *a2, float *a3, size_t size, ACTIVATION a, float *dst)
{
    const int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < size) {
        float val = 0;
        if (a1) val += a1[index];
        if (a2) val += a2[index];
        if (a3) val += a3[index];
        if (a == LOGISTIC) val = 1.f / (1.f + expf(-val));
        else if (a == TANH) val = (2 / (1 + expf(-2 * val)) - 1);
        else if (a == LEAKY) val = (val < 0) ? val*0.1 : val;
        dst[index] = val;
    }
}

extern "C" void add_3_arrays_activate(float *a1, float *a2, float *a3, size_t size, ACTIVATION a, float *dst)
{
    const int block_size = BLOCK;
    const int num_blocks = get_number_of_blocks(size, block_size);
    if (!(a == LOGISTIC || a == TANH || a == LEAKY || a == LINEAR)) {
        printf(" add_3_arrays_activate() doesn't support activation %d, it supports only LOGISTIC and TANH \n", a);
        exit(EXIT_FAILURE);
    }
    add_3_arrays_activate_kernel << <num_blocks, block_size, 0, get_cuda_stream() >> >(a1, a2, a3, size, a, dst);
}


__global__ void sum_of_mults_kernel(float *a1, float *a2, float *b1, float *b2, size_t size, float *dst)
{
    const int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < size) {
        dst[index] = a1[index] * a2[index] + b1[index] * b2[index];
    }
}

extern "C" void sum_of_mults(float *a1, float *a2, float *b1, float *b2,  size_t size, float *dst)
{
    const int block_size = BLOCK;
    const int num_blocks = get_number_of_blocks(size, block_size);
    sum_of_mults_kernel << <num_blocks, block_size, 0, get_cuda_stream() >> >(a1, a2, b1, b2, size, dst);
}


__global__ void activate_and_mult_kernel(float *a1, float *a2, size_t size, ACTIVATION a, float *dst)
{
    const int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < size) {
        float val = a1[index];
        if (a == TANH) val = (2 / (1 + expf(-2 * val)) - 1);
        else if (a == LEAKY) val = (val < 0) ? val*0.1 : val;
        dst[index] = val * a2[index];
    }
}

extern "C" void activate_and_mult(float *a1, float *a2, size_t size, ACTIVATION a, float *dst)
{
    const int block_size = BLOCK;
    const int num_blocks = get_number_of_blocks(size, block_size);
    if (!(a == TANH || a == LEAKY || a == LINEAR)) {
        printf(" activat_and_mult() doesn't support activation %d, it supports only TANH \n", a);
        exit(EXIT_FAILURE);
    }
    activate_and_mult_kernel << <num_blocks, block_size, 0, get_cuda_stream() >> >(a1, a2, size, a, dst);
}



__global__ void scale_channels_kernel(float *in_w_h_c, int size, int channel_size, int batch_size, int scale_wh, float *scales_c, float *out)
{
    const int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < size) {
        if (scale_wh) {
            int osd_index = index % channel_size + (index / batch_size)*channel_size;

            out[index] = in_w_h_c[index] * scales_c[osd_index];
        }
        else {
            out[index] = in_w_h_c[index] * scales_c[index / channel_size];
        }
    }
}

extern "C" void scale_channels_gpu(float *in_w_h_c, int size, int channel_size, int batch_size, int scale_wh, float *scales_c, float *out)
{
    const int block_size = BLOCK;
    const int num_blocks = get_number_of_blocks(size, block_size);
    scale_channels_kernel << <num_blocks, block_size, 0, get_cuda_stream() >> >(in_w_h_c, size, channel_size, batch_size, scale_wh, scales_c, out);
    CHECK_CUDA(hipPeekAtLastError());
}




__global__ void backward_scale_channels_kernel(float *in_w_h_c_delta, int size, int channel_size, int batch_size, int scale_wh,
    float *in_scales_c, float *out_from_delta,
    float *in_from_output, float *out_state_delta)
{
    const int index = blockIdx.x*blockDim.x + threadIdx.x;

    if (index < size) {

        if (scale_wh)
        {
            int osd_index = index % channel_size + (index / batch_size)*channel_size;

            //out_state_delta[osd_index] += in_w_h_c_delta[index] * in_from_output[index]; // l.delta * from  (should be divided by channel_size?)
            atomicAdd(&out_state_delta[osd_index], in_w_h_c_delta[index] * in_from_output[index] / channel_size); // l.delta * from

            out_from_delta[index] += in_scales_c[osd_index] * in_w_h_c_delta[index]; // input * l.delta  // atomic isn't required here

        }
        else {
            int osd_index = index / channel_size;
            //out_state_delta[osd_index] += in_w_h_c_delta[index] * in_from_output[index]; // l.delta * from  (should be divided by channel_size?)

            int warp_id = index / 32;
            int index_warp_start = warp_id * 32;
            int osd_index_warp_start = index_warp_start / channel_size;
            int osd_index_warp_end = (index_warp_start + 31) / channel_size;

            if (osd_index_warp_start == osd_index_warp_end) // all thread in warp process the same channel
            {
                float sum = warpAllReduceSum(in_w_h_c_delta[index] * in_from_output[index]); // l.delta * from
                if (threadIdx.x % 32 == 0) {
                    atomicAdd(&out_state_delta[osd_index], sum);
                    //out_state_delta[osd_index] += sum;
                }
            }
            else {
                atomicAdd(&out_state_delta[osd_index], in_w_h_c_delta[index] * in_from_output[index]); // l.delta * from
            }

            out_from_delta[index] += in_scales_c[osd_index] * in_w_h_c_delta[index]; // input * l.delta  // atomic isn't required here
        }
    }
}

extern "C" void backward_scale_channels_gpu(float *in_w_h_c_delta, int size, int channel_size, int batch_size, int scale_wh,
    float *in_scales_c, float *out_from_delta,
    float *in_from_output, float *out_state_delta)
{
    const int block_size = BLOCK;
    const int num_blocks = get_number_of_blocks(size, block_size);
    backward_scale_channels_kernel << <num_blocks, block_size, 0, get_cuda_stream() >> > (in_w_h_c_delta, size, channel_size, batch_size, scale_wh,
        in_scales_c, out_from_delta,
        in_from_output, out_state_delta);

    CHECK_CUDA(hipPeekAtLastError());
}


__global__ void sam_kernel(float *in_w_h_c, int size, int channel_size, float *scales_c, float *out)
{
    const int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < size) {
        out[index] = in_w_h_c[index] * scales_c[index];
    }
}

extern "C" void sam_gpu(float *in_w_h_c, int size, int channel_size, float *scales_c, float *out)
{
    const int block_size = BLOCK;
    const int num_blocks = get_number_of_blocks(size, block_size);
    sam_kernel << <num_blocks, block_size, 0, get_cuda_stream() >> >(in_w_h_c, size, channel_size, scales_c, out);
    CHECK_CUDA(hipPeekAtLastError());
}


__global__ void backward_sam_kernel(float *in_w_h_c_delta, int size, int channel_size,
    float *in_scales_c, float *out_from_delta,
    float *in_from_output, float *out_state_delta)
{
    const int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < size) {
        out_state_delta[index] += in_w_h_c_delta[index] * in_from_output[index]; // l.delta * from  (should be divided by channel_size?)
        out_from_delta[index] += in_scales_c[index] * in_w_h_c_delta[index]; // input * l.delta

                                                                             //out_state_delta[index] += in_w_h_c_delta[index];
                                                                             //out_from_delta[index] = in_w_h_c_delta[index];
    }
}

extern "C" void backward_sam_gpu(float *in_w_h_c_delta, int size, int channel_size,
    float *in_scales_c, float *out_from_delta,
    float *in_from_output, float *out_state_delta)
{
    const int block_size = BLOCK;
    const int num_blocks = get_number_of_blocks(size, block_size);
    backward_sam_kernel << <num_blocks, block_size, 0, get_cuda_stream() >> > (in_w_h_c_delta, size, channel_size,
        in_scales_c, out_from_delta,
        in_from_output, out_state_delta);

    CHECK_CUDA(hipPeekAtLastError());
}


__global__  void smooth_rotate_weights_kernel(const float *src_weight_gpu, float *weight_deform_gpu, int nweights, int n, int kernel_size, int angle, int reverse)
{
    const int index = blockIdx.x*blockDim.x + threadIdx.x;
    const int kernel_area = kernel_size * kernel_size;
    const int i = index * kernel_area;

    const int stage_step = (nweights / kernel_area) / 4;  // 4 stages
    const int stage_id = index / stage_step;

    // nweights = (c / groups) * n * size * size;
    // kernel_area = size*size

    if (i < nweights)
    {
        // rotate left or right
        if (reverse) angle = -angle;

        const float cos_a = cosf(angle * 3.14159265 / 180);
        const float sin_a = sinf(angle * 3.14159265 / 180);
        const int x_c = kernel_size / 2;
        const int y_c = kernel_size / 2;

        float dropout_sum = 0;

        for (int y = 0; y < kernel_size; ++y) {
            for (int x = 0; x < kernel_size; ++x) {
                // Xsource = x*cos(alpha) + y*sin(alpha)
                // Ysource = -x*sin(alpha) + y*cos(alpha)

                float x_s = x_c + (x - x_c)*cos_a + (y - y_c)*sin_a;
                float y_s = y_c - (x - x_c)*sin_a + (y - y_c)*cos_a;

                int x_0 = floorf(x_s);   // round down
                int x_1 = ceilf(x_s);    // round up
                if (x_0 == x_1) x_1 = x_0 + 1;
                int y_0 = floorf(y_s);
                int y_1 = ceilf(y_s);
                if (y_0 == y_1) y_1 = y_0 + 1;

                float c_x_0 = x_1 - x_s;
                float c_x_1 = x_s - x_0;
                float c_y_0 = y_1 - y_s;
                float c_y_1 = y_s - y_0;


                float val = 0;
                if (x_0 >= 0 && x_0 < kernel_size && y_0 >= 0 && y_0 < kernel_size) val += src_weight_gpu[x_0 + y_0*kernel_size + i] * c_x_0 * c_y_0;
                else dropout_sum += c_x_0 * c_y_0;

                if (x_1 >= 0 && x_1 < kernel_size && y_0 >= 0 && y_0 < kernel_size) val += src_weight_gpu[x_1 + y_0*kernel_size + i] * c_x_1 * c_y_0;
                else dropout_sum += c_x_1 * c_y_0;

                if (x_0 >= 0 && x_0 < kernel_size && y_1 >= 0 && y_1 < kernel_size) val += src_weight_gpu[x_0 + y_1*kernel_size + i] * c_x_0 * c_y_1;
                else dropout_sum += c_x_0 * c_y_1;

                if (x_1 >= 0 && x_1 < kernel_size && y_1 >= 0 && y_1 < kernel_size) val += src_weight_gpu[x_1 + y_1*kernel_size + i] * c_x_1 * c_y_1;
                else dropout_sum += c_x_1 * c_y_1;

                weight_deform_gpu[x + y*kernel_size + i] = val;
            }
        }

        // compensate for dropped items
        const float coef = (kernel_size*kernel_size) / (kernel_size*kernel_size - dropout_sum);
        for (int y = 0; y < kernel_size; ++y) {
            for (int x = 0; x < kernel_size; ++x) {
                weight_deform_gpu[x + y*kernel_size + i] *= coef;
            }
        }
    }
}


extern "C" void smooth_rotate_weights_gpu(const float *src_weight_gpu, float *weight_deform_gpu, int nweights, int n, int size, int angle, int reverse)
{
    const int kernel_area = size*size;
    const int block_size = BLOCK;
    const int num_blocks = get_number_of_blocks(nweights / kernel_area, block_size);
    smooth_rotate_weights_kernel << <num_blocks, block_size, 0, get_cuda_stream() >> > (src_weight_gpu, weight_deform_gpu, nweights, n, size, angle, reverse);

    CHECK_CUDA(hipPeekAtLastError());
}



__global__  void stretch_weights_kernel(const float *src_weight_gpu, float *weight_deform_gpu, int nweights, int n, int kernel_size, float scale, int reverse)
{
    const int index = blockIdx.x*blockDim.x + threadIdx.x;
    const int kernel_area = kernel_size * kernel_size;
    const int i = index * kernel_area;

    const int stage_step = (nweights / kernel_area) / 4;  // 4 stages
    const int stage_id = index / stage_step;

    // nweights = (c / groups) * n * size * size;
    // kernel_area = size*size

    if (i < nweights)
    {

        if (stage_id == 0) {
            // simple copy
            for (int x = 0; x < kernel_size; ++x) {
                for (int y = 0; y < kernel_size; ++y) {
                    weight_deform_gpu[x + y*kernel_size + i] = src_weight_gpu[x + y*kernel_size + i];
                }
            }
        }
        else if (stage_id > 0)
        {
            if (stage_id == 1) scale = 0.65;
            else if (stage_id == 2) scale = 0.8;
            else if (stage_id == 3) scale = 1.3;

            if (reverse) scale = 1 / scale;

            const int x_c = kernel_size / 2;
            const int y_c = kernel_size / 2;

            float dropout_sum = 0;

            for (int y = 0; y < kernel_size; ++y) {
                for (int x = 0; x < kernel_size; ++x) {
                    // Xsource = x_c + (x_d - x_c) / scale
                    // Ysource = y_c + (y_d - y_c) / scale

                    float x_s = x_c + (x - x_c) / scale;
                    float y_s = y_c + (y - y_c) / scale;

                    int x_0 = floorf(x_s);   // round down
                    int x_1 = ceilf(x_s);    // round up
                    if (x_0 == x_1) x_1 = x_0 + 1;
                    int y_0 = floorf(y_s);
                    int y_1 = ceilf(y_s);
                    if (y_0 == y_1) y_1 = y_0 + 1;

                    float c_x_0 = x_1 - x_s;
                    float c_x_1 = x_s - x_0;
                    float c_y_0 = y_1 - y_s;
                    float c_y_1 = y_s - y_0;

                    float val = 0;
                    if (x_0 >= 0 && x_0 < kernel_size && y_0 >= 0 && y_0 < kernel_size) val += src_weight_gpu[x_0 + y_0*kernel_size + i] * c_x_0 * c_y_0;
                    else dropout_sum += c_x_0 * c_y_0;

                    if (x_1 >= 0 && x_1 < kernel_size && y_0 >= 0 && y_0 < kernel_size) val += src_weight_gpu[x_1 + y_0*kernel_size + i] * c_x_1 * c_y_0;
                    else dropout_sum += c_x_1 * c_y_0;

                    if (x_0 >= 0 && x_0 < kernel_size && y_1 >= 0 && y_1 < kernel_size) val += src_weight_gpu[x_0 + y_1*kernel_size + i] * c_x_0 * c_y_1;
                    else dropout_sum += c_x_0 * c_y_1;

                    if (x_1 >= 0 && x_1 < kernel_size && y_1 >= 0 && y_1 < kernel_size) val += src_weight_gpu[x_1 + y_1*kernel_size + i] * c_x_1 * c_y_1;
                    else dropout_sum += c_x_1 * c_y_1;

                    weight_deform_gpu[x + y*kernel_size + i] = val;
                }
            }

            // compensate for dropped items
            //const float coef = (kernel_size*kernel_size) / (kernel_size*kernel_size - dropout_sum);
            for (int y = 0; y < kernel_size; ++y) {
                for (int x = 0; x < kernel_size; ++x) {
                    //if (scale < 1) weight_deform_gpu[x + y*kernel_size + i] /= scale;// *= coef;
                    weight_deform_gpu[x + y*kernel_size + i] /= scale;// *= coef;
                }
            }
        }
    }
}


extern "C" void stretch_weights_gpu(const float *src_weight_gpu, float *weight_deform_gpu, int nweights, int n, int size, float scale, int reverse)
{
    const int kernel_area = size*size;
    const int block_size = BLOCK;
    const int num_blocks = get_number_of_blocks(nweights / kernel_area, block_size);
    stretch_weights_kernel << <num_blocks, block_size, 0, get_cuda_stream() >> > (src_weight_gpu, weight_deform_gpu, nweights, n, size, scale, reverse);

    CHECK_CUDA(hipPeekAtLastError());
}



__global__  void sway_and_flip_weights_kernel(const float *src_weight_gpu, float *weight_deform_gpu, int nweights, int n, int kernel_size, int angle, int reverse)
{
    const int index = blockIdx.x*blockDim.x + threadIdx.x;
    const int kernel_area = kernel_size * kernel_size;
    const int i = index * kernel_area;

    const int stage_step = (nweights / kernel_area) / 4;  // 4 stages
    const int stage_id = index / stage_step;

    // nweights = (c / groups) * n * size * size;
    // kernel_area = size*size

    if (i < nweights)
    {

        if (stage_id == 0) {
            // simple copy
            for (int x = 0; x < kernel_size; ++x) {
                for (int y = 0; y < kernel_size; ++y) {
                    weight_deform_gpu[x + y*kernel_size + i] = src_weight_gpu[x + y*kernel_size + i];
                }
            }
        }
        else if (stage_id == 1 || stage_id == 2)
        {
            // rotate left or right
            if (stage_id == 2) angle = -angle;
            if (reverse) angle = -angle;

            const float cos_a = cosf(angle * 3.14159265 / 180);
            const float sin_a = sinf(angle * 3.14159265 / 180);
            const int x_c = kernel_size / 2;
            const int y_c = kernel_size / 2;

            float dropout_sum = 0;

            for (int y = 0; y < kernel_size; ++y) {
                for (int x = 0; x < kernel_size; ++x) {
                    // Xsource = x*cos(alpha) + y*sin(alpha)
                    // Ysource = -x*sin(alpha) + y*cos(alpha)

                    float x_s = x_c + (x - x_c)*cos_a + (y - y_c)*sin_a;
                    float y_s = y_c - (x - x_c)*sin_a + (y - y_c)*cos_a;

                    int x_0 = floorf(x_s);   // round down
                    int x_1 = ceilf(x_s);    // round up
                    if (x_0 == x_1) x_1 = x_0 + 1;
                    int y_0 = floorf(y_s);
                    int y_1 = ceilf(y_s);
                    if (y_0 == y_1) y_1 = y_0 + 1;

                    float c_x_0 = x_1 - x_s;
                    float c_x_1 = x_s - x_0;
                    float c_y_0 = y_1 - y_s;
                    float c_y_1 = y_s - y_0;

                    float val = 0;
                    if (x_0 >= 0 && x_0 < kernel_size && y_0 >= 0 && y_0 < kernel_size) val += src_weight_gpu[x_0 + y_0*kernel_size + i] * c_x_0 * c_y_0;
                    else dropout_sum += c_x_0 * c_y_0;

                    if (x_1 >= 0 && x_1 < kernel_size && y_0 >= 0 && y_0 < kernel_size) val += src_weight_gpu[x_1 + y_0*kernel_size + i] * c_x_1 * c_y_0;
                    else dropout_sum += c_x_1 * c_y_0;

                    if (x_0 >= 0 && x_0 < kernel_size && y_1 >= 0 && y_1 < kernel_size) val += src_weight_gpu[x_0 + y_1*kernel_size + i] * c_x_0 * c_y_1;
                    else dropout_sum += c_x_0 * c_y_1;

                    if (x_1 >= 0 && x_1 < kernel_size && y_1 >= 0 && y_1 < kernel_size) val += src_weight_gpu[x_1 + y_1*kernel_size + i] * c_x_1 * c_y_1;
                    else dropout_sum += c_x_1 * c_y_1;

                    weight_deform_gpu[x + y*kernel_size + i] = val;
                }
            }

            // compensate for dropped items
            const float coef = (kernel_size*kernel_size) / (kernel_size*kernel_size - dropout_sum);
            for (int y = 0; y < kernel_size; ++y) {
                for (int x = 0; x < kernel_size; ++x) {
                    weight_deform_gpu[x + y*kernel_size + i] *= coef;
                }
            }
        }
        else if (stage_id == 3)
        {
            // flip
            for (int y = 0; y < kernel_size; ++y) {
                for (int x = 0; x < kernel_size; ++x) {
                    weight_deform_gpu[(kernel_size - x - 1) + y*kernel_size + i] = src_weight_gpu[x + y*kernel_size + i];
                }
            }
        }
    }
}


extern "C" void sway_and_flip_weights_gpu(const float *src_weight_gpu, float *weight_deform_gpu, int nweights, int n, int size, int angle, int reverse)
{
    const int kernel_area = size*size;
    const int block_size = BLOCK;
    const int num_blocks = get_number_of_blocks(nweights / kernel_area, block_size);
    sway_and_flip_weights_kernel << <num_blocks, block_size, 0, get_cuda_stream() >> > (src_weight_gpu, weight_deform_gpu, nweights, n, size, angle, reverse);

    CHECK_CUDA(hipPeekAtLastError());
}







__global__  void rotate_weights_kernel(const float *src_weight_gpu, float *weight_deform_gpu, int nweights, int n, int kernel_size, int reverse)
{
    const int index = blockIdx.x*blockDim.x + threadIdx.x;
    const int kernel_area = kernel_size * kernel_size;
    const int i = index * kernel_area;

    const int stage_step = (nweights / kernel_area) / 4;  // 4 stages
    const int stage_id = index / stage_step;

    // nweights = (c / groups) * n * size * size;
    // kernel_area = size*size

    if (i < nweights)
    {
        // if(reverse)

        if (stage_id == 0) {
            // simple copy
            for (int y = 0; y < kernel_size; ++y) {
                for (int x = 0; x < kernel_size; ++x) {
                    const int src_i = x + y*kernel_size + i;
                    const int dst_i = x + y*kernel_size + i;
                    if (reverse) weight_deform_gpu[src_i] = src_weight_gpu[dst_i];
                    else weight_deform_gpu[dst_i] = src_weight_gpu[src_i];
                }
            }
        }
        else if (stage_id == 1)
        {
            // 90 degree clockwise rotation - 1
            for (int y = 0; y < kernel_size; ++y) {
                for (int x = 0; x < kernel_size; ++x) {
                    const int src_i = x + y*kernel_size + i;
                    const int dst_i = (kernel_size - 1 - y) + x*kernel_size + i;
                    if (reverse) weight_deform_gpu[src_i] = src_weight_gpu[dst_i];
                    else weight_deform_gpu[dst_i] = src_weight_gpu[src_i];
                }
            }
        }
        else if (stage_id == 2)
        {
            // 180 degree clockwise rotation - 2
            for (int y = 0; y < kernel_size; ++y) {
                for (int x = 0; x < kernel_size; ++x) {
                    const int src_i = x + y*kernel_size + i;
                    const int dst_i = (kernel_size - 1 - x) + (kernel_size - 1 - y)*kernel_size + i;
                    if (reverse) weight_deform_gpu[src_i] = src_weight_gpu[dst_i];
                    else weight_deform_gpu[dst_i] = src_weight_gpu[src_i];
                }
            }
        }
        else if (stage_id == 3)
        {
            // 270 degree clockwise rotation - 3
            for (int y = 0; y < kernel_size; ++y) {
                for (int x = 0; x < kernel_size; ++x) {
                    const int src_i = x + y*kernel_size + i;
                    const int dst_i = y + (kernel_size - 1 - x)*kernel_size + i;
                    if (reverse) weight_deform_gpu[src_i] = src_weight_gpu[dst_i];
                    else weight_deform_gpu[dst_i] = src_weight_gpu[src_i];
                }
            }
        }
    }
}


extern "C" void rotate_weights_gpu(const float *src_weight_gpu, float *weight_deform_gpu, int nweights, int n, int size, int reverse)
{
    const int kernel_area = size*size;
    const int block_size = BLOCK;
    const int num_blocks = get_number_of_blocks(nweights / kernel_area, block_size);
    rotate_weights_kernel << <num_blocks, block_size, 0, get_cuda_stream() >> > (src_weight_gpu, weight_deform_gpu, nweights, n, size, reverse);

    CHECK_CUDA(hipPeekAtLastError());
}



__global__  void stretch_sway_flip_weights_kernel(const float *src_weight_gpu, float *weight_deform_gpu, int nweights, int n, int kernel_size, float angle, int reverse)
{
    const int index = blockIdx.x*blockDim.x + threadIdx.x;
    const int kernel_area = kernel_size * kernel_size;
    const int i = index * kernel_area;

    const int stage_step = (nweights / kernel_area) / 8;  // 8 stages
    const int stage_id = index / stage_step;

    // nweights = (c / groups) * n * size * size;
    // kernel_area = size*size

    if (i < nweights)
    {

        if (stage_id == 0) {
            // simple copy
            for (int x = 0; x < kernel_size; ++x) {
                for (int y = 0; y < kernel_size; ++y) {
                    weight_deform_gpu[x + y*kernel_size + i] = src_weight_gpu[x + y*kernel_size + i];
                }
            }
        }
        else if (stage_id == 1 || stage_id == 2 || stage_id == 3 || stage_id == 4)
        {
            float scale = 0.5;
            if (stage_id == 1) scale = 0.65;
            else if (stage_id == 2) scale = 0.8;
            else if (stage_id == 3) scale = 1.2;
            else if (stage_id == 4) scale = 1.4;

            if (reverse) scale = 1 / scale;

            const int x_c = kernel_size / 2;
            const int y_c = kernel_size / 2;

            float dropout_sum = 0;

            for (int y = 0; y < kernel_size; ++y) {
                for (int x = 0; x < kernel_size; ++x) {
                    // Xsource = x_c + (x_d - x_c) / scale
                    // Ysource = y_c + (y_d - y_c) / scale

                    float x_s = x_c + (x - x_c) / scale;
                    float y_s = y_c + (y - y_c) / scale;

                    int x_0 = floorf(x_s);   // round down
                    int x_1 = ceilf(x_s);    // round up
                    if (x_0 == x_1) x_1 = x_0 + 1;
                    int y_0 = floorf(y_s);
                    int y_1 = ceilf(y_s);
                    if (y_0 == y_1) y_1 = y_0 + 1;

                    float c_x_0 = x_1 - x_s;
                    float c_x_1 = x_s - x_0;
                    float c_y_0 = y_1 - y_s;
                    float c_y_1 = y_s - y_0;

                    float val = 0;
                    if (x_0 >= 0 && x_0 < kernel_size && y_0 >= 0 && y_0 < kernel_size) val += src_weight_gpu[x_0 + y_0*kernel_size + i] * c_x_0 * c_y_0;
                    else dropout_sum += c_x_0 * c_y_0;

                    if (x_1 >= 0 && x_1 < kernel_size && y_0 >= 0 && y_0 < kernel_size) val += src_weight_gpu[x_1 + y_0*kernel_size + i] * c_x_1 * c_y_0;
                    else dropout_sum += c_x_1 * c_y_0;

                    if (x_0 >= 0 && x_0 < kernel_size && y_1 >= 0 && y_1 < kernel_size) val += src_weight_gpu[x_0 + y_1*kernel_size + i] * c_x_0 * c_y_1;
                    else dropout_sum += c_x_0 * c_y_1;

                    if (x_1 >= 0 && x_1 < kernel_size && y_1 >= 0 && y_1 < kernel_size) val += src_weight_gpu[x_1 + y_1*kernel_size + i] * c_x_1 * c_y_1;
                    else dropout_sum += c_x_1 * c_y_1;

                    weight_deform_gpu[x + y*kernel_size + i] = val;
                }
            }

            // compensate for dropped items
            //const float coef = (kernel_size*kernel_size) / (kernel_size*kernel_size - dropout_sum);
            for (int y = 0; y < kernel_size; ++y) {
                for (int x = 0; x < kernel_size; ++x) {
                    if(scale > 1)
                        weight_deform_gpu[x + y*kernel_size + i] /= scale;// *= coef;
                }
            }
        }
        else if (stage_id == 5 || stage_id == 6)
        {
            // rotate left or right
            if (stage_id == 6) angle = -angle;
            if (reverse) angle = -angle;

            const float cos_a = cosf(angle * 3.14159265 / 180);
            const float sin_a = sinf(angle * 3.14159265 / 180);
            const int x_c = kernel_size / 2;
            const int y_c = kernel_size / 2;

            float dropout_sum = 0;

            for (int y = 0; y < kernel_size; ++y) {
                for (int x = 0; x < kernel_size; ++x) {
                    // Xsource = x*cos(alpha) + y*sin(alpha)
                    // Ysource = -x*sin(alpha) + y*cos(alpha)

                    float x_s = x_c + (x - x_c)*cos_a + (y - y_c)*sin_a;
                    float y_s = y_c - (x - x_c)*sin_a + (y - y_c)*cos_a;

                    int x_0 = floorf(x_s);   // round down
                    int x_1 = ceilf(x_s);    // round up
                    if (x_0 == x_1) x_1 = x_0 + 1;
                    int y_0 = floorf(y_s);
                    int y_1 = ceilf(y_s);
                    if (y_0 == y_1) y_1 = y_0 + 1;

                    float c_x_0 = x_1 - x_s;
                    float c_x_1 = x_s - x_0;
                    float c_y_0 = y_1 - y_s;
                    float c_y_1 = y_s - y_0;

                    float val = 0;
                    if (x_0 >= 0 && x_0 < kernel_size && y_0 >= 0 && y_0 < kernel_size) val += src_weight_gpu[x_0 + y_0*kernel_size + i] * c_x_0 * c_y_0;
                    else dropout_sum += c_x_0 * c_y_0;

                    if (x_1 >= 0 && x_1 < kernel_size && y_0 >= 0 && y_0 < kernel_size) val += src_weight_gpu[x_1 + y_0*kernel_size + i] * c_x_1 * c_y_0;
                    else dropout_sum += c_x_1 * c_y_0;

                    if (x_0 >= 0 && x_0 < kernel_size && y_1 >= 0 && y_1 < kernel_size) val += src_weight_gpu[x_0 + y_1*kernel_size + i] * c_x_0 * c_y_1;
                    else dropout_sum += c_x_0 * c_y_1;

                    if (x_1 >= 0 && x_1 < kernel_size && y_1 >= 0 && y_1 < kernel_size) val += src_weight_gpu[x_1 + y_1*kernel_size + i] * c_x_1 * c_y_1;
                    else dropout_sum += c_x_1 * c_y_1;

                    weight_deform_gpu[x + y*kernel_size + i] = val;
                }
            }

            // compensate for dropped items
            const float coef = (kernel_size*kernel_size) / (kernel_size*kernel_size - dropout_sum);
            for (int y = 0; y < kernel_size; ++y) {
                for (int x = 0; x < kernel_size; ++x) {
                    weight_deform_gpu[x + y*kernel_size + i] *= coef;
                }
            }
        }
        else if (stage_id == 7)
        {
            // flip
            for (int y = 0; y < kernel_size; ++y) {
                for (int x = 0; x < kernel_size; ++x) {
                    weight_deform_gpu[(kernel_size - x - 1) + y*kernel_size + i] = src_weight_gpu[x + y*kernel_size + i];
                }
            }
        }
    }
}


extern "C" void stretch_sway_flip_weights_gpu(const float *src_weight_gpu, float *weight_deform_gpu, int nweights, int n, int size, int angle, int reverse)
{
    const int kernel_area = size*size;
    const int block_size = BLOCK;
    const int num_blocks = get_number_of_blocks(nweights / kernel_area, block_size);
    stretch_sway_flip_weights_kernel << <num_blocks, block_size, 0, get_cuda_stream() >> > (src_weight_gpu, weight_deform_gpu, nweights, n, size, angle, reverse);

    CHECK_CUDA(hipPeekAtLastError());
}



__global__  void reduce_and_expand_array_kernel(const float *src_gpu, float *dst_gpu, int current_size, int groups)
{
    const int index = blockIdx.x*blockDim.x + threadIdx.x;

    if (index < current_size) {
        float val = 0;
        for (int i = 0; i < groups; ++i) {
            val += src_gpu[index + i*current_size];
        }
        for (int i = 0; i < groups; ++i) {
            dst_gpu[index + i*current_size] = val / groups;
        }
    }
}

extern "C" void reduce_and_expand_array_gpu(const float *src_gpu, float *dst_gpu, int size, int groups)
{
    const int current_size = size / groups;
    const int block_size = BLOCK;
    const int num_blocks = get_number_of_blocks(current_size, block_size);
    reduce_and_expand_array_kernel << <num_blocks, block_size, 0, get_cuda_stream() >> > (src_gpu, dst_gpu, current_size, groups);

    CHECK_CUDA(hipPeekAtLastError());
}



__global__  void expand_array_kernel(const float *src_gpu, float *dst_gpu, int current_size, int groups)
{
    const int index = blockIdx.x*blockDim.x + threadIdx.x;

    if (index < current_size) {
        for (int i = 0; i < groups; ++i) {
            dst_gpu[index + i*current_size] = src_gpu[index];
        }
    }
}

extern "C" void expand_array_gpu(const float *src_gpu, float *dst_gpu, int size, int groups)
{
    const int current_size = size / groups;
    const int block_size = BLOCK;
    const int num_blocks = get_number_of_blocks(current_size, block_size);
    expand_array_kernel << <num_blocks, block_size, 0, get_cuda_stream() >> > (src_gpu, dst_gpu, current_size, groups);

    CHECK_CUDA(hipPeekAtLastError());
}



__global__  void mult_inverse_array_kernel(const float *src_gpu, float *dst_gpu, int size, const float eps,
    float divider, const float clip, const float abs_add)
{
    const int index = blockIdx.x*blockDim.x + threadIdx.x;

    if (index < size) {
        float val = src_gpu[index];
        float sign = (val < 0) ? -1 : 1;
        // eps = 1 by default
        // eps = 2 - lower delta
        // eps = 0 - higher delta (linear)
        // eps = -1 - high delta (inverse number)
        // = (abs(x)*10+1)^(-1)
        float unsigned_val = powf(fabs(val)*10 + abs_add, eps);
        unsigned_val = unsigned_val / divider;
        if (unsigned_val > clip && clip != 0.0) unsigned_val = clip;
        if (isnan(unsigned_val) || isinf(unsigned_val)) unsigned_val = 0;
        dst_gpu[index] = unsigned_val * sign;
    }
}

extern "C" void mult_inverse_array_gpu(const float *src_gpu, float *dst_gpu, int size, float eps, float divider, float clip, float abs_add)
{
    const int block_size = BLOCK;
    const int num_blocks = get_number_of_blocks(size, block_size);
    mult_inverse_array_kernel << <num_blocks, block_size, 0, get_cuda_stream() >> > (src_gpu, dst_gpu, size, eps, divider, clip, abs_add);

    CHECK_CUDA(hipPeekAtLastError());
}



__global__ void P_constrastive_f_det_kernel(int *labels, unsigned int feature_size, float temperature, contrastive_params *contrast_p, const int contrast_p_size)
{
    const int il = blockIdx.x*blockDim.x + threadIdx.x;

    if (il < contrast_p_size) {
        const float sim = contrast_p[il].sim;
        const size_t i = contrast_p[il].i;
        const size_t j = contrast_p[il].j;

        const float numerator = expf(sim / temperature);

        float denominator = 0;
        int k;
        for (k = 0; k < contrast_p_size; ++k) {
            contrastive_params cp = contrast_p[k];
            //if (k != i && labels[k] != labels[i]) {
            //if (k != i) {
            if (cp.i != i && cp.j == j) {
                //const float sim_den = cp.sim;
                ////const float sim_den = find_sim(k, l, contrast_p, contrast_p_size); // cosine_similarity(z[k], z[l], feature_size);
                //denominator += expf(sim_den / temperature);
                denominator += cp.exp_sim;
            }
        }

        float result = 0.9999;
        if (denominator != 0) result = numerator / denominator;
        if (result > 1) result = 0.9999;

        contrast_p[il].P = result;
    }
}


extern "C" void P_constrastive_f_det_gpu(int *labels, unsigned int feature_size, float temperature, contrastive_params *contrast_p, const int contrast_p_size)
{
    const int block_size = BLOCK;
    const int num_blocks = get_number_of_blocks(contrast_p_size, block_size);
    P_constrastive_f_det_kernel << <num_blocks, block_size, 0, get_cuda_stream() >> > (labels, feature_size, temperature, contrast_p, contrast_p_size);

    CHECK_CUDA(hipPeekAtLastError());
}




__global__ void coord_conv_kernel(float *dst, int w, int h, int chan, int batch, int type)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;

    const int x = i % w;
    i = i / w;
    const int y = i % h;
    i = i / h;
    const int c = i % chan;
    //i = i / chan;
    //const int b = i % batch;

    if (type == 0) {
        if (c == 0) {
            const float x_val = (2.0f * x) / w - 1.0f;  // [-1; 1)
            dst[i] = x_val; // x - coord
        }
        else if (c == 1) {
            const float y_val = (2.0f * y) / h - 1.0f;  // [-1; 1)
            dst[i] = y_val; // y - coord
        }
        else if (c == 2) {
            const float x_val = (2.0f * x) / w - 1.0f;  // [-1; 1)
            const float y_val = (2.0f * y) / h - 1.0f;  // [-1; 1)
            const float rad_val = sqrtf(x_val*x_val + y_val*y_val);  // [0; 1.414)
            dst[i] = rad_val; // rad - coord
        }
    }
    else if (type == 1) {
        if (c >= 0 && c <= 2) {
            dst[i] = 0;
        }
    }
}

extern "C" void coord_conv_gpu(float *dst, int size, int w, int h, int chan, int b, int type)
{
    const int block_size = BLOCK;
    const int num_blocks = get_number_of_blocks(size, block_size);
    coord_conv_kernel << <num_blocks, block_size, 0, get_cuda_stream() >> > (dst, w, h, chan, b, type);

    CHECK_CUDA(hipPeekAtLastError());
}


__global__ void forward_implicit_kernel(int size, int batch, int nweights, float *weight_gpu, float *output_gpu)
{
    const int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= size) return;

    output_gpu[id] = weight_gpu[id % nweights];
}

extern "C" void forward_implicit_gpu(int batch, int nweights, float *weight_gpu, float *output_gpu)
{
    int size = batch * nweights;
    forward_implicit_kernel << <cuda_gridsize(size), BLOCK, 0, get_cuda_stream() >> > (size, batch, nweights, weight_gpu, output_gpu);
    CHECK_CUDA(hipPeekAtLastError());
}



__global__ void backward_implicit_kernel(int size, int batch, int nweights, float *weight_updates_gpu, float *delta_gpu)
{
    const int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= size) return;

    for (int i = 0; i < batch; ++i) {
        weight_updates_gpu[id] += delta_gpu[id + i * nweights];
    }
}

extern "C" void backward_implicit_gpu(int batch, int nweights, float *weight_updates_gpu, float *delta_gpu)
{
    int size = nweights;
    backward_implicit_kernel << <cuda_gridsize(size), BLOCK, 0, get_cuda_stream() >> > (size, batch, nweights, weight_updates_gpu, delta_gpu);
    CHECK_CUDA(hipPeekAtLastError());
}
