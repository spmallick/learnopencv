#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipblas.h>
#include <stdint.h>

#include "im2col.h"
#include "dark_cuda.h"

#include <stdio.h>
#include <assert.h>


template<typename T1, typename T2>
__device__ inline T1 __shfl_custom(T1 val, T2 lane) {
#if CUDART_VERSION >= 9000
    return __shfl_sync(FULL_MASK, val, lane);
#else
    return __shfl(val, lane);
#endif
}

template<typename T>
__device__ inline uint32_t __ballot_custom(T val) {
#if CUDART_VERSION >= 9000
    return __ballot_sync(FULL_MASK, val);
#else
    return __ballot(val);
#endif
}


// src: https://github.com/BVLC/caffe/blob/master/src/caffe/util/im2col.cu
// You may also want to read: https://github.com/BVLC/caffe/blob/master/LICENSE

__global__ void im2col_gpu_kernel(const int n, const float* data_im,
        const int height, const int width, const int ksize,
        const int pad,
        const int stride,
        const int height_col, const int width_col,
        float *data_col) {
    int index = blockIdx.x*blockDim.x+threadIdx.x;
    for(; index < n; index += blockDim.x*gridDim.x){
        int w_out = index % width_col;
        int h_index = index / width_col;
        int h_out = h_index % height_col;
        int channel_in = h_index / height_col;
        int channel_out = channel_in * ksize * ksize;
        int h_in = h_out * stride - pad;
        int w_in = w_out * stride - pad;
        float* data_col_ptr = data_col;
        data_col_ptr += (channel_out * height_col + h_out) * width_col + w_out;
        const float* data_im_ptr = data_im;
        data_im_ptr += (channel_in * height + h_in) * width + w_in;
        for (int i = 0; i < ksize; ++i) {
            for (int j = 0; j < ksize; ++j) {
                int h = h_in + i;
                int w = w_in + j;

                *data_col_ptr = (h >= 0 && w >= 0 && h < height && w < width) ?
                    data_im_ptr[i * width + j] : 0;

                //data_im[(channel_in * height + h_in) * width + w_in + i * width + j];
                //(*data_col_ptr) = data_im_ptr[ii * width + jj];

                data_col_ptr += height_col * width_col;
            }
        }
    }
}

void im2col_ongpu(float *im,
         int channels, int height, int width,
         int ksize, int stride, int pad, float *data_col){
    // We are going to launch channels * height_col * width_col kernels, each
    // kernel responsible for copying a single-channel grid.
    int height_col = (height + 2 * pad - ksize) / stride + 1;
    int width_col = (width + 2 * pad - ksize) / stride + 1;
    int num_kernels = channels * height_col * width_col;
    im2col_gpu_kernel<<<(num_kernels+BLOCK-1)/BLOCK,
        BLOCK, 0, get_cuda_stream()>>>(
                num_kernels, im, height, width, ksize, pad,
                stride, height_col,
                width_col, data_col);

    CHECK_CUDA(hipPeekAtLastError());
}
// --------------------------------

/*
__global__ void im2col_align_gpu_kernel(const int n, const float* data_im,
    const int height, const int width, const int ksize,
    const int pad,
    const int stride,
    const int height_col, const int width_col,
    float *data_col, const int bit_align)
{
    //__shared__ float tmp_s[1];

    int index = blockIdx.x*blockDim.x + threadIdx.x;
    for (; index < n; index += blockDim.x*gridDim.x) {
        int w_out = index % width_col;
        int h_index = index / width_col;
        int h_out = h_index % height_col;
        int channel_in = h_index / height_col;
        int channel_out = channel_in * ksize * ksize;
        int h_in = h_out * stride - pad;
        int w_in = w_out * stride - pad;
        float* data_col_ptr = data_col;
        //data_col_ptr += (channel_out * height_col + h_out) * width_col + w_out;
        data_col_ptr += channel_out * bit_align + h_out * width_col + w_out;
        float* data_col_ptr_32 = data_col + (channel_out * bit_align + h_out * width_col + w_out)/32;
        const float* data_im_ptr = data_im;
        data_im_ptr += (channel_in * height + h_in) * width + w_in;
        for (int i = 0; i < ksize; ++i) {
            for (int j = 0; j < ksize; ++j) {
                int h = h_in + i;
                int w = w_in + j;

                float val = (h >= 0 && w >= 0 && h < height && w < width) ?
                    data_im_ptr[i * width + j] : 0;

                *data_col_ptr = val;
                //tmp_s[0] = val;

                //(*data_col_ptr) = (h >= 0 && w >= 0 && h < height && w < width) ?
                //    data_im_ptr[i * width + j] : 0;

                //float src_val = (h >= 0 && w >= 0 && h < height && w < width) ? data_im_ptr[i * width + j] : 0;
                //unsigned int bit_mask = __ballot_sync(0xffffffff, src_val > 0);
                //if (threadIdx.x % WARP_SIZE == 0) *((unsigned int*)data_col_ptr_32) = bit_mask;
                // use atomicOr() // *dst_ptr |= (mask << (col_index % 8));
                //data_col_ptr_32 += bit_align / 32;

                //data_col_ptr += height_col * width_col;
                data_col_ptr += bit_align;
            }
        }
    }
}
*/

// float 32
__global__ void im2col_align_gpu_kernel(const int n, const float* data_im,
    const int height, const int width, const int ksize,
    const int pad,
    const int stride,
    const int height_col, const int width_col,
    float *data_col, const int bit_align)
{
    //__shared__ float tmp_s[1];


    int index = blockIdx.x*blockDim.x + threadIdx.x;
    for (; index < n; index += blockDim.x*gridDim.x) {
        int w_out = index % width_col;
        int h_index = index / width_col;
        int h_out = h_index % height_col;
        int channel_in = h_index / height_col;
        int channel_out = channel_in * ksize * ksize;
        int h_in = h_out * stride - pad;
        int w_in = w_out * stride - pad;
        //float* data_col_ptr = data_col;
        //float* data_col_ptr_32 = data_col + (channel_out * bit_align + h_out * width_col + w_out) / 32;
        //data_col_ptr += (channel_out * height_col + h_out) * width_col + w_out;
        //data_col_ptr += channel_out * bit_align + h_out * width_col + w_out;
        float* data_col_ptr = &data_col[channel_out * bit_align + h_out * width_col + w_out];
        const float* data_im_ptr = data_im;
        data_im_ptr += (channel_in * height + h_in) * width + w_in;
        for (int i = 0; i < ksize; ++i) {
            for (int j = 0; j < ksize; ++j) {
                int h = h_in + i;
                int w = w_in + j;

                float val = (h >= 0 && w >= 0 && h < height && w < width) ?
                    data_im_ptr[i * width + j] : 0;

                int pre_out_index = index % (width_col*height_col);
                int out_index = (channel_out + i*ksize + j) * bit_align + pre_out_index;// h_out * width_col + w_out;
                data_col[out_index] = val;

                //(*data_col_ptr) = val;
                //dst_s[threadIdx.x] = val;
                //tmp_s[0] = val;

                //(*data_col_ptr) = (h >= 0 && w >= 0 && h < height && w < width) ?
                //    data_im_ptr[i * width + j] : 0;

                //float src_val = (h >= 0 && w >= 0 && h < height && w < width) ? data_im_ptr[i * width + j] : 0;
                //unsigned int bit_mask = __ballot_sync(0xffffffff, src_val > 0);
                //if (threadIdx.x % WARP_SIZE == 0) *((unsigned int*)data_col_ptr_32) = bit_mask;
                // use atomicOr() // *dst_ptr |= (mask << (col_index % 8));
                //data_col_ptr_32 += bit_align / 32;

                //data_col_ptr += height_col * width_col;
                data_col_ptr += bit_align;
            }
        }
    }
}

void im2col_align_ongpu(float *im,
    int channels, int height, int width,
    int ksize, int stride, int pad, float *data_col, int bit_align) {
    // We are going to launch channels * height_col * width_col kernels, each
    // kernel responsible for copying a single-channel grid.
    int height_col = (height + 2 * pad - ksize) / stride + 1;
    int width_col = (width + 2 * pad - ksize) / stride + 1;
    int num_kernels = channels * height_col * width_col;
    im2col_align_gpu_kernel << <(num_kernels + BLOCK - 1) / BLOCK,
        BLOCK, 0, get_cuda_stream() >> >(
            num_kernels, im, height, width, ksize, pad,
            stride, height_col,
            width_col, data_col, bit_align);

    CHECK_CUDA(hipPeekAtLastError());
}


// --------------------------------



// binary im2col - stride=1
__global__ void im2col_align_bin_gpu_kernel(const int n, const float* data_im,
    const int height, const int width, const int ksize, const int channels,
    const int pad,
    const int stride,
    const int height_col, const int width_col,
    float *data_col, const int bit_align)
{
    //__shared__ float tmp_s[1];
    //__shared__ ulonglong4 tmp256_s[1];


    //#define SHRED_VALS ((BLOCK / 169) * )
    //__shared__ float dst_s[1024];
    //__shared__ float dst_s[1024];
    //__shared__ uint32_t bit_s[32];
    //__shared__ uint8_t bit_s[128];

    int index = blockIdx.x*blockDim.x + threadIdx.x;
    //for (; index < n; index += blockDim.x*gridDim.x)
    {
        int c_index = index;
        int channel_in = c_index % channels;

        //int h_out = index % height_col;
        //int c_index = index / height_col;
        //int channel_in = c_index % channels;

        int channel_out = channel_in * ksize * ksize;

        int j_index = c_index / channels;
        int j = j_index % ksize;
        int i = j_index / ksize;

        int pre_out_index = (channel_out + i*ksize + j) * bit_align;
        int j_pad = (j - pad);
        int i_pad = (i - pad);

        for(int wh_index = 0; wh_index < (height_col*width_col); wh_index += 32)
        //for (int h_out = 0; h_out < height_col; ++h_out)
        {

            // the end of padding
            //if(0)
            //for (int w_out = 0; w_out < (width_col); w_out += 32)
            {
                const int w_out = wh_index % width_col;
                const int h_out = wh_index / width_col;

                const int w = w_out + j_pad;
                const int h = h_out + i_pad;

                int pre_in_index = channel_in * height * width;
                int pre_in_wh_index = h * width + w;

                int send_wh_index = wh_index;
                if (i >= ksize) send_wh_index = height_col*width_col;

                #pragma unroll
                for (int t = 0; t < WARP_SIZE; ++t)
                {
                    const int lane_id = threadIdx.x % WARP_SIZE;

                    const int cur_wh_index = __shfl_custom(send_wh_index, t) + lane_id;

                    if (cur_wh_index < (width_col*height_col))// && (cur_i_pad+pad) < ksize)
                    {
                        const int cur_pre_out_index = __shfl_custom(pre_out_index, t);

                        const int cur_pre_in_index = __shfl_custom(pre_in_index, t);
                        const int cur_pre_in_wh_index = __shfl_custom(pre_in_wh_index, t) + lane_id;

                        int w = cur_pre_in_wh_index % width;
                        int h = cur_pre_in_wh_index / width;
                        int in_index = cur_pre_in_index + cur_pre_in_wh_index;

                        int out_index = cur_pre_out_index + cur_wh_index;

                        float val = (w >= 0 && w < width && h >= 0 && h < height) ?
                            data_im[in_index] : float();

                        //data_col[out_index] = val;
                        //tmp_s[0] = val;

                        uint32_t bit_mask = __ballot_custom(val > 0);
                        if (lane_id == 0) {
                            uint8_t *bit8_ptr = &(((uint8_t *)data_col)[out_index / 8]);
                            uint32_t *bit32_ptr = (uint32_t *)bit8_ptr;
                            *bit32_ptr = bit_mask;
                        }
                    }


                }

            }// w_out

        }
    }
}


void im2col_align_bin_ongpu(float *im,
    int channels, int height, int width,
    int ksize, int stride, int pad, float *data_col, int bit_align) {
    // We are going to launch channels * height_col * width_col kernels, each
    // kernel responsible for copying a single-channel grid.
    int height_col = (height + 2 * pad - ksize) / stride + 1;
    int width_col = (width + 2 * pad - ksize) / stride + 1;
    //int num_kernels = channels * height_col * width_col * ksize * ksize;
    //int num_kernels = channels * ksize * ksize * height_col;
    int num_kernels = channels * ksize * ksize;
    int num_blocks = num_kernels / BLOCK + 1;

    //im2col_align_bin_gpu_kernel << <(num_kernels + BLOCK - 1) / BLOCK,
    im2col_align_bin_gpu_kernel << <num_blocks,
        BLOCK, 0, get_cuda_stream() >> >(
            num_kernels, im, height, width, ksize, channels, pad,
            stride, height_col,
            width_col, data_col, bit_align);

    CHECK_CUDA(hipPeekAtLastError());
}
// --------------------------------

/*
__global__ void float_to_bit_gpu_kernel(float *src, unsigned char *dst, size_t size)
{
    //const int size_aligned = size + (WARP_SIZE - size % WARP_SIZE);

    int index = blockIdx.x*blockDim.x + threadIdx.x;
    float src_val;

    //for (; index < size_aligned; index += blockDim.x*gridDim.x)
    {
        //src_val = src[index];
        if(index < size) src_val = src[index];
        else src_val = 0;
        //unsigned int bit_mask = __ballot_sync(0xffffffff, src_val > 0);
        unsigned int bit_mask = __ballot_custom(src_val > 0);
        if (threadIdx.x % WARP_SIZE == 0) ((unsigned int*)dst)[index / 32] = bit_mask;
    }
}
*/

/*
__global__ void float_to_bit_gpu_kernel(float *src, unsigned char *dst, size_t size)
{
    //const int size_aligned = size + (WARP_SIZE - size % WARP_SIZE);
    __shared__ uint32_t tmp[WARP_SIZE];

    int index = blockIdx.x*blockDim.x + threadIdx.x;
    float src_val;
    uint32_t *dst32_ptr = ((unsigned int*)dst);

    //for (; index < size_aligned; index += blockDim.x*gridDim.x)
    {
        //src_val = src[index];
        if (index < size) src_val = src[index];
        else src_val = 0;
        //unsigned int bit_mask = __ballot_sync(0xffffffff, src_val > 0);
        const int num_of_warps = blockDim.x / WARP_SIZE;
        const int warp_id = threadIdx.x / WARP_SIZE;
        const int lane_id = threadIdx.x % WARP_SIZE;

        uint32_t bit_mask = __ballot_custom(src_val > 0);

        if (lane_id == 0) tmp[warp_id] = bit_mask;

        __syncthreads();
        if (warp_id == 0) {
            if (lane_id < num_of_warps) {
                dst32_ptr[index / 32 + lane_id] = tmp[lane_id];
            }
        }
        __syncthreads();
    }
}
*/

__global__ void float_to_bit_gpu_kernel(float *src, unsigned char *dst, size_t size)
{
    __shared__ uint32_t tmp[WARP_SIZE*32];

    int index = 32*blockIdx.x*blockDim.x + threadIdx.x;
    float src_val;
    uint32_t *dst32_ptr = ((unsigned int*)dst);

    int i;
    for(i = 0; i < 32; ++i)
    {
        if ((index + i * 1024) < size) src_val = src[index + i*1024];
        else src_val = 0;
        //unsigned int bit_mask = __ballot_sync(0xffffffff, src_val > 0);
        //const int num_of_warps = blockDim.x / WARP_SIZE;
        const int warp_id = threadIdx.x / WARP_SIZE;
        const int lane_id = threadIdx.x % WARP_SIZE;

        uint32_t bit_mask = __ballot_custom(src_val > 0);
        if (lane_id == 0) tmp[i * 32 + warp_id] = bit_mask;
    }
    __syncthreads();
    dst32_ptr[blockIdx.x*blockDim.x + threadIdx.x] = tmp[threadIdx.x];
}


void float_to_bit_gpu(float *src, unsigned char *dst, size_t size)
{
    //const int num_blocks = size / 1024 + 1;
    //const int num_blocks = size / (32*1024) + 1;
    const int num_blocks = get_number_of_blocks(size, 32 * 1024);
    float_to_bit_gpu_kernel<<<num_blocks, 1024, 0, get_cuda_stream()>>>(src, dst, size);
    CHECK_CUDA(hipPeekAtLastError());
}
// --------------------------------

/*
__device__ __host__ static inline void remove_bit(unsigned char *const dst, size_t index) {
    size_t dst_i = index / 8;
    int dst_shift = index % 8;
    dst[dst_i] &= ~(1 << dst_shift);
}

__device__ __host__ static inline void set_bit(unsigned char *const dst, size_t index) {
    size_t dst_i = index / 8;
    int dst_shift = index % 8;
    dst[dst_i] |= 1 << dst_shift;
    //dst[dst_i] |= 1 << (8 - dst_shift);
}
*/

__device__ __host__ static inline unsigned char get_bit(unsigned char const*const src, size_t index) {
    size_t src_i = index / 8;
    int src_shift = index % 8;
    unsigned char val = (src[src_i] & (1 << src_shift)) > 0;
    //unsigned char val = (src[src_i] & (1 << (8 - src_shift))) > 0;
    return val;
}

// Intel CPUs and nVidia CUDA GPU are little endian
__device__ __host__ unsigned char reverse_byte(unsigned char a)
{
    return ((a & 0x1) << 7) | ((a & 0x2) << 5) |
        ((a & 0x4) << 3) | ((a & 0x8) << 1) |
        ((a & 0x10) >> 1) | ((a & 0x20) >> 3) |
        ((a & 0x40) >> 5) | ((a & 0x80) >> 7);
}

__device__ __host__ unsigned char reverse_byte_2(unsigned char a)
{
    return ((a * 0x0802LU & 0x22110LU) | (a * 0x8020LU & 0x88440LU)) * 0x10101LU >> 16;
}

__device__ unsigned char reverse_byte_CUDA(unsigned char a)
{
    uint32_t tmp = __brev(a);
    return tmp >> 24;
}

__device__ void transpose8rS32_reversed_diagonale(unsigned char* A, unsigned char* B, int m, int n)
{
    unsigned x, y, t;

    // Load the array and pack it into x and y.
    x = (A[0] << 24) | (A[m] << 16) | (A[2 * m] << 8) | A[3 * m];
    y = (A[4 * m] << 24) | (A[5 * m] << 16) | (A[6 * m] << 8) | A[7 * m];

    t = (x ^ (x >> 7)) & 0x00AA00AA;  x = x ^ t ^ (t << 7);
    t = (y ^ (y >> 7)) & 0x00AA00AA;  y = y ^ t ^ (t << 7);

    t = (x ^ (x >> 14)) & 0x0000CCCC;  x = x ^ t ^ (t << 14);
    t = (y ^ (y >> 14)) & 0x0000CCCC;  y = y ^ t ^ (t << 14);

    t = (x & 0xF0F0F0F0) | ((y >> 4) & 0x0F0F0F0F);
    y = ((x << 4) & 0xF0F0F0F0) | (y & 0x0F0F0F0F);
    x = t;

    B[7 * n] = reverse_byte_CUDA(x >> 24);  B[6 * n] = reverse_byte_CUDA(x >> 16);  B[5 * n] = reverse_byte_CUDA(x >> 8);  B[4 * n] = reverse_byte_CUDA(x);
    B[3 * n] = reverse_byte_CUDA(y >> 24);  B[2 * n] = reverse_byte_CUDA(y >> 16);  B[1 * n] = reverse_byte_CUDA(y >> 8);  B[0 * n] = reverse_byte_CUDA(y);

    //__device__ ​ unsigned int 	__brev(unsigned int  x)
    //Reverse the bit order of a 32 bit unsigned integer.
    // https://docs.nvidia.com/cuda/cuda-math-api/group__CUDA__MATH__INTRINSIC__INT.html
}


// transpose 8x8 bit
__global__ void transpose_bin_gpu_kernel(unsigned char *A, unsigned char *B, const int n, const int m,
    const int lda, const int ldb, const int block_size)
{
    int i;
    int index = blockIdx.x*blockDim.x + threadIdx.x;

    //for (i = 0; i < n; i += 8)
    {
        i = (index*8) % n;
        int j;
        //for (j = 0; j < m - 8; j += 8)
        {
            j = ((index * 8) / n) * 8;
            if (j < m) {
                int a_index = i*lda + j;
                int b_index = j*ldb + i;
                transpose8rS32_reversed_diagonale(&A[a_index / 8], &B[b_index / 8], lda / 8, ldb / 8);
            }
            //else if (j < m) {
            //    for (; j < m; ++j) {
            //        if (get_bit(A, i*lda + j)) set_bit(B, j*ldb + i);
            //        else remove_bit(B, j*ldb + i);
            //    }
            //}
        }
    }
}



__device__ __host__ uint8_t reverse_8_bit(uint8_t a) {
    return ((a * 0x0802LU & 0x22110LU) | (a * 0x8020LU & 0x88440LU)) * 0x10101LU >> 16;
}

__device__ uint32_t reverse_32_bit(uint32_t a)
{
    // __device__ ​ unsigned int __brev(unsigned int  x) // CUDA
    // unsigned int __rbit(unsigned int val) // for ARM    //__asm__("rbit %0, %1\n" : "=r"(output) : "r"(input));
    return __brev(a);
    //return (reverse_8_bit(a >> 24) << 0) |
    //    (reverse_8_bit(a >> 16) << 8) |
    //    (reverse_8_bit(a >> 8) << 16) |
    //    (reverse_8_bit(a >> 0) << 24);
}

#define swap(a0, a1, j, m) t = (a0 ^ (a1 >>j)) & m; a0 = a0 ^ t; a1 = a1 ^ (t << j);

__device__ void transpose32_optimized(uint32_t A[32]) {
    int j, k;
    unsigned m, t;

    //m = 0x0000FFFF;
    //for (j = 16; j != 0; j = j >> 1, m = m ^ (m << j)) {
    //    for (k = 0; k < 32; k = (k + j + 1) & ~j) {
    //        t = (A[k] ^ (A[k + j] >> j)) & m;
    //        A[k] = A[k] ^ t;
    //        A[k + j] = A[k + j] ^ (t << j);
    //    }
    //}

    j = 16;
    m = 0x0000FFFF;
    for (k = 0; k < 32; k = (k + j + 1) & ~j) { swap(A[k], A[k + j], j, m); }

    j = 8;
    m = 0x00ff00ff;
    for (k = 0; k < 32; k = (k + j + 1) & ~j) { swap(A[k], A[k + j], j, m); }

    j = 4;
    m = 0x0f0f0f0f;
    for (k = 0; k < 32; k = (k + j + 1) & ~j) { swap(A[k], A[k + j], j, m); }

    j = 2;
    m = 0x33333333;
    for (k = 0; k < 32; k = (k + j + 1) & ~j) { swap(A[k], A[k + j], j, m); }

    j = 1;
    m = 0x55555555;
    for (k = 0; k < 32; k = (k + j + 1) & ~j) { swap(A[k], A[k + j], j, m); }

    // reverse Y
    for (j = 0; j < 16; ++j) {
        uint32_t tmp = A[j];
        A[j] = reverse_32_bit(A[31 - j]);
        A[31 - j] = reverse_32_bit(tmp);
    }
}

extern "C" {
__device__ void transpose_32x32_bits_reversed_diagonale(uint32_t *A, uint32_t *B, int m, int n)
{
    //unsigned A_tmp[32];
    //int i;
    //#pragma unroll
    //for (i = 0; i < 32; ++i) A_tmp[i] = A[i * m];
    //transpose32_optimized(A_tmp);
    //#pragma unroll
    //for (i = 0; i < 32; ++i) B[i*n] = A_tmp[i];

    __shared__ uint32_t A_shared[32 * BLOCK_TRANSPOSE32];
    uint32_t *A_tmp = &A_shared[32 * threadIdx.x];

    int i;
    #pragma unroll 32
    for (i = 0; i < 32; ++i) A_tmp[i] = A[i * m];
    transpose32_optimized(A_tmp);
    #pragma unroll 32
    for (i = 0; i < 32; ++i) B[i*n] = A_tmp[i];
}
}

// transpose 32x32 bit
__global__ void transpose_bin_gpu_kernel_32(uint32_t *A, uint32_t *B, const int n, const int m,
    const int lda, const int ldb, const int block_size)
{
    int i;
    int index = (blockIdx.x*blockDim.x + threadIdx.x) * 32;

    //for (i = 0; i < n; i += 8)
    {
        i = index % n;
        int j;
        //for (j = 0; j < m - 8; j += 8)
        {
            j = (index / n) * 32;
            if (j < m) {
                int a_index = i*lda + j;
                int b_index = j*ldb + i;
                transpose_32x32_bits_reversed_diagonale(&A[a_index / 32], &B[b_index / 32], lda / 32, ldb / 32);
            }
        }
    }
}

void transpose_bin_gpu(unsigned char *A, unsigned char *B, const int n, const int m,
    const int lda, const int ldb, const int block_size)
{
    //int size = n*m/ (8*8) + 1;
    int size32 = n*m / (32*32) + 1;
    //const int num_blocks = size / BLOCK + 1;
    const int num_blocks32 = size32 / BLOCK_TRANSPOSE32 + 1;
    transpose_bin_gpu_kernel_32 << <num_blocks32, BLOCK_TRANSPOSE32, 0, get_cuda_stream() >> >((uint32_t *)A, (uint32_t *)B, n, m, lda, ldb, block_size);
    //transpose_bin_gpu_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> >(A, B, n, m, lda, ldb, block_size);
    CHECK_CUDA(hipPeekAtLastError());
}
// --------------------------------

__global__ void transpose_uint32_kernel(uint32_t *src, uint32_t *dst, int src_h, int src_w, int src_align, int dst_align)
{
    //l.bit_align - algined (n) by 32
    //new_ldb - aligned (k) by 256
    int index = blockIdx.x*blockDim.x + threadIdx.x;

    //for (i = 0; i < src_h; i += 1)
    int i = index % src_h;  // l.size*l.size*l.c;
    {
        //for (j = 0; j < src_w; j += 1)
        int j = index / src_h;  // out_h*out_w;
        if(j < src_w)
        {
            ((uint32_t *)dst)[j*dst_align / 32 + i] = ((uint32_t *)src)[i*src_align + j];
        }
    }
}

void transpose_uint32_gpu(uint32_t *src, uint32_t *dst, int src_h, int src_w, int src_align, int dst_align)
{
    int size = src_w * src_h;
    const int num_blocks = size / BLOCK + 1;
    transpose_uint32_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> >(src, dst, src_h, src_w, src_align, dst_align);
    CHECK_CUDA(hipPeekAtLastError());
}
// --------------------------------

//#define TRANS_LOOP 10

__global__ void transpose_uint32_kernel_2(uint32_t *src, uint32_t *dst, int src_h, int src_w, int src_align, int dst_align)
{
    __shared__ uint32_t tmp[33 * 32];   // misaligned_array[32x32]
    const int w_align = 33;
    //const int shared_size = w_align * 32;

    //l.bit_align - algined (n) by 32
    //new_ldb - aligned (k) by 256

    const int src_w_align = src_w + (32 - src_w % 32);
    //const int src_h_align = src_h + (32 - src_h % 32);

    const int warps_in_width = src_w_align / 32;
    //const int warps_in_height = src_h_align / 32;



    const int local_x = threadIdx.x % 32;   // index % 32;
    const int local_x_index = threadIdx.x / 32; // index / 32;
    const int local_y = local_x_index % 32;

//#pragma unroll TRANS_LOOP
    //for (int i = 0; i < TRANS_LOOP; ++i)
    {
        const int global_index = blockIdx.x;// blockIdx.x*TRANS_LOOP + i;// local_x_index / 32;
        const int global_x_index = global_index % warps_in_width;
        const int global_y_index = global_index / warps_in_width;

        const int global_x = global_x_index * 32 + local_x;
        const int global_y = global_y_index * 32 + local_y;

        uint32_t val = 0;
        if (global_x < src_w && global_y < src_h) {
            val = src[global_y * src_align + global_x];
        }
        //dst[global_x * dst_align / 32 + global_y] = val;
        //tmp[local_y * 32 + local_x] = val;

        tmp[local_x * w_align + local_y] = val;
        __syncthreads();
        val = tmp[local_y * w_align + local_x];

        const int new_global_x = global_y_index * 32 + local_x;
        const int new_global_y = global_x_index * 32 + local_y;

        if (new_global_x < src_h && new_global_y < src_w) {
            dst[new_global_y * (dst_align / 32) + new_global_x] = val;
        }
    }
}

#define TRANS_BLOCK 1024
void transpose_uint32_gpu_2(uint32_t *src, uint32_t *dst, int src_h, int src_w, int src_align, int dst_align)
{
    int src_w_align = src_w + (32 - src_w % 32);
    int src_h_align = src_h + (32 - src_h % 32);

    int size = src_w_align * src_h_align;
    int num_blocks = size / TRANS_BLOCK;
    transpose_uint32_kernel_2 << <num_blocks, TRANS_BLOCK, 0, get_cuda_stream() >> >(src, dst, src_h, src_w, src_align, dst_align);
    CHECK_CUDA(hipPeekAtLastError());
}
// --------------------------------


// 32 channels -> 1 channel (with 32 floats)
// 256 channels -> 8 channels (with 32 floats)
__global__ void repack_input_kernel(float *input, float *re_packed_input, int w, int h, int c)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;

    const int items_per_channel = w * h;

    int c_pack = index % 32;
    int chan_index = index / 32;
    int chan = (chan_index * 32) % c;
    int i = (chan_index * 32) / c;

    //for (chan = 0; chan < c; chan += 32)
    {
        //for (i = 0; i < items_per_channel; ++i)
        if(i < items_per_channel)
        {
            //for (c_pack = 0; c_pack < 32; ++c_pack)
            {
                float src = input[(chan + c_pack)*items_per_channel + i];

                re_packed_input[chan*items_per_channel + i * 32 + c_pack] = src;
            }
        }
    }
}

void repack_input_gpu(float *input, float *re_packed_input, int w, int h, int c)
{
    int size = w * h * c;
    const int num_blocks = size / BLOCK + 1;
    repack_input_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> >(input, re_packed_input, w, h, c);
    CHECK_CUDA(hipPeekAtLastError());
}
// --------------------------------


// 32 channels -> 1 channel (with 32 floats)
// 256 channels -> 8 channels (with 32 floats)
__global__ void repack_input_kernel_2(float *input, float *re_packed_input, int w, int h, int c)
{
    //__shared__ uint32_t tmp[33 * 32];  // 33x32 is misaligned 32 x 32 to avoid bank conflicts

    int index = blockIdx.x*blockDim.x + threadIdx.x;

    const int items_per_channel = w * h;

    int c_pack = index % 32;
    int chan_index = index / 32;
    int chan = (chan_index * 32) % c;
    int i = (chan_index * 32) / c;

    //for (chan = 0; chan < c; chan += 32)
    {
        //for (i = 0; i < items_per_channel; ++i)
        if (i < items_per_channel)
        {
            //for (c_pack = 0; c_pack < 32; ++c_pack)
            {
                float src = input[(chan + c_pack)*items_per_channel + i];

                re_packed_input[chan*items_per_channel + i * 32 + c_pack] = src;
            }
        }
    }
}

void repack_input_gpu_2(float *input, float *re_packed_input, int w, int h, int c)
{
    int size = w * h * c;
    const int num_blocks = size / BLOCK + 1;
    repack_input_kernel_2 << <num_blocks, BLOCK, 0, get_cuda_stream() >> >(input, re_packed_input, w, h, c);
    CHECK_CUDA(hipPeekAtLastError());
}
// --------------------------------


// 32 channels -> 1 channel (with 32 floats)
// 256 channels -> 8 channels (with 32 floats)
__global__ void repack_input_kernel_bin(float *input, uint32_t *re_packed_input_bin, int w, int h, int c)
{
    //__shared__ uint32_t tmp[32];
    const int index = blockIdx.x*blockDim.x + threadIdx.x;

    const int global_warp_id = index / WARP_SIZE;
    const int lane_id = threadIdx.x % WARP_SIZE;

    const int items_per_channel = w * h;
    const int items_per_channel_aligned = items_per_channel + WARP_SIZE - (items_per_channel % WARP_SIZE);

    int i = 32 * (global_warp_id % (items_per_channel_aligned / WARP_SIZE));
    int chan = 32 * (global_warp_id / (items_per_channel_aligned / WARP_SIZE));

    if (chan < c)
    {
        uint32_t result_bits = 0;

        for (int c_pack = 0; c_pack < 32; ++c_pack)
        {
            float src = 0;
            if ((i + lane_id) < items_per_channel) {
                src = input[(chan + c_pack)*items_per_channel + (i + lane_id)];
            }
            uint32_t bit_mask = __ballot_custom(src > 0);

            uint32_t cur_bit = (bit_mask >> lane_id) & uint32_t(1);

            result_bits |= (cur_bit << c_pack);
        }
        if ((i + lane_id) < items_per_channel) {
            re_packed_input_bin[chan*items_per_channel / 32 + (i + lane_id)] = result_bits;
        }
    }
}

void repack_input_gpu_bin(float *input, uint32_t *re_packed_input_bin, int w, int h, int c)
{
    int size = (w * h * c) / 32 + 1;
    const int block_size = BLOCK;
    const int num_blocks = get_number_of_blocks(size, block_size);
    //printf("\n num_blocks = %d, num_blocks/32 = %d,  block_size = %d \n", num_blocks, num_blocks / 32, block_size);
    repack_input_kernel_bin << <num_blocks, block_size, 0, get_cuda_stream() >> >(input, re_packed_input_bin, w, h, c);
    CHECK_CUDA(hipPeekAtLastError());
}

/*
// 32 channels -> 1 channel (with 32 floats)
// 256 channels -> 8 channels (with 32 floats)
__global__ void repack_input_kernel_bin(float *input, uint32_t *re_packed_input_bin, int w, int h, int c)
{
    //__shared__ uint32_t tmp[32];
    int index = blockIdx.x*blockDim.x + threadIdx.x;

    //const int num_of_warps = blockDim.x / WARP_SIZE;
    //const int warp_id = threadIdx.x / WARP_SIZE;
    //const int lane_id = threadIdx.x % WARP_SIZE;

    const int items_per_channel = w * h;

    int c_pack = index % 32;
    int chan_index = index / 32;
    //int chan = (chan_index * 32) % c;
    //int i = (chan_index * 32) / c;

    int i = (chan_index) % items_per_channel;
    int chan = ((chan_index ) / items_per_channel)*32;


    //for (chan = 0; chan < c; chan += 32)
    if(chan < c)
    {
        //for (i = 0; i < items_per_channel; ++i)
        //if (i < items_per_channel)
        {
            //for (c_pack = 0; c_pack < 32; ++c_pack)
            {
                float src = input[(chan + c_pack)*items_per_channel + i];

                uint32_t bit_mask = __ballot_custom(src > 0);
                if (threadIdx.x % 32 == 0)
                    re_packed_input_bin[chan*items_per_channel / 32 + i] = bit_mask;
            }
        }
    }
}

void repack_input_gpu_bin(float *input, uint32_t *re_packed_input_bin, int w, int h, int c)
{
    int size = w * h * c;
    const int block_size = 256;// 128;
    const int num_blocks = get_number_of_blocks(size, block_size);
    printf("\n num_blocks = %d, num_blocks/32 = %d,  block_size = %d \n", num_blocks, num_blocks/32, block_size);
    repack_input_kernel_bin << <num_blocks, block_size, 0, get_cuda_stream() >> >(input, re_packed_input_bin, w, h, c);
    CHECK_CUDA(hipPeekAtLastError());
}
*/



__global__ void fill_int8_gpu_kernel(unsigned char *src, unsigned char val, size_t size) {
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    if(index < size) src[index] = 0;
}

void fill_int8_gpu(unsigned char *src, unsigned char val, size_t size) {
    const int num_blocks = size / BLOCK + 1;
    fill_int8_gpu_kernel<<<num_blocks, BLOCK, 0, get_cuda_stream()>>>(src, val, size);
    CHECK_CUDA(hipPeekAtLastError());
}
// --------------------------------

//typedef unsigned long long int uint64_t;
//typedef unsigned int uint32_t;
//typedef unsigned char uint8_t;
//typedef char int8_t;
/*
__device__ __host__ static inline uint64_t broadcast_bit_1_to_64(uint8_t src) {
    return (src > 0) ? 0xFFFFFFFFFFFFFFFF : 0;
}
*/
__device__ __host__ static inline uint8_t xnor_bit1(uint8_t a, uint8_t b) {
    return ~(a^b) & 0b1;
}
/*
__device__ __host__ static inline uint32_t xnor_int32(uint32_t a, uint32_t b) {
    return ~(a^b);
}

__device__ __host__ static inline uint64_t xnor_int64(uint64_t a, uint64_t b) {
    return ~(a^b);
}

__device__ __host__ static inline uint4 xnor_int128(uint4 a, uint4 b) {
    uint4 res;
    res.w = ~(a.w^b.w);
    res.x = ~(a.x^b.x);
    res.y = ~(a.y^b.y);
    res.z = ~(a.z^b.z);
    return res;
}

__device__ __host__ static inline ulonglong4 xnor_int256(ulonglong4 a, ulonglong4 b) {
    ulonglong4 res;
    res.w = ~(a.w^b.w);
    res.x = ~(a.x^b.x);
    res.y = ~(a.y^b.y);
    res.z = ~(a.z^b.z);
    return res;
}
*/
//-------
/*
__device__ __host__ static inline uint8_t xor_bit1(uint8_t a, uint8_t b) {
    return (a^b) & 0b1;
}
*/
__device__ __host__ static inline uint32_t xor_int32(uint32_t a, uint32_t b) {
    return (a^b);
}

__device__ __host__ static inline uint64_t xor_int64(uint64_t a, uint64_t b) {
    return (a^b);
}
/*
__device__ __host__ static inline uint4 xor_int128(uint4 a, uint4 b) {
    uint4 res;
    res.w = (a.w^b.w);
    res.x = (a.x^b.x);
    res.y = (a.y^b.y);
    res.z = (a.z^b.z);
    return res;
}
*/
__device__ __host__ static inline ulonglong4 xor_int256(ulonglong4 a, ulonglong4 b) {
    ulonglong4 res;
    res.w = (a.w^b.w);
    res.x = (a.x^b.x);
    res.y = (a.y^b.y);
    res.z = (a.z^b.z);
    return res;
}

/*
__device__ static inline int popcnt_256(ulonglong4 a) {
    return __popcll(a.w) + __popcll(a.x) + __popcll(a.y) + __popcll(a.z);
}

__global__ void gemm_nn_custom_bin_mean_transposed_gpu_kernel(int M, int N, int K,
    unsigned char *A, int lda,
    unsigned char *B, int ldb,
    float *C, int ldc, float *mean_arr)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;

    //if (index == 0)
    {
        int i, j, k, h;

        //#pragma omp parallel for
        //for (i = 0; i < M; ++i)
        i = index % M;
        //if(i < M)
        {   // l.n - filters [16 - 55 - 1024]
            float mean_val = mean_arr[i];

            //for (j = 0; j < N; ++j)
            j = index / M;
            if(j < N)
            { // out_h*out_w - one channel output size [169 - 173056]
                int count = 0;

                for (k = 0; k < K; k += 64) {   // l.size*l.size*l.c - one filter size [27 - 9216]
                    uint64_t a_bit64 = *((uint64_t *)(A + (i*lda + k) / 8));
                    uint64_t b_bit64 = *((uint64_t *)(B + (j*ldb + k) / 8));
                    uint64_t c_bit64 = xnor_int64(a_bit64, b_bit64);

                    int tmp_count = __popcll(c_bit64);

                    if (K - k < 64)  tmp_count = tmp_count - (64 - (K - k));    // remove extra bits
                    count += tmp_count;
                    //binary_int64_printf(c_bit64);
                    //printf(", count = %d \n\n", tmp_count);
                }

                C[i*ldc + j] = (2 * count - K) * mean_val;
            }
        }
    }
}
*/


/*
// B (input) in the shared_memory
__global__ void gemm_nn_custom_bin_mean_transposed_gpu_kernel(int M, int N, int K,
    unsigned char *A, int lda,
    unsigned char *B, int ldb,
    float *C, int ldc, float *mean_arr)
{

    __shared__ uint64_t B_s[4096];  // 32 KB // [ldb x N`] // max = 262 144 bits

    int start_j = blockIdx.x*blockDim.x / M;
    {
        int end_j = (blockIdx.x*blockDim.x + blockDim.x) / M + 1;

        size_t shared_size = ldb * (end_j - start_j);

        //float tmp_shared_size = ldb * (blockDim.x / M);
        //int passes = (4096 * 64) / tmp_shared_size - 1;
        //size_t shared_size = tmp_shared_size * passes;

        int k;
        for (int k = threadIdx.x * 256; k < shared_size; k += blockDim.x * 256) {
            int x = start_j*ldb + k;
            if (x < (N*ldb)) *((ulonglong4 *)(B_s + k / 8)) = *((ulonglong4 *)(B + x / 8));
        }

        ////if (j_cur < N && (index % M == 0 || threadIdx.x == 0)) {
          ////  for (int k = 0; k < K; k += 64) {   // l.size*l.size*l.c - one filter size [27 - 9216]
            ////    *((uint64_t *)(B_s + (local_j*ldb + k) / 8)) = *((uint64_t *)(B + (j_cur*ldb + k) / 8));    // input
            ////}
        ////}
    }
    __syncthreads();

    int index = blockIdx.x*blockDim.x + threadIdx.x;


    //if (index == 0)
    //for(int in_tmp = threadIdx.x; in_tmp < 1*blockDim.x; in_tmp += blockDim.x)
    {
        //int index = blockIdx.x*blockDim.x*1 + in_tmp;

        int j_cur = index / M;
        int local_j = j_cur - start_j;

        int i, j, h;

        //#pragma omp parallel for
        //for (i = 0; i < M; ++i)
        i = index % M;
        //if(i < M)
        {   // l.n - filters [16 - 55 - 1024]
            // further improvements: for (l.n == 1024) iterate several (j)
            float mean_val = mean_arr[i];

            //for (j = 0; j < N; ++j)
            j = index / M;
            if (j < N)
            { // out_h*out_w - one channel output size [169 - 173056]
                const int bit_step = 256;
                int count = 0;
                int k = 0;
                for (k = 0; k < K; k += bit_step) {   // l.size*l.size*l.c - one filter size [27 - 144 - 9216]
                    ulonglong4 a_bit256 = *((ulonglong4 *)(A + (i*lda + k) / 8));    // weights
                    //ulonglong4 b_bit256 = *((ulonglong4 *)(B + (j*ldb + k) / 8));
                    ulonglong4 b_bit256 = *((ulonglong4 *)(B_s + (local_j*ldb + k) / 8));    // input
                    ulonglong4 c_bit256 = xnor_int256(a_bit256, b_bit256);

                    count += __popcll(c_bit256.w) + __popcll(c_bit256.x) +
                        __popcll(c_bit256.y) + __popcll(c_bit256.z);
                }

                int f1 = (K % bit_step == 0) ? 0 : (bit_step - (K % bit_step));
                //C[i*ldc + j] += 2 * count*mean_val;
                //C[i*ldc + j] += -2 * f1*mean_val;
                //C[i*ldc + j] += - K*mean_val;

                count = count - f1;    // remove extra bits (from empty space for align only)
                C[i*ldc + j] = (2 * count - K) * mean_val;

                //B_s[0] = (2 * count - K) * mean_val;
            }
        }
    }
}
*/

/*
// A (weights) in the shared_memory
__global__ void gemm_nn_custom_bin_mean_transposed_gpu_kernel(int M, int N, int K,
    unsigned char *A, int lda,
    unsigned char *B, int ldb,
    float *C, int ldc, float *mean_arr)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;

    __shared__ uint64_t A_s[6144];  // 48 KB // [lda x M`]
                                    //__shared__ uint8_t A_s[6144*8];  // 48 KB // [lda x M`]

    int start_i = blockIdx.x*blockDim.x / N;
    int end_i = (blockIdx.x*blockDim.x + blockDim.x) / N + 1;

    size_t shared_size = lda * (end_i - start_i);

    int i_cur = index / N;
    int local_i = i_cur - start_i;

    for (int k = threadIdx.x * 64; k < shared_size; k += blockDim.x * 64) {
        int x = start_i*lda + k;
        if (x < (M*lda)) *((uint64_t *)(A_s + k / 8)) = *((uint64_t *)(A + x / 8));
    }

    //if (i_cur < M && (index % N == 0 || threadIdx.x == 0)) {
    //for (int k = 0; k < K; k += 64) {   // l.size*l.size*l.c - one filter size [27 - 9216]
    //(*(uint64_t *)(A_s + (local_i*lda + k) / 8)) = *((uint64_t *)(A + (i_cur*lda + k) / 8));    // weights
    //  }
    //}

    __syncthreads();

    int i, j, k, h;

    j = index % N;
    {    // out_h*out_w - one channel output size [169 - 173056]
        i = index / N;
        if (i < M)  // l.n - filters [16 - 55 - 1024]
        {
            float mean_val = mean_arr[i];
            int count = 0;

            for (k = 0; k < K; k += 64) {   // l.size*l.size*l.c - one filter size [27 - 9216]
                //uint64_t a_bit64 = *((uint64_t *)(A + (i*lda + k) / 8));    // weights
                uint64_t a_bit64 = *((uint64_t *)(A_s + (local_i*lda + k) / 8));    // weights
                uint64_t b_bit64 = *((uint64_t *)(B + (j*ldb + k) / 8));            // input
                uint64_t c_bit64 = xnor_int64(a_bit64, b_bit64);

                int tmp_count = __popcll(c_bit64);

                if (K - k < 64)  tmp_count = tmp_count - (64 - (K - k));    // remove extra bits
                count += tmp_count;
            }

            C[i*ldc + j] = (2 * count - K) * mean_val;
        }
    }
}
*/

__inline__ __device__
int warpAllReduceSum(int val) {
    for (int mask = WARP_SIZE / 2; mask > 0; mask /= 2)
#if CUDART_VERSION >= 9000
        val += __shfl_xor_sync(FULL_MASK, val, mask);
#else
        val += __shfl_xor(val, mask);
#endif

    return val;
}

// Tensor Cores binary (CC >= 7.3 && CUDA >= 10.0) - __CUDA_SUBBYTE_IMMA__
#if CUDART_VERSION >= 10000
#include <mma.h>

#define WMMA_M 8
#define WMMA_N 8
#define WMMA_K 128
#define WMMA_K32 (WMMA_K/32)

#define WMMA_Nx2 (WMMA_N*2)

// Tensor Cores are used for XOR-GEMM
__global__ void gemm_nn_custom_bin_mean_transposed_tensor_kernel(int M, int N, int K,
    unsigned char *A, int lda,
    unsigned char *B, int ldb,
    float *C, int ldc, float *mean_arr, float *bias_arr, int leaky_activation,
    float *shortcut_in_gpu, float *shortcut_out_gpu)
{
    // total 57%
    int index = blockIdx.x*blockDim.x + threadIdx.x;

    __shared__ int C_s[WMMA_N * WMMA_M * 32 * 2];    // 2 * 8 KB - Temprorary result of GEMM WMMA for 32 warps

    const int lane_id = threadIdx.x % 32;
    const int warp_id = threadIdx.x / 32;
    const int global_warp_id = index / 32;

    const int N_aligned = N + WMMA_Nx2 - (N % WMMA_Nx2);

    /*
    __syncthreads();
    __shared__ uint32_t A_s[8 * 512];   // 8x512 = 8 x 16384 bits, instead of 8x4
    const int start_global_warp_id = blockIdx.x*blockDim.x / 32;
    int start_i = start_global_warp_id / (N_aligned / WMMA_N);
    start_i = start_i * WMMA_M;
    if (start_i + WMMA_M > M) start_i = M - WMMA_M;   // must be: i+7 < M
    for (int tmp_index = threadIdx.x; tmp_index < (8 * 512); tmp_index += blockDim.x)
    {
        int k_tmp = tmp_index % 512;
        int local_i = tmp_index / 512;

        uint32_t a_val = ((uint32_t *)(A))[(start_i + local_i)*lda/32 + k_tmp];
        A_s[local_i * 512 + k_tmp] = a_val;
    }
    __syncthreads();
    */


    int i, j, k;//, h;
    // 47% = 29 + 10 + 8
    j = global_warp_id % (N_aligned / WMMA_Nx2);
    j = j * WMMA_Nx2;
    {    // out_h*out_w - one channel output size [169 - 173056]
        i = global_warp_id / (N_aligned / WMMA_Nx2);
        i = i * WMMA_M;

        //int count = 0;
        k = 0;

        if (i < M)  //if (i < M)  // l.n - filters [16 - 55 - 1024]
        {
            if (j + WMMA_Nx2 > N) j = N - WMMA_Nx2;   // must be: j+7 < N
            if (i + WMMA_M > M) i = M - WMMA_M;   // must be: i+7 < M

#if __CUDA_ARCH__ >= 730
            // Tensor Cores
            using namespace nvcuda;

            wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, wmma::experimental::precision::b1, wmma::row_major> a_frag;
            wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, wmma::experimental::precision::b1, wmma::col_major> b_frag;
            wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, int> c1_frag, c2_frag;
            wmma::fill_fragment(c1_frag, 0); // !!!! XOR isn't XNOR !!!!!!!!!!
            wmma::fill_fragment(c2_frag, 0); // !!!! XOR isn't XNOR !!!!!!!!!!

            // 8 x 8 x 4 (uint32_t, 4 * 32 = 128 bit)
            for (; k < K; k += 128)  // l.size*l.size*l.c - one filter size [27 - 144 - 9216]
            {
                int64_t A_cur_index = (i*lda + k) / 8;  // index in bits
                int64_t B1_cur_index = (j*ldb + k) / 8;  // index in bits
                int64_t B2_cur_index = ((j + 8)*ldb + k) / 8;  // index in bits

                // try to use A that is cached in shared memory - poor performance
                //if (i == start_i) wmma::load_matrix_sync(a_frag, &A_s[k / 32], (512 * 32));   // lda = (128*32) bits
                //else wmma::load_matrix_sync(a_frag, (uint32_t *)(A + A_cur_index), lda);   // lda = M

                // lda, ldb - are in bits
                wmma::load_matrix_sync(a_frag, (uint32_t *)(A + A_cur_index), lda);   // lda = M

                wmma::load_matrix_sync(b_frag, (uint32_t *)(B + B1_cur_index), ldb);   // ldb = K
                wmma::bmma_sync(c1_frag, a_frag, b_frag, c1_frag);    // XOR-GEMM

                wmma::load_matrix_sync(b_frag, (uint32_t *)(B + B2_cur_index), ldb);   // ldb = K
                wmma::bmma_sync(c2_frag, a_frag, b_frag, c2_frag);    // XOR-GEMM
            }
            // C[i*ldc + j]
            wmma::store_matrix_sync(&C_s[warp_id*WMMA_M*WMMA_N], c1_frag, WMMA_N, wmma::mem_row_major);
            wmma::store_matrix_sync(&C_s[warp_id*WMMA_M*WMMA_N + WMMA_M*WMMA_N*32], c2_frag, WMMA_N, wmma::mem_row_major);
#else // __CUDA_ARCH__ >= 730

            // Custom XOR-GEMM
            int k_d = lane_id % 4;
            int i_d = lane_id / 4;
            //int j_d = lane_id / 4;

            int32_t accum_c_val[8*2]; // wmma::fill_fragment(c_frag, 0);
            for (int local_j = 0; local_j < 8*2; ++local_j) {
                accum_c_val[local_j] = 0;
            }

            // 8 x 8 x 4 (uint32_t, 4 * 32 = 128 bit)
            for (; k < K; k += 128)  // l.size*l.size*l.c - one filter size [27 - 144 - 9216]
            {
                //int64_t A_cur_index = (i*lda + k) / 8;
                //int64_t A_cur_index = (local_i*lda + k) / 8;
                //int64_t B_cur_index = (j*ldb + k) / 8;

                // lda, ldb - are in bits
                // 8*4 = 32
                // 8*8 = 64
                int k_d = lane_id % 4;
                int i_d = lane_id / 4;
                int j_d = lane_id / 4;
                uint32_t a_val = *(uint32_t *)(A + ((i + i_d)*lda + (k + k_d*32)) / 8); // wmma::load_matrix_sync(a_frag, (uint32_t *)(A + A_cur_index), lda);

                for (int c_x = 0; c_x < 2; c_x++)
                {
                    uint32_t b_val = *(uint32_t *)(B + ((c_x * 8 + j + j_d)*ldb + (k + k_d * 32)) / 8); // wmma::load_matrix_sync(b_frag, (uint32_t *)(B + B_cur_index), ldb);

                    // wmma::bmma_sync(c_frag, a_frag, b_frag, c_frag);
                    int32_t c_val[8];  // 8 x 32 threads = 256
                    #pragma UNROLL
                    for (int local_j = 0; local_j < 8; ++local_j)
                    {
                        uint32_t b_val_cur = __shfl_custom(b_val, local_j * 4 + k_d);
                        c_val[local_j] = __popc(xor_int32(a_val, b_val_cur));
                    }

                    #pragma UNROLL
                    for (int local_j = 0; local_j < 8; ++local_j)
                    {
                        #pragma UNROLL
                        for (int local_k = 0; local_k < 4; ++local_k) {
                            accum_c_val[local_j + c_x*8] += __shfl_custom(c_val[local_j], i_d * 4 + local_k);
                        }
                    }
                }
            }

            // only the first 8 threads (i) contain 8 good values each, in c_val[8] (j) = 8 x 8 =64
            // wmma::store_matrix_sync(&C_s[warp_id*WMMA_M*WMMA_N], c_frag, WMMA_N, wmma::mem_row_major);
            if (k_d == 0) {
                for (int c_x = 0; c_x < 2; c_x++)
                {
                    for (int local_j = 0; local_j < 8; ++local_j)
                    {
                        C_s[warp_id*WMMA_M*WMMA_N + i_d*WMMA_N + local_j + WMMA_M*WMMA_N*32 * c_x] = accum_c_val[local_j + c_x*8];
                    }
                }
            }
#endif // __CUDA_ARCH__ >= 730

            for(int c_x = 0; c_x < 2; c_x++)
            {
                int j_d = lane_id % WMMA_N;
                {
                    #pragma UNROLL
                    for (int i_d = lane_id / WMMA_N; i_d < WMMA_M; i_d += WMMA_M / 2)
                    {
                        int count = C_s[warp_id*WMMA_M*WMMA_N + i_d*WMMA_N + j_d + WMMA_M*WMMA_N*32*c_x];

                        const int bit_step = 128;
                        int f1 = (K % bit_step == 0) ? 0 : (bit_step - (K % bit_step));
                        count = count - f1;    // remove extra bits (from empty space for align only)

                        count = (2 * count - K);

                        float mean_val = mean_arr[i + i_d];
                        float bias_val = bias_arr[i + i_d];
                        float dst_val = count *mean_val + bias_val;
                        if (leaky_activation)
                            dst_val = (dst_val >= 0) ? (dst_val) : (0.1f*dst_val);    // Leaky activation

                        size_t out_index = (i + i_d)*ldc + (c_x * 8 + j + j_d);
                        C[out_index] = dst_val;

                        if (shortcut_out_gpu) {
                            shortcut_out_gpu[out_index] = shortcut_in_gpu[out_index] + dst_val;
                        }
                    }

                }
            }
        }
    }
}
#endif  // CUDART_VERSION >= 10000

/*
// Tensor Cores are used for XOR-GEMM
__global__ void gemm_nn_custom_bin_mean_transposed_tensor_kernel(int M, int N, int K,
    unsigned char *A, int lda,
    unsigned char *B, int ldb,
    float *C, int ldc, float *mean_arr, float *bias_arr, int leaky_activation)
{
    // total 57%
    int index = blockIdx.x*blockDim.x + threadIdx.x;

    __shared__ int C_s[8*8 * 32];    // Temprorary result of GEMM WMMA

    const int lane_id = threadIdx.x % 32;
    const int warp_id = threadIdx.x / 32;
    const int global_warp_id = index / 32;

    const int N_aligned = N + WMMA_N - (N % WMMA_N);

    int i, j, k, h;
    // 47% = 29 + 10 + 8
    j = global_warp_id % (N_aligned / WMMA_N);
    j = j * WMMA_N;
    {    // out_h*out_w - one channel output size [169 - 173056]
        i = global_warp_id / (N_aligned / WMMA_N);
        i = i * WMMA_M;

        int count = 0;
        k = 0;

        if (i < M)  //if (i < M)  // l.n - filters [16 - 55 - 1024]
        {
            if (j + WMMA_N > N) j = N - WMMA_N;   // must be: j+7 < N
            if (i + WMMA_M > M) i = M - WMMA_M;   // must be: i+7 < M

#if __CUDA_ARCH__ >= 730
            // Tensor Cores
            using namespace nvcuda;

            wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, wmma::experimental::precision::b1, wmma::row_major> a_frag;
            wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, wmma::experimental::precision::b1, wmma::col_major> b_frag;
            wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, int> c_frag;
            wmma::fill_fragment(c_frag, 0); // !!!! XOR isn't XNOR !!!!!!!!!!

            // 8 x 8 x 4 (uint32_t, 4 * 32 = 128 bit)
            for (; k < K; k += 128)  // l.size*l.size*l.c - one filter size [27 - 144 - 9216]
            {
                int64_t A_cur_index = (i*lda + k) / 8;
                //int64_t A_cur_index = (local_i*lda + k) / 8;
                int64_t B_cur_index = (j*ldb + k) / 8;

                // lda, ldb - are in bits
                wmma::load_matrix_sync(a_frag, (uint32_t *)(A + A_cur_index), lda);   // lda = M
                wmma::load_matrix_sync(b_frag, (uint32_t *)(B + B_cur_index), ldb);   // ldb = K

                wmma::bmma_sync(c_frag, a_frag, b_frag, c_frag);    // XOR-GEMM
            }
            // C[i*ldc + j]
            wmma::store_matrix_sync(&C_s[warp_id*WMMA_M*WMMA_N], c_frag, WMMA_N, wmma::mem_row_major);
#else // __CUDA_ARCH__ >= 730

            // Custom XOR-GEMM
            int k_d = lane_id % 4;
            int i_d = lane_id / 4;
            int j_d = lane_id / 4;

            int32_t accum_c_val[8]; // wmma::fill_fragment(c_frag, 0);
            for (int local_j = 0; local_j < 8; ++local_j) {
                accum_c_val[local_j] = 0;
            }

            // 8 x 8 x 4 (uint32_t, 4 * 32 = 128 bit)
            for (; k < K; k += 128)  // l.size*l.size*l.c - one filter size [27 - 144 - 9216]
            {
                int64_t A_cur_index = (i*lda + k) / 8;
                //int64_t A_cur_index = (local_i*lda + k) / 8;
                int64_t B_cur_index = (j*ldb + k) / 8;

                // lda, ldb - are in bits
                // 8*4 = 32
                // 8*8 = 64
                int k_d = lane_id % 4;
                int i_d = lane_id / 4;
                int j_d = lane_id / 4;
                uint32_t a_val = *(uint32_t *)(A + ((i + i_d)*lda + (k + k_d*32)) / 8); // wmma::load_matrix_sync(a_frag, (uint32_t *)(A + A_cur_index), lda);
                uint32_t b_val = *(uint32_t *)(B + ((j + j_d)*ldb + (k + k_d*32)) / 8); // wmma::load_matrix_sync(b_frag, (uint32_t *)(B + B_cur_index), ldb);

                // wmma::bmma_sync(c_frag, a_frag, b_frag, c_frag);
                int32_t c_val[8];  // 8 x 32 threads = 256
                #pragma UNROLL
                for (int local_j = 0; local_j < 8; ++local_j)
                {
                    uint32_t b_val_cur = __shfl_custom(b_val, local_j *4 + k_d);
                    c_val[local_j] = __popc(xor_int32(a_val, b_val_cur));
                }

                #pragma UNROLL
                for (int local_j = 0; local_j < 8; ++local_j)
                {
                    #pragma UNROLL
                    for (int local_k = 0; local_k < 4; ++local_k) {
                        accum_c_val[local_j] += __shfl_custom(c_val[local_j], i_d * 4 + local_k);
                    }
                }
            }

            // only the first 8 threads (i) contain 8 good values each, in c_val[8] (j) = 8 x 8 =64
            // wmma::store_matrix_sync(&C_s[warp_id*WMMA_M*WMMA_N], c_frag, WMMA_N, wmma::mem_row_major);
            if (k_d == 0) {
                for (int local_j = 0; local_j < 8; ++local_j)
                {
                    C_s[warp_id*WMMA_M*WMMA_N + i_d*WMMA_N + local_j] = accum_c_val[local_j];
                }
            }
#endif // __CUDA_ARCH__ >= 730

            {
                int i_d = lane_id % WMMA_M;
                {

                    for (int j_d = lane_id / WMMA_M; j_d < WMMA_N; j_d += WMMA_N / 2)
                    {
                        int count = C_s[warp_id*WMMA_M*WMMA_N + i_d*WMMA_N + j_d];

                        const int bit_step = 128;
                        int f1 = (K % bit_step == 0) ? 0 : (bit_step - (K % bit_step));
                        count = count - f1;    // remove extra bits (from empty space for align only)

                        count = (2 * count - K);

                        float mean_val = mean_arr[i + i_d];
                        float bias_val = bias_arr[i + i_d];
                        float dst_val = count *mean_val + bias_val;
                        if (leaky_activation)
                            dst_val = (dst_val > 0) ? (dst_val) : (0.1f*dst_val);    // Leaky activation

                        C[(i + i_d)*ldc + (j + j_d)] = dst_val;
                    }

                }
            }
        }
    }
}
*/


// Coalescing
// A (weights) in the shared_memory - GOOD
__global__ void gemm_nn_custom_bin_mean_transposed_gpu_kernel(int M, int N, int K,
    unsigned char *A, int lda,
    unsigned char *B, int ldb,
    float *C, int ldc, float *mean_arr, float *bias_arr, int leaky_activation,
    float *shortcut_in_gpu, float *shortcut_out_gpu)
{
    // total 57%
    int index = blockIdx.x*blockDim.x + threadIdx.x;

    __shared__ uint8_t A_s[6144*8/4];
    //__shared__ uint64_t A_s[6144];  // 48 KB // [lda x M`]
    //__shared__ uint8_t A_s[6144*8];  // 48 KB // [lda x M`]

    int start_i = blockIdx.x*blockDim.x / N;
    int end_i = (blockIdx.x*blockDim.x + blockDim.x) / N + 1;

    size_t shared_size = lda * (end_i - start_i);

    int i_cur = index / N;
    int local_i = i_cur - start_i;
    // ~10%
    for (int k = threadIdx.x * 64; k < shared_size; k += blockDim.x * 64) {
        int x = start_i*lda + k;
        if (x < (M*lda)) *((uint64_t *)(A_s + k / 8)) = *((uint64_t *)(A + x / 8));
    }
    __syncthreads();

    int i, j, k; //, h;
    // 47% = 29 + 10 + 8
    j = index % N;
    {    // out_h*out_w - one channel output size [169 - 173056]
        i = index / N;
        //if (i < M)  // l.n - filters [16 - 55 - 1024]
        {
            int count = 0;
            k = 0;

#ifdef NOT_USED
            // 32 thread X 256 bit = 8192 bit
            for (; k < (K - 8192); k += 8192) {   // l.size*l.size*l.c - one filter size [27 - 9216]
                ulonglong4 c_bit256;

                //int64_t A_cur_index = (i*lda + k) / 8;
                int64_t A_cur_index = (local_i*lda + k) / 8;
                int64_t B_cur_index = (j*ldb + k) / 8;
                if (i >= M) A_cur_index = 0;

#pragma unroll
                for (int t = 0; t < WARP_SIZE; ++t) {
                    const int lane_id = threadIdx.x % WARP_SIZE;

                    const int64_t A_i = __shfl_custom(A_cur_index, t) + 32 * lane_id;
                    const int64_t B_i = __shfl_custom(B_cur_index, t) + 32 * lane_id;

                    {
                        //ulonglong4 a_bit256 = *((ulonglong4 *)(A + A_i));    // weights
                        ulonglong4 a_bit256 = *((ulonglong4 *)(A_s + A_i));    // weights
                        ulonglong4 b_bit256 = *((ulonglong4 *)(B + B_i));    // input
                        c_bit256 = xor_int256(a_bit256, b_bit256);
                        int tmp_count = __popcll(c_bit256.w) + __popcll(c_bit256.x) +
                            __popcll(c_bit256.y) + __popcll(c_bit256.z);

                        int sum_count = warpAllReduceSum(tmp_count);
                        if (lane_id == t) count += sum_count;
                    }
                }
            }
#endif


//#ifdef NOT_USED
            // 32 thread X 64 bit = 2048 bit // 29%
            for (; k < (K - 2048); k += 2048) {   // l.size*l.size*l.c - one filter size [27 - 9216]
                uint64_t c_bit64;

                //int64_t A_cur_index = (i*lda + k) / 8;
                int64_t A_cur_index = (local_i*lda + k) / 8;
                int64_t B_cur_index = (j*ldb + k) / 8;
                if (i >= M) A_cur_index = 0;

                #pragma unroll
                for (int t = 0; t < WARP_SIZE; ++t) {
                    const int lane_id = threadIdx.x % WARP_SIZE;

                    const int64_t A_i = __shfl_custom(A_cur_index, t) + 8 * lane_id;
                    const int64_t B_i = __shfl_custom(B_cur_index, t) + 8 * lane_id;

                    {
                        //uint64_t a_bit64 = *((uint64_t *)(A + A_i));    // weights
                        uint64_t a_bit64 = *((uint64_t *)(A_s + A_i));    // weights
                        uint64_t b_bit64 = *((uint64_t *)(B + B_i));    // input
                        c_bit64 = xor_int64(a_bit64, b_bit64);
                        int tmp_count = __popcll(c_bit64);

                        int sum_count = warpAllReduceSum(tmp_count);
                        if (lane_id == t) count += sum_count;
                    }
                }
            }
//#endif

//#ifdef NOT_USED
            // 32 thread X 32 bit = 1024 bit // 10%
            for (; k < (K - 1024); k += 1024) {   // l.size*l.size*l.c - one filter size [27 - 9216]

                //int64_t A_cur_index = (i*lda + k) / 8;
                int64_t A_cur_index = (local_i*lda + k) / 8;
                int64_t B_cur_index = (j*ldb + k) / 8;
                if (i >= M) A_cur_index = 0;

                #pragma unroll
                for (int t = 0; t < WARP_SIZE; ++t) {
                    const int lane_id = threadIdx.x % WARP_SIZE;

                    const int64_t A_i = __shfl_custom(A_cur_index, t) + 4 * lane_id;
                    const int64_t B_i = __shfl_custom(B_cur_index, t) + 4 * lane_id;

                    {
                        //uint64_t a_bit64 = *((uint64_t *)(A + A_i));    // weights
                        uint32_t a_bit32 = *((uint32_t *)(A_s + A_i));    // weights
                        uint32_t b_bit32 = *((uint32_t *)(B + B_i));    // input
                        uint32_t c_bit32 = xor_int32(a_bit32, b_bit32);
                        int tmp_count = __popc(c_bit32);

                        int sum_count = warpAllReduceSum(tmp_count);
                        if (lane_id == t) count += sum_count;
                    }
                }
            }
//#endif

            if (i < M)
            {
                float mean_val = mean_arr[i];
                float bias_val = bias_arr[i];

//#ifdef NOT_USED
                // 8%
                for (; k < K; k += 256) {   // l.size*l.size*l.c - one filter size [27 - 144 - 9216]
                    //ulonglong4 a_bit256 = *((ulonglong4 *)(A + (i*lda + k) / 8));    // weights
                    ulonglong4 a_bit256 = *((ulonglong4 *)(A_s + (local_i*lda + k) / 8));    // weights
                    ulonglong4 b_bit256 = *((ulonglong4 *)(B + (j*ldb + k) / 8));    // input
                    ulonglong4 c_bit256 = xor_int256(a_bit256, b_bit256);

                    count += __popcll(c_bit256.w) + __popcll(c_bit256.x) +
                        __popcll(c_bit256.y) + __popcll(c_bit256.z);
                }
//#endif

#ifdef NOT_USED
                for (; k < K; k += 64) {   // l.size*l.size*l.c - one filter size [27 - 9216]
                    //uint64_t a_bit64 = *((uint64_t *)(A + (i*lda + k) / 8));    // weights
                    uint64_t a_bit64 = *((uint64_t *)(A_s + (local_i*lda + k) / 8));    // weights
                    uint64_t b_bit64 = *((uint64_t *)(B + (j*ldb + k) / 8));            // input
                    uint64_t c_bit64 = xor_int64(a_bit64, b_bit64);

                    count += __popcll(c_bit64);
                }
#endif

                const int bit_step = 256;
                int f1 = (K % bit_step == 0) ? 0 : (bit_step - (K % bit_step));
                count = count - f1;    // remove extra bits (from empty space for align only)
                float dst_val = (2 * count - K) *mean_val + bias_val;
                if(leaky_activation)
                    dst_val = (dst_val >= 0) ? (dst_val) : (0.1f*dst_val);    // Leaky activation
                size_t out_index = i*ldc + j;
                C[out_index] = dst_val;

                if (shortcut_out_gpu) {
                    shortcut_out_gpu[out_index] = shortcut_in_gpu[out_index] + dst_val;
                }
            }
        }
    }
}


// further optimization - use WMMA GEMM for using Tensor Cores
// https://github.com/NVIDIA-developer-blog/code-samples/blob/master/posts/tensor-cores/simpleTensorCoreGEMM.cu
// https://github.com/NVIDIA/cuda-samples/blob/master/Samples/cudaTensorCoreGemm/cudaTensorCoreGemm.cu
// https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#wmma-subbyte
// nvcuda::wmma::col_major ->  cutlass::MatrixLayout::kColumnMajor (matrix is not transposed)

// Matrix A	Matrix B	Accumulator	Matrix Size (m-n-k)
// precision::b1	precision::b1	int	8x8x128

// The only dimensions currently supported by WMMA for XNOR
// const int WMMA_M = 8;
// const int WMMA_N = 8;
// const int WMMA_K = 128;


// GOOD
void gemm_nn_custom_bin_mean_transposed_gpu(int M, int N, int K,
    unsigned char *A, int lda,
    unsigned char *B, int ldb,
    float *C, int ldc, float *mean_arr, float *bias, int leaky_activation,
    float *shortcut_in_gpu, float *shortcut_out_gpu)
{
    int size = M*N;
    const int num_blocks = get_number_of_blocks(size, BLOCK);

    //printf("\n M = %d, N = %d, M %% 8 = %d, N %% 8 = %d \n", M, N, M % 8, N % 8);

    /*
    printf("\n gemm_bin size = %d, num_blocks = %d, M*K = %d KB, N*K = %d KB \n (w) M*K/num_blocks = %d KB, (i) N*K/num_blocks = %d KB \n",
        size, num_blocks, M*K / 1024, N*K / 1024, M*lda / num_blocks / 1024, N*ldb / num_blocks / 1024);
    printf(" M / 512 = %d, N / 512 = %d, M*lda / 512 = %d, N*ldb / 512 = %d \n", M / 512, N / 512, M*lda/512, N*ldb/512);
    */
    //printf(" shared_memory: (w) lda*BLOCK/N = %d, (i) ldb*BLOCK/M = %d, \t lda = %d \n\n", lda*BLOCK / N, ldb*BLOCK / M, lda);


    //if (M % 8 == 0 && N % 8 == 0 && M == 128)
    //if (M >= 32)    // l.n >= 32
#if CUDART_VERSION >= 10000
    if (1)
    {
        const int M_aligned = M + (8 - (M % 8));
        const int N_aligned = N + (16 - (N % 16));
        int size = (M_aligned / 8)*(N_aligned / 16)*WARP_SIZE;
        const int num_blocks = get_number_of_blocks(size, BLOCK);

        //printf(" lda = %d, ldb = %d, ldc = %d, lda/32 = %d, ldb/32 = %d, ldc/32 = %d \n", lda, ldb, ldc, lda / 32, ldb / 32, ldc / 32);
        //printf("  l.c (K/9) = %d, M (l.n) = %d \n", (K%9 == 0)? K / 9: K, M);
        gemm_nn_custom_bin_mean_transposed_tensor_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> > (
            M, N, K,
            A, lda,
            B, ldb,
            C, ldc,
            mean_arr, bias, leaky_activation,
            shortcut_in_gpu, shortcut_out_gpu);

        //hipDeviceSynchronize();
        //getchar();
    }
    else
#endif  //# CUDART_VERSION >= 10000
    {
        gemm_nn_custom_bin_mean_transposed_gpu_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> > (
            M, N, K,
            A, lda,
            B, ldb,
            C, ldc,
            mean_arr, bias, leaky_activation,
            shortcut_in_gpu, shortcut_out_gpu);
    }
    CHECK_CUDA(hipPeekAtLastError());
}
// --------------------------------

/*
void convolve_cpu(float *input, float *weights, float *output, int in_w, int in_h, int in_c, int n, int size, int pad)
{
    int fil;
    // filter index
#pragma omp parallel for      // "omp parallel for" - automatic parallelization of loop by using OpenMP
    for (fil = 0; fil < n; ++fil) {
        int chan, y, x, f_y, f_x;
        // channel index
        for (chan = 0; chan < in_c; ++chan)
            // input - y
            for (y = 0; y < in_h; ++y)
                // input - x
                for (x = 0; x < in_w; ++x)
                {
                    int const output_index = fil*in_w*in_h + y*in_w + x;
                    int const weights_pre_index = fil*in_c*size*size + chan*size*size;
                    int const input_pre_index = chan*in_w*in_h;
                    float sum = 0;

                    // filter - y
                    for (f_y = 0; f_y < size; ++f_y)
                    {
                        int input_y = y + f_y - pad;
                        // filter - x
                        for (f_x = 0; f_x < size; ++f_x)
                        {
                            int input_x = x + f_x - pad;
                            if (input_y < 0 || input_x < 0 || input_y >= in_h || input_x >= in_w) continue;

                            int input_index = input_pre_index + input_y*in_w + input_x;
                            int weights_index = weights_pre_index + f_y*size + f_x;

                            sum += input[input_index] * weights[weights_index];
                        }
                    }
                    // l.output[filters][width][height] +=
                    //        state.input[channels][width][height] *
                    //        l.weights[filters][channels][filter_width][filter_height];
                    output[output_index] += sum;
                }
    }


}
// --------------------------------


void convolve_bin_cpu(float *input, float *weights, float *output, int in_w, int in_h, int in_c, int n,
    int size, int pad, int new_lda, float *mean_arr_gpu)
{
    int fil;
    // filter index
#pragma omp parallel for      // "omp parallel for" - automatic parallelization of loop by using OpenMP
    for (fil = 0; fil < n; ++fil) {
        float mean_val = mean_arr_gpu[fil];
        int chan, y, x, f_y, f_x;
        // channel index
        for (chan = 0; chan < in_c; ++chan)
            // input - y
            for (y = 0; y < in_h; ++y)
                // input - x
                for (x = 0; x < in_w; ++x)
                {
                    int const output_index = fil*in_w*in_h + y*in_w + x;
                    int const weights_pre_index = fil*in_c*size*size + chan*size*size;
                    int const input_pre_index = chan*in_w*in_h;
                    int sum = 0;
                    int good_val = 0;

                    // filter - y
                    for (f_y = 0; f_y < size; ++f_y)
                    {
                        int input_y = y + f_y - pad;
                        // filter - x
                        for (f_x = 0; f_x < size; ++f_x)
                        {
                            int input_x = x + f_x - pad;
                            if (input_y < 0 || input_x < 0 || input_y >= in_h || input_x >= in_w) continue;

                            int input_index = input_pre_index + input_y*in_w + input_x;
                            //int weights_index = weights_pre_index + f_y*size + f_x;
                            //int weights_index = fil*in_c*size*size + chan*size*size + f_y*size + f_x;
                            int weights_index = fil*new_lda + chan*size*size + f_y*size + f_x;

                            //sum += input[input_index] * weights[weights_index];

                            int8_t in_bit = get_bit((uint8_t *)input, input_index);
                            int8_t w_bit = get_bit((uint8_t *)weights, weights_index);
                            int res = xnor_bit1(in_bit, w_bit);
                            sum += res;
                            good_val++;
                            //sum += (res > 0) ? 1 : -1;
                            //in_bit = (in_bit > 0) ? 1 : -1;
                            //w_bit = (w_bit > 0) ? 1 : -1;
                            //int8_t res = in_bit*w_bit;
                            //sum += res;
                            //printf("\n i: %d x w: %d = res: %d \t sum: %d \t mean = %f \n", in_bit, w_bit, res, sum, mean_val);
                        }
                    }
                    //printf("sum = %d, ", sum);
                    sum = sum - (good_val - sum);
                    //printf(" size = %d, sum = %d \n", size, sum);

                    // l.output[filters][width][height] +=
                    //        state.input[channels][width][height] *
                    //        l.weights[filters][channels][filter_width][filter_height];
                    output[output_index] += sum*mean_val;
                }
    }
}
*/
// --------------------------------

__global__ void convolve_gpu_kernel(float *input, float *weights, float *output, int in_w, int in_h, int in_c, int n, int size, int pad)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;

    int fil;
    // filter index
    //for (fil = 0; fil < n; ++fil)
    int chan, y, x, f_y, f_x;
    // channel index
    //for (chan = 0; chan < in_c; ++chan)
    // input - y
    //for (y = 0; y < in_h; ++y)
    // input - x
    //for (x = 0; x < in_w; ++x)
    x = index % in_w;
    int index2 = index / in_w;
    y = index2 % in_h;
    fil = index2 / in_h;
    if (fil < n)
    {

        int const output_index = fil*in_w*in_h + y*in_w + x;
        float sum = 0;

        for (chan = 0; chan < in_c; ++chan)
        {
            int const weights_pre_index = fil*in_c*size*size + chan*size*size;
            int const input_pre_index = chan*in_w*in_h;

            // filter - y
            for (f_y = 0; f_y < size; ++f_y)
            {
                int input_y = y + f_y - pad;
                // filter - x
                for (f_x = 0; f_x < size; ++f_x)
                {
                    int input_x = x + f_x - pad;
                    if (input_y < 0 || input_x < 0 || input_y >= in_h || input_x >= in_w) continue;

                    int input_index = input_pre_index + input_y*in_w + input_x;
                    int weights_index = weights_pre_index + f_y*size + f_x;

                    sum += input[input_index] * weights[weights_index];

                }
            }
            // l.output[filters][width][height] +=
            //        state.input[channels][width][height] *
            //        l.weights[filters][channels][filter_width][filter_height];
            //output[output_index] += sum;
        }
        output[output_index] = sum;
    }

}

void convolve_gpu(float *input, float *weights, float *output, int in_w, int in_h, int in_c, int n, int size, int pad)
{
    int array_size = in_w*in_h*n;    // width X height X filters
    const int num_blocks = array_size / BLOCK + 1;
    //printf("\n array_size = %d, num_blocks = %d, w = %d, h = %d, n = %d, c = %d, pad = %d \n", array_size, num_blocks, in_w, in_h, n, in_c, pad);

    convolve_gpu_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> > (input, weights, output, in_w, in_h, in_c, n, size, pad);
    CHECK_CUDA(hipPeekAtLastError());
}

// --------------------------------

/*
__global__ void convolve_bin_gpu_kernel(float *input, float *weights, float *output, int in_w, int in_h, int in_c, int n,
    int size, int pad, int new_lda, float *mean_arr_gpu)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;

    int fil;
    // filter index
    //for (fil = 0; fil < n; ++fil)
    int chan, y, x, f_y, f_x;
    // channel index
    //for (chan = 0; chan < in_c; ++chan)
    // input - y
    //for (y = 0; y < in_h; ++y)
    // input - x
    //for (x = 0; x < in_w; ++x)
    x = index % in_w;
    int index2 = index / in_w;
    y = index2 % in_h;
    fil = index2 / in_h;
    if (fil < n)    // (1-6 for one BLOCK)
    {
                //float mean_val = mean_arr_gpu[fil];
                int const output_index = fil*in_w*in_h + y*in_w + x;
                int sum = 0;
                int good_val = 0;

                for (chan = 0; chan < in_c; ++chan)
                {
                    //int const weights_pre_index = fil*in_c*size*size + chan*size*size;
                    int const weights_pre_index = fil*new_lda + chan*size*size;
                    int const input_pre_index = chan*in_w*in_h;

                    // filter - y
                    for (f_y = 0; f_y < size; ++f_y)
                    {
                        int input_y = y + f_y - pad;
                        // filter - x
                        for (f_x = 0; f_x < size; ++f_x)
                        {
                            int input_x = x + f_x - pad;
                            if (input_y < 0 || input_x < 0 || input_y >= in_h || input_x >= in_w) continue;

                            int input_index = input_pre_index + input_y*in_w + input_x;
                            int weights_index = weights_pre_index + f_y*size + f_x;
                            //int weights_index = fil*in_c*size*size + chan*size*size + f_y*size + f_x;
                            //int weights_index = fil*new_lda + chan*size*size + f_y*size + f_x;

                            uint8_t in_bit = get_bit((uint8_t *)input, input_index);
                            uint8_t w_bit = get_bit((uint8_t *)weights, weights_index);
                            int res = xnor_bit1(in_bit, w_bit);
                            sum += res;
                            good_val++;

                            //sum += input[input_index] *weights[weights_index];

                        }
                    }
                    // l.output[filters][width][height] +=
                    //        state.input[channels][width][height] *
                    //        l.weights[filters][channels][filter_width][filter_height];
                    //output[output_index] += sum;
                }
                sum = sum - (good_val - sum);
                output[output_index] = sum * mean_arr_gpu[fil]; // atoimcAdd for inter-BLOCK sum
    }

}
*/

__global__ void convolve_bin_gpu_kernel(float *input, float *weights, float *output, int in_w, int in_h, int in_c, int n,
    int size, int pad, int new_lda, float *mean_arr_gpu)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;

    int fil;
    // filter index
    //for (fil = 0; fil < n; ++fil)
    int chan, y, x, f_y, f_x;
    // channel index
    //for (chan = 0; chan < in_c; ++chan)
    // input - y
    //for (y = 0; y < in_h; ++y)
    // input - x
    //for (x = 0; x < in_w; ++x)
    x = index % in_w;
    int index2 = index / in_w;
    y = index2 % in_h;
    fil = index2 / in_h;
    //if (fil < n)    // (1-6 for one BLOCK)
    {
        //float mean_val = mean_arr_gpu[fil];
        int const output_index = fil*in_w*in_h + y*in_w + x;
        int sum = 0;
        int good_val = 0;

        int min_index = blockIdx.x*blockDim.x;
        int min_fil = (min_index / in_w) / in_h;
        int max_index = (blockIdx.x+1)*blockDim.x - 1;
        int max_fil = (max_index / in_w) / in_h;

        __shared__ uint32_t weights_shared[3*3*1024*6/32 + 1];  // 7 KB (6 filters) - use (new_lda) for size calculation
        //const int weights_size = size*size*in_c/8;
        const int weights_size = size*size*in_c / 32 + 1;

        for (int tmp_fil = min_fil; tmp_fil <= max_fil; tmp_fil++) {
            for (int s = threadIdx.x; s < weights_size; s += blockDim.x) {
                //weights_shared[s + (tmp_fil - min_fil)*new_lda / 8] = ((uint8_t *)weights)[tmp_fil*new_lda / 8 + s];
                weights_shared[s + (tmp_fil - min_fil)*new_lda/32] = ((uint32_t *)weights)[tmp_fil*new_lda / 32 + s];
            }
        }
        __syncthreads();

        for (chan = 0; chan < in_c; ++chan)
        {
            //int const weights_pre_index = fil*in_c*size*size + chan*size*size;
            //int const weights_pre_index = fil*new_lda + chan*size*size;
            int const input_pre_index = chan*in_w*in_h;

            __shared__ uint32_t input_shared[416*416/32 + 1];   // 21.2 KB bytes (for input size 832x832)
            const int input_shared_size = in_w*in_h / 32 + 1;
            const int add_input_index = input_pre_index % 32;
            __syncthreads();    // why??? but is required

            for (int s = threadIdx.x; s < input_shared_size; s += blockDim.x) {
                input_shared[s] = ((uint32_t *)input)[input_pre_index / 32 + s];
            }
            __syncthreads();

            /*
            __shared__ uint8_t input_shared[208 * 208 / 8 + 1];   // 5.4 KB bytes (for input size 416x416)
            const int input_shared_size = in_w*in_h / 8 + 1;
            const int add_input_index = input_pre_index % 8;
            __syncthreads();

            for (int s = threadIdx.x; s < input_shared_size; s += blockDim.x) {
                ((uint8_t *)input_shared)[s] = ((uint8_t *)input)[input_pre_index / 8 + s];
            }
            __syncthreads();
            */
            //int src_index = -1;
            //uint32_t input_byte;

            if (fil < n)    // (1-6 for one BLOCK)
            {
                // filter - y
                for (f_y = 0; f_y < size; ++f_y)
                {
                    int input_y = y + f_y - pad;
                    // filter - x
                    for (f_x = 0; f_x < size; ++f_x)
                    {
                        int input_x = x + f_x - pad;
                        if (input_y < 0 || input_x < 0 || input_y >= in_h || input_x >= in_w) continue;

                        //int input_index = input_pre_index + input_y*in_w + input_x;
                        //int weights_index = weights_pre_index + f_y*size + f_x;
                        //int weights_index = fil*in_c*size*size + chan*size*size + f_y*size + f_x;
                        //int weights_index = fil*new_lda + chan*size*size + f_y*size + f_x;

                        //uint8_t in_bit = get_bit((uint8_t *)input, input_index);
                        //uint8_t w_bit = get_bit((uint8_t *)weights, weights_index);

                        //int weights_index = fil*in_c*size*size + chan*size*size + f_y*size + f_x;
                        int weights_shared_index = (fil - min_fil)*new_lda + chan*size*size + f_y*size + f_x;
                        //uint8_t in_bit = get_bit((uint8_t *)weights_shared, weights_shared_index);
                        uint8_t w_bit = get_bit((uint8_t *)weights_shared, weights_shared_index);

                        //int input_index = input_pre_index + input_y*in_w + input_x;
                        int input_shared_index = /*input_pre_index +*/ input_y*in_w + input_x + add_input_index;
                        uint8_t in_bit = get_bit((uint8_t *)input_shared, input_shared_index);
                        /*
                        int new_src_index = input_shared_index / 32;
                        int src_shift = input_shared_index % 32;
                        //if (new_src_index != src_index)
                        {
                            src_index = new_src_index;
                            input_byte = ((uint32_t *)input_shared)[src_index];
                        }
                        uint8_t in_bit = (input_byte & (1 << src_shift)) >> src_shift;
                        */

                        int res = xnor_bit1(in_bit, w_bit);
                        sum += res;
                        good_val++;

                        //sum += input[input_index] *weights[weights_index];

                    }
                }
            }
            // l.output[filters][width][height] +=
            //        state.input[channels][width][height] *
            //        l.weights[filters][channels][filter_width][filter_height];
            //output[output_index] += sum;
        }
        sum = sum - (good_val - sum);
        //output[output_index] = sum * mean_arr_gpu[fil]; // atoimcAdd for inter-BLOCK sum
        atomicAdd(&output[output_index], sum * mean_arr_gpu[fil]);
    }

}

void convolve_bin_gpu(float *input, float *weights, float *output, int in_w, int in_h, int in_c, int n,
    int size, int pad, int new_lda, float *mean_arr_gpu)
{
    int array_size = in_w*in_h*n;    // width X height X filters
    const int num_blocks = array_size / BLOCK + 1;
    //printf("\n array_size = %d, num_blocks = %d, w = %d, h = %d, n = %d, c = %d, pad = %d \n", array_size, num_blocks, in_w, in_h, n, in_c, pad);

    convolve_bin_gpu_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> > (input, weights, output, in_w, in_h, in_c, n, size, pad, new_lda, mean_arr_gpu);
    CHECK_CUDA(hipPeekAtLastError());
}

// --------------------------------

// CUDA: use 512 threads per block
const int CAFFE_CUDA_NUM_THREADS = 512;

// CUDA: number of blocks for threads.
inline int CAFFE_GET_BLOCKS(const int N) {
    return (N + CAFFE_CUDA_NUM_THREADS - 1) / CAFFE_CUDA_NUM_THREADS;
}

// CUDA: grid stride looping
#define CUDA_KERNEL_LOOP(i, n) \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; \
       i < (n); \
       i += blockDim.x * gridDim.x)

// https://github.com/BVLC/caffe/blob/master/src/caffe/util/im2col.cu
__global__ void im2col_gpu_kernel_ext(const int n, const float* data_im,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w,
    const int stride_h, const int stride_w,
    const int dilation_h, const int dilation_w,
    const int height_col, const int width_col,
    float* data_col) {
    CUDA_KERNEL_LOOP(index, n) {
        const int h_index = index / width_col;
        const int h_col = h_index % height_col;
        const int w_col = index % width_col;
        const int c_im = h_index / height_col;
        const int c_col = c_im * kernel_h * kernel_w;
        const int h_offset = h_col * stride_h - pad_h;
        const int w_offset = w_col * stride_w - pad_w;
        float* data_col_ptr = data_col;
        data_col_ptr += (c_col * height_col + h_col) * width_col + w_col;
        const float* data_im_ptr = data_im;
        data_im_ptr += (c_im * height + h_offset) * width + w_offset;
        for (int i = 0; i < kernel_h; ++i) {
            for (int j = 0; j < kernel_w; ++j) {
                int h_im = h_offset + i * dilation_h;
                int w_im = w_offset + j * dilation_w;
                *data_col_ptr =
                    (h_im >= 0 && w_im >= 0 && h_im < height && w_im < width) ?
                    data_im_ptr[i * dilation_h * width + j * dilation_w] : 0;
                data_col_ptr += height_col * width_col;
            }
        }
    }
}


void im2col_gpu_ext(const float* data_im, const int channels,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w,
    const int stride_h, const int stride_w,
    const int dilation_h, const int dilation_w,
    float* data_col)
{
    // We are going to launch channels * height_col * width_col kernels, each
    // kernel responsible for copying a single-channel grid.
    int height_col = (height + 2 * pad_h -
        (dilation_h * (kernel_h - 1) + 1)) / stride_h + 1;
    int width_col = (width + 2 * pad_w -
        (dilation_w * (kernel_w - 1) + 1)) / stride_w + 1;
    int num_kernels = channels * height_col * width_col;
    // NOLINT_NEXT_LINE(whitespace/operators)
    im2col_gpu_kernel_ext << <CAFFE_GET_BLOCKS(num_kernels),
        CAFFE_CUDA_NUM_THREADS >> >(
            num_kernels, data_im, height, width, kernel_h, kernel_w, pad_h,
            pad_w, stride_h, stride_w, dilation_h, dilation_w, height_col,
            width_col, data_col);

    CHECK_CUDA(hipPeekAtLastError());
}