#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipblas.h>
#include <cstring>

#include "dropout_layer.h"
#include "dark_cuda.h"
#include "utils.h"
#include "blas.h"

#include "image_opencv.h"
#include "image.h"


__global__ void dropblock_fast_kernel(float *rand, float prob, int w, int h, int spatial, int filters, int batch, int block_size, float *drop_blocks_scale, float *output)
{
    const int threads = BLOCK;
    const int id = threadIdx.x;
    const int f = blockIdx.x % filters;
    const int b = blockIdx.x / filters;

    __shared__ int prob_block;
    __shared__ int index_block;

    if (id == 0) {
        prob_block = 1.0 * 1000000;
        index_block = -1;
    }
    __syncthreads();

    int i;
    for (i = id; i < spatial; i += threads) {
        int index = b*spatial*f + f*spatial + i;

        if (rand[index] < prob) {
            //Chose with the lowest rand[i]
            int new_val = rand[index] * 1000000;
            rand[index] = 1;
            int old_val = atomicMin(&prob_block, new_val);
            if (new_val < old_val) {
                index_block = i;
                //if (b == 0) printf("\n rand[i] = %f, prob = %f, b = %d, f = %d, i = %d, index_block = %d \n", rand[i], prob, b, f, i, index_block);
            }
        }

    }
    __syncthreads();
    if (index_block == -1) return;


    int b_x = index_block % w;
    int b_y = index_block / w;

    if (b_x > (w - block_size)) b_x = b_x - (w - block_size);
    if (b_y > (h - block_size)) b_y = b_y - (h - block_size);

    b_x = max(0, min(b_x, w - block_size));
    b_y = max(0, min(b_y, h - block_size));

    int block_square_size = block_size * block_size;

    for (i = id; i < block_square_size; i += threads)
    {
        int i_x = i % block_size;
        int i_y = i / block_size;

        int x = b_x + i_x;
        int y = b_y + i_y;

        if (x >= 0 && x < w && y >= 0 && y < h) {
            int new_index = b*filters*spatial + f*spatial + y*w + x;

            output[new_index] = 0;
            rand[new_index] = 0;
        }
    }

    //if (id == 0 && b == 0) printf(" f = %d, b = %d \n", f, b);

    if (id == 0 && drop_blocks_scale) {
        atomicAdd(&drop_blocks_scale[b], block_square_size);
        //if(b == 0) printf("\n index_block = %d \n", index_block);
    }

}

__global__ void set_scales_dropblock_kernel(float *drop_blocks_scale, int block_size_w, int block_size_h, int outputs, int batch)
{
    const int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index >= batch) return;

    //printf(" drop_blocks_scale[index] = %f \n", drop_blocks_scale[index]);
    const float prob = drop_blocks_scale[index] / (float)outputs;
    const float scale = 1.0f / (1.0f - prob);
    drop_blocks_scale[index] = scale;
}

__global__ void scale_dropblock_kernel(float *output, int size, int outputs, float *drop_blocks_scale)
{
    const int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index >= size) return;

    const int b = index / outputs;
    output[index] *= drop_blocks_scale[b];
}


__global__ void backward_dropblock_kernel(float *pass, float *delta, int size)
{
    const int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index >= size) return;

    if (pass[index] == 0) delta[index] = 0;
}


__global__ void yoloswag420blazeit360noscope(float *input, int size, float *rand, float prob, float scale)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id < size) input[id] = (rand[id] < prob) ? 0 : input[id]*scale;
}


void forward_dropout_layer_gpu(dropout_layer l, network_state state)
{
    if (!state.train) return;
    int iteration_num = get_current_iteration(state.net); // (*state.net.seen) / (state.net.batch*state.net.subdivisions);
    //if (iteration_num < state.net.burn_in) return;

    // We gradually increase the block size and the probability of dropout - during the first half of the training
    float multiplier = 1.0;
    if(iteration_num < (state.net.max_batches*0.85))
        multiplier = (iteration_num / (float)(state.net.max_batches*0.85));

    // dropblock
    if (l.dropblock) {
        //l.probability = 1 / keep_prob
        //const int max_blocks_per_channel = 10;
        const float cur_prob = l.probability * multiplier;
        const float cur_scale = 1.f / (1.f - cur_prob);

        int block_width = l.dropblock_size_abs *multiplier;
        int block_height = l.dropblock_size_abs *multiplier;

        if (l.dropblock_size_rel) {
            block_width = l.dropblock_size_rel * l.w * multiplier;
            block_height = l.dropblock_size_rel * l.h * multiplier;
        }

        block_width = max_val_cmp(1, block_width);
        block_height = max_val_cmp(1, block_height);

        block_width = min_val_cmp(l.w, block_width);
        block_height = min_val_cmp(l.h, block_height);

        const int block_size = min_val_cmp(block_width, block_height);
        const float block_prob = cur_prob / (block_size*block_size);
        assert(block_size <= l.w && block_size <= l.h);

        const int size = l.inputs*l.batch;
        cuda_random(l.rand_gpu, size);

        fill_ongpu(l.batch, 0, l.drop_blocks_scale_gpu, 1);

        //fill_ongpu(l.outputs * l.batch, 1, state.input, 1); // remove!!!

        int num_blocks = l.batch * l.c;
        dropblock_fast_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> > (l.rand_gpu, block_prob, l.w, l.h, l.w*l.h, l.c, l.batch, block_size, l.drop_blocks_scale_gpu, state.input);
        CHECK_CUDA(hipPeekAtLastError());

        num_blocks = get_number_of_blocks(l.batch, BLOCK);
        set_scales_dropblock_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> > (l.drop_blocks_scale_gpu, block_size, block_size, l.outputs, l.batch);
        CHECK_CUDA(hipPeekAtLastError());

        /*
        {
            cuda_pull_array(l.drop_blocks_scale_gpu, l.drop_blocks_scale, l.batch);

            float avg_scale = 0;

            for (int b = 0; b < l.batch; ++b) {
                const float scale = l.drop_blocks_scale[b];
                avg_scale += scale;
                printf(" %d x %d - block_size = %d, block_size*block_size = %d , ", l.w, l.h, block_size, block_size*block_size);
                printf(" , l.drop_blocks_scale[b] = %f, scale = %f \t cur_prob = %f, cur_scale = %f \n",
                    l.drop_blocks_scale[b], scale, cur_prob, cur_scale);
            }
            avg_scale = avg_scale / l.batch;
            printf(" avg_scale = %f \n", avg_scale);

            float *output = (float *)calloc(l.outputs * l.batch, sizeof(float));
            cuda_pull_array(state.input, output, l.outputs * l.batch);

            printf(" l.w = %d, l.h = %d, l.c = %d \n", l.w, l.h, l.c);

            image img = float_to_image(l.w, l.h, l.c, output);
            img = collapse_image_layers(img, 1);
            //normalize_image(img);

            show_image(img, "dropout - forward");
            wait_key_cv(0);
            //free_image(img);
            //free(output);
        }
        */

        num_blocks = get_number_of_blocks(l.outputs * l.batch, BLOCK);
        scale_dropblock_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> > (state.input, l.outputs * l.batch, l.outputs, l.drop_blocks_scale_gpu);
        CHECK_CUDA(hipPeekAtLastError());

    }
    // dropout
    else {
        int size = l.inputs*l.batch;
        cuda_random(l.rand_gpu, size);
        /*
        int i;
        for(i = 0; i < size; ++i){
            layer.rand[i] = rand_uniform();
        }
        cuda_push_array(layer.rand_gpu, layer.rand, size);
        */

        yoloswag420blazeit360noscope << <cuda_gridsize(size), BLOCK, 0, get_cuda_stream() >> > (state.input, size, l.rand_gpu, l.probability, l.scale);
        CHECK_CUDA(hipPeekAtLastError());
    }
}

void backward_dropout_layer_gpu(dropout_layer l, network_state state)
{
    if(!state.delta) return;
    //int iteration_num = get_current_iteration(state.net); //(*state.net.seen) / (state.net.batch*state.net.subdivisions);
    //if (iteration_num < state.net.burn_in) return;

    const int size = l.inputs*l.batch;

    // dropblock
    if (l.dropblock) {
        int iteration_num = get_current_iteration(state.net); //(*state.net.seen) / (state.net.batch*state.net.subdivisions);
        float multiplier = 1.0;
        if (iteration_num < (state.net.max_batches*0.85))
            multiplier = (iteration_num / (float)(state.net.max_batches*0.85));

        const float cur_prob = l.probability * multiplier;
        const float cur_scale = 1.f / (1.f - cur_prob);

        int block_width = l.dropblock_size_abs * multiplier;
        int block_height = l.dropblock_size_abs * multiplier;

        if (l.dropblock_size_rel) {
            block_width = l.dropblock_size_rel * l.w * multiplier;
            block_height = l.dropblock_size_rel * l.h * multiplier;
        }

        block_width = max_val_cmp(1, block_width);
        block_height = max_val_cmp(1, block_height);

        block_width = min_val_cmp(l.w, block_width);
        block_height = min_val_cmp(l.h, block_height);

        const int block_size = min_val_cmp(block_width, block_height);
        const float block_prob = cur_prob / (block_size*block_size);

        //fill_ongpu(l.outputs * l.batch, 1, state.delta, 1); // remove!!!

        int num_blocks = get_number_of_blocks(l.outputs * l.batch, BLOCK);
        backward_dropblock_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> >(l.rand_gpu, state.delta, l.outputs * l.batch);
        CHECK_CUDA(hipPeekAtLastError());

        scale_dropblock_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> > (state.delta, l.outputs * l.batch, l.outputs, l.drop_blocks_scale_gpu);
        CHECK_CUDA(hipPeekAtLastError());

        /*
        {
            cuda_pull_array(l.drop_blocks_scale_gpu, l.drop_blocks_scale, l.batch);

            float avg_scale = 0;

            for (int b = 0; b < l.batch; ++b) {
                const float scale = l.drop_blocks_scale[b];
                avg_scale += scale;
                printf(" %d x %d - block_size = %d, block_size*block_size = %d , ", l.w, l.h, block_size, block_size*block_size);
                printf(" , l.drop_blocks_scale[b] = %f, scale = %f \t cur_prob = %f, cur_scale = %f \n",
                    l.drop_blocks_scale[b], scale, cur_prob, cur_scale);
            }
            avg_scale = avg_scale / l.batch;
            printf(" avg_scale = %f \n", avg_scale);

            float *output = (float *)calloc(l.outputs * l.batch, sizeof(float));
            cuda_pull_array(state.delta, output, l.outputs * l.batch);

            printf(" l.w = %d, l.h = %d, l.c = %d \n", l.w, l.h, l.c);

            image img = float_to_image(l.w, l.h, l.c, output);
            img = collapse_image_layers(img, 1);
            //normalize_image(img);

            show_image(img, "dropout - delta");
            wait_key_cv(0);
            //free_image(img);
            //free(output);
        }
        */

    }
    // dropout
    else {
        yoloswag420blazeit360noscope << <cuda_gridsize(size), BLOCK, 0, get_cuda_stream() >> > (state.delta, size, l.rand_gpu, l.probability, l.scale);
        CHECK_CUDA(hipPeekAtLastError());
    }
}
