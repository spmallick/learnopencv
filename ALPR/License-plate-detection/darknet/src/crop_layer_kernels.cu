#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipblas.h>

#include "crop_layer.h"
#include "utils.h"
#include "dark_cuda.h"
#include "image.h"

__device__ float get_pixel_kernel(float *image, int w, int h, int x, int y, int c)
{
    if(x < 0 || x >= w || y < 0 || y >= h) return 0;
    return image[x + w*(y + c*h)];
}

__device__ float3 rgb_to_hsv_kernel(float3 rgb)
{
    float r = rgb.x;
    float g = rgb.y;
    float b = rgb.z;

    float h, s, v;
    float max = (r > g) ? ( (r > b) ? r : b) : ( (g > b) ? g : b);
    float min = (r < g) ? ( (r < b) ? r : b) : ( (g < b) ? g : b);
    float delta = max - min;
    v = max;
    if(max == 0){
        s = 0;
        h = -1;
    }else{
        s = delta/max;
        if(r == max){
            h = (g - b) / delta;
        } else if (g == max) {
            h = 2 + (b - r) / delta;
        } else {
            h = 4 + (r - g) / delta;
        }
        if (h < 0) h += 6;
    }
    return make_float3(h, s, v);
}

__device__ float3 hsv_to_rgb_kernel(float3 hsv)
{
    float h = hsv.x;
    float s = hsv.y;
    float v = hsv.z;

    float r, g, b;
    float f, p, q, t;

    if (s == 0) {
        r = g = b = v;
    } else {
        int index = (int) floorf(h);
        f = h - index;
        p = v*(1-s);
        q = v*(1-s*f);
        t = v*(1-s*(1-f));
        if(index == 0){
            r = v; g = t; b = p;
        } else if(index == 1){
            r = q; g = v; b = p;
        } else if(index == 2){
            r = p; g = v; b = t;
        } else if(index == 3){
            r = p; g = q; b = v;
        } else if(index == 4){
            r = t; g = p; b = v;
        } else {
            r = v; g = p; b = q;
        }
    }
    r = (r < 0) ? 0 : ((r > 1) ? 1 : r);
    g = (g < 0) ? 0 : ((g > 1) ? 1 : g);
    b = (b < 0) ? 0 : ((b > 1) ? 1 : b);
    return make_float3(r, g, b);
}

__device__ float bilinear_interpolate_kernel(float *image, int w, int h, float x, float y, int c)
{
    int ix = (int) floorf(x);
    int iy = (int) floorf(y);

    float dx = x - ix;
    float dy = y - iy;

    float val = (1-dy) * (1-dx) * get_pixel_kernel(image, w, h, ix, iy, c) +
        dy     * (1-dx) * get_pixel_kernel(image, w, h, ix, iy+1, c) +
        (1-dy) *   dx   * get_pixel_kernel(image, w, h, ix+1, iy, c) +
        dy     *   dx   * get_pixel_kernel(image, w, h, ix+1, iy+1, c);
    return val;
}

__global__ void levels_image_kernel(float *image, float *rand, int batch, int w, int h, int train, float saturation, float exposure, float translate, float scale, float shift)
{
    int size = batch * w * h;
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id >= size) return;
    int x = id % w;
    id /= w;
    int y = id % h;
    id /= h;
    float rshift = rand[0];
    float gshift = rand[1];
    float bshift = rand[2];
    float r0 = rand[8*id + 0];
    float r1 = rand[8*id + 1];
    float r2 = rand[8*id + 2];
    float r3 = rand[8*id + 3];

    saturation = r0*(saturation - 1) + 1;
    saturation = (r1 > .5) ? 1./saturation : saturation;
    exposure = r2*(exposure - 1) + 1;
    exposure = (r3 > .5) ? 1./exposure : exposure;

    size_t offset = id * h * w * 3;
    image += offset;
    float r = image[x + w*(y + h*0)];
    float g = image[x + w*(y + h*1)];
    float b = image[x + w*(y + h*2)];
    float3 rgb = make_float3(r,g,b);
    if(train){
        float3 hsv = rgb_to_hsv_kernel(rgb);
        hsv.y *= saturation;
        hsv.z *= exposure;
        rgb = hsv_to_rgb_kernel(hsv);
    } else {
        shift = 0;
    }
    image[x + w*(y + h*0)] = rgb.x*scale + translate + (rshift - .5)*shift;
    image[x + w*(y + h*1)] = rgb.y*scale + translate + (gshift - .5)*shift;
    image[x + w*(y + h*2)] = rgb.z*scale + translate + (bshift - .5)*shift;
}

__global__ void forward_crop_layer_kernel(float *input, float *rand, int size, int c, int h, int w, int crop_height, int crop_width, int train, int flip, float angle, float *output)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id >= size) return;

    float cx = w/2.;
    float cy = h/2.;

    int count = id;
    int j = id % crop_width;
    id /= crop_width;
    int i = id % crop_height;
    id /= crop_height;
    int k = id % c;
    id /= c;
    int b = id;

    float r4 = rand[8*b + 4];
    float r5 = rand[8*b + 5];
    float r6 = rand[8*b + 6];
    float r7 = rand[8*b + 7];

    float dw = (w - crop_width)*r4;
    float dh = (h - crop_height)*r5;
    flip = (flip && (r6 > .5));
    angle = 2*angle*r7 - angle;
    if(!train){
        dw = (w - crop_width)/2.;
        dh = (h - crop_height)/2.;
        flip = 0;
        angle = 0;
    }

    input += w*h*c*b;

    float x = (flip) ? w - dw - j - 1 : j + dw;
    float y = i + dh;

    float rx = cos(angle)*(x-cx) - sin(angle)*(y-cy) + cx;
    float ry = sin(angle)*(x-cx) + cos(angle)*(y-cy) + cy;

    output[count] = bilinear_interpolate_kernel(input, w, h, rx, ry, k);
}

extern "C" void forward_crop_layer_gpu(crop_layer layer, network_state state)
{
    cuda_random(layer.rand_gpu, layer.batch*8);

    float radians = layer.angle*3.14159265/180.;

    float scale = 2;
    float translate = -1;
    if(layer.noadjust){
        scale = 1;
        translate = 0;
    }

    int size = layer.batch * layer.w * layer.h;

    levels_image_kernel<<<cuda_gridsize(size), BLOCK, 0, get_cuda_stream() >>>(state.input, layer.rand_gpu, layer.batch, layer.w, layer.h, state.train, layer.saturation, layer.exposure, translate, scale, layer.shift);
    CHECK_CUDA(hipPeekAtLastError());

    size = layer.batch*layer.c*layer.out_w*layer.out_h;

    forward_crop_layer_kernel<<<cuda_gridsize(size), BLOCK, 0, get_cuda_stream() >>>(state.input, layer.rand_gpu, size, layer.c, layer.h, layer.w, layer.out_h, layer.out_w, state.train, layer.flip, radians, layer.output_gpu);
    CHECK_CUDA(hipPeekAtLastError());

/*
       cuda_pull_array(layer.output_gpu, layer.output, size);
       image im = float_to_image(layer.crop_width, layer.crop_height, layer.c, layer.output + 0*(size/layer.batch));
       image im2 = float_to_image(layer.crop_width, layer.crop_height, layer.c, layer.output + 1*(size/layer.batch));
       image im3 = float_to_image(layer.crop_width, layer.crop_height, layer.c, layer.output + 2*(size/layer.batch));

       translate_image(im, -translate);
       scale_image(im, 1/scale);
       translate_image(im2, -translate);
       scale_image(im2, 1/scale);
       translate_image(im3, -translate);
       scale_image(im3, 1/scale);

       show_image(im, "cropped");
       show_image(im2, "cropped2");
       show_image(im3, "cropped3");
       cvWaitKey(0);
       */
}
