#include "hip/hip_runtime.h"
#include "dark_cuda.h"

#include <stdio.h>
#include <time.h>
#include <assert.h>

#include "network.h"
#include "image.h"
#include "data.h"
#include "utils.h"
#include "parser.h"

#include "crop_layer.h"
#include "connected_layer.h"
#include "rnn_layer.h"
#include "gru_layer.h"
#include "crnn_layer.h"
#include "detection_layer.h"
#include "region_layer.h"
#include "convolutional_layer.h"
#include "activation_layer.h"
#include "maxpool_layer.h"
#include "reorg_layer.h"
#include "avgpool_layer.h"
#include "normalization_layer.h"
#include "batchnorm_layer.h"
#include "cost_layer.h"
#include "local_layer.h"
#include "softmax_layer.h"
#include "dropout_layer.h"
#include "route_layer.h"
#include "shortcut_layer.h"
#include "blas.h"

//#ifdef OPENCV
//#include <opencv2/highgui/highgui_c.h>
//#endif

#include "http_stream.h"

float * get_network_output_gpu_layer(network net, int i);
float * get_network_delta_gpu_layer(network net, int i);
float * get_network_output_gpu(network net);

typedef struct time_benchmark_layers {
    float time;
    int layer_id, layer_type;
} time_benchmark_layers;

int time_comparator(const void *pa, const void *pb)
{
    time_benchmark_layers a = *(time_benchmark_layers *)pa;
    time_benchmark_layers b = *(time_benchmark_layers *)pb;
    float diff = a.time - b.time;
    if (diff < 0) return 1;
    else if (diff > 0) return -1;
    return 0;
}

void forward_network_gpu(network net, network_state state)
{
    static time_benchmark_layers *avg_time_per_layer = NULL;
    static time_benchmark_layers *sorted_avg_time_per_layer = NULL;
    double start_time, end_time;
    if (net.benchmark_layers) {
        if (!avg_time_per_layer) {
            avg_time_per_layer = (time_benchmark_layers *)calloc(net.n, sizeof(time_benchmark_layers));
            sorted_avg_time_per_layer = (time_benchmark_layers *)calloc(net.n, sizeof(time_benchmark_layers));
        }
        hipDeviceSynchronize();
    }

    //printf("\n");
    state.workspace = net.workspace;
    int i;
    for(i = 0; i < net.n; ++i){
        state.index = i;
        layer l = net.layers[i];
        if(l.delta_gpu && state.train){
            fill_ongpu(l.outputs * l.batch, 0, l.delta_gpu, 1);
        }

        if (net.benchmark_layers) {
            start_time = get_time_point();
        }

        l.forward_gpu(l, state);

        if (net.benchmark_layers) {
            CHECK_CUDA(hipDeviceSynchronize());
            end_time = get_time_point();
            const double took_time = (end_time - start_time) / 1000;
            const double alpha = 0.9;
            if (avg_time_per_layer[i].time == 0) {
                avg_time_per_layer[i].layer_id = i;
                avg_time_per_layer[i].layer_type = l.type;
                avg_time_per_layer[i].time = took_time;
            }
            else avg_time_per_layer[i].time = avg_time_per_layer[i].time * alpha + took_time * (1 - alpha);

            sorted_avg_time_per_layer[i] = avg_time_per_layer[i];
            printf("\n fw-layer %d - type: %d - %lf ms - avg_time %lf ms \n", i, l.type, took_time, avg_time_per_layer[i].time);
        }

        if(net.wait_stream)
            hipStreamSynchronize(get_cuda_stream());
        state.input = l.output_gpu;
        //hipDeviceSynchronize();

        /*
        cuda_pull_array(l.output_gpu, l.output, l.outputs);
        hipStreamSynchronize(get_cuda_stream());
        float avg_val = 0;
        int k;
        for (k = 0; k < l.outputs; ++k) avg_val += l.output[k];
        printf(" i: %d - avg_val = %f \n", i, avg_val / l.outputs);
        */

/*
        cuda_pull_array(l.output_gpu, l.output, l.batch*l.outputs);
        if (l.out_w >= 0 && l.out_h >= 1 && l.c >= 3) {
            int j;
            for (j = 0; j < l.out_c; ++j) {
                image img = make_image(l.out_w, l.out_h, 3);
                memcpy(img.data, l.output + l.out_w*l.out_h*j, l.out_w*l.out_h * 1 * sizeof(float));
                memcpy(img.data + l.out_w*l.out_h * 1, l.output + l.out_w*l.out_h*j, l.out_w*l.out_h * 1 * sizeof(float));
                memcpy(img.data + l.out_w*l.out_h * 2, l.output + l.out_w*l.out_h*j, l.out_w*l.out_h * 1 * sizeof(float));
                char buff[256];
                sprintf(buff, "layer-%d slice-%d", i, j);
                show_image(img, buff);
                save_image(img, buff);
            }
            cvWaitKey(0); // wait press-key in console
            cvDestroyAllWindows();
        }
*/
    }

    if (net.benchmark_layers) {
        printf("\n\nSorted by time (forward):\n");
        qsort(sorted_avg_time_per_layer, net.n, sizeof(time_benchmark_layers), time_comparator);
        for (i = 0; i < net.n; ++i) {
            //printf("layer %d - type: %d - avg_time %lf ms \n", avg_time_per_layer[i].layer_id, avg_time_per_layer[i].layer_type, avg_time_per_layer[i].time);
            printf("%d - fw-sort-layer %d - type: %d - avg_time %lf ms \n", i, sorted_avg_time_per_layer[i].layer_id, sorted_avg_time_per_layer[i].layer_type, sorted_avg_time_per_layer[i].time);
        }
    }

    //hipStreamSynchronize(get_cuda_stream());   // sync CUDA-functions
    //hipDeviceSynchronize();
}

void backward_network_gpu(network net, network_state state)
{
    static time_benchmark_layers *avg_time_per_layer = NULL;
    static time_benchmark_layers *sorted_avg_time_per_layer = NULL;
    double start_time, end_time;
    if (net.benchmark_layers) {
        if (!avg_time_per_layer) {
            avg_time_per_layer = (time_benchmark_layers *)calloc(net.n, sizeof(time_benchmark_layers));
            sorted_avg_time_per_layer = (time_benchmark_layers *)calloc(net.n, sizeof(time_benchmark_layers));
        }
        hipDeviceSynchronize();
    }

    state.workspace = net.workspace;
    int i;
    float * original_input = state.input;
    float * original_delta = state.delta;
    for(i = net.n-1; i >= 0; --i){
        state.index = i;
        layer l = net.layers[i];
        if (l.stopbackward == 1) break;
        if (l.stopbackward > get_current_iteration(net)) break;
        if(i == 0){
            state.input = original_input;
            state.delta = original_delta;
        }else{
            layer prev = net.layers[i-1];
            state.input = prev.output_gpu;
            state.delta = prev.delta_gpu;
            if (net.optimized_memory && !prev.keep_delta_gpu) {
                state.delta = net.state_delta_gpu;
            }
        }
        if (l.onlyforward) continue;

        if (net.benchmark_layers) {
            start_time = get_time_point();
        }

        l.backward_gpu(l, state);

        if (net.benchmark_layers) {
            CHECK_CUDA(hipDeviceSynchronize());
            end_time = get_time_point();
            const double took_time = (end_time - start_time) / 1000;
            const double alpha = 0.9;
            if (avg_time_per_layer[i].time == 0) {
                avg_time_per_layer[i].layer_id = i;
                avg_time_per_layer[i].layer_type = l.type;
                avg_time_per_layer[i].time = took_time;
            }
            else avg_time_per_layer[i].time = avg_time_per_layer[i].time * alpha + took_time * (1 - alpha);

            sorted_avg_time_per_layer[i] = avg_time_per_layer[i];
            printf("\n bw-layer %d - type: %d - %lf ms - avg_time %lf ms \n", i, l.type, took_time, avg_time_per_layer[i].time);
        }

        if (i != 0) {
            layer prev = net.layers[i - 1];
            if (net.optimized_memory && state.delta && !prev.keep_delta_gpu) {
                if (prev.delta_gpu != state.delta) simple_copy_ongpu(prev.outputs*prev.batch, state.delta, prev.delta_gpu);
                fill_ongpu(prev.outputs*prev.batch, 0, net.state_delta_gpu, 1);
            }
        }

        /*
        if(i != 0)
        {
            layer l = net.layers[i - 1];
            int state_delta_nan_inf = is_nan_or_inf(state.delta, l.outputs * l.batch);
            int state_input_nan_inf = is_nan_or_inf(state.input, l.outputs * l.batch);
            printf("\n i - %d  is_nan_or_inf(s.delta) = %d \n", i, state_delta_nan_inf);
            printf(" i - %d  is_nan_or_inf(s.input) = %d \n", i, state_input_nan_inf);
            if (state_delta_nan_inf || state_input_nan_inf) { printf(" found "); getchar(); }
        }
        */
    }

    if (net.adversarial && net.attention)
    {
        int img_size = net.w * net.h * net.c;
        float *original_input_cpu = (float *)xcalloc(img_size, sizeof(float));
        float *original_delta_cpu = (float *)xcalloc(img_size, sizeof(float));
        cuda_pull_array(original_input, original_input_cpu, img_size);
        cuda_pull_array(original_delta, original_delta_cpu, img_size);

        image attention_img = make_attention_image(img_size, original_delta_cpu, original_input_cpu, net.w, net.h, net.c, 0.7);
        show_image(attention_img, "attention_img");
        resize_window_cv("attention_img", 500, 500);

        //static int img_counter = 0;
        //img_counter++;
        //char buff[256];
        //sprintf(buff, "attention_img_%d.png", img_counter);
        //save_image_png(attention_img, buff);
        free_image(attention_img);

        image attention_mask_img = make_attention_image(img_size, original_delta_cpu, original_delta_cpu, net.w, net.h, net.c, 1.0);
        show_image(attention_mask_img, "attention_mask_img");
        resize_window_cv("attention_mask_img", 500, 500);

        //sprintf(buff, "attention_mask_img_%d.png", img_counter);
        //save_image_png(attention_mask_img, buff);
        free_image(attention_mask_img);

        free(original_input_cpu);
        free(original_delta_cpu);
    }
    if (net.adversarial) {
        int x_size = get_network_input_size(net)*net.batch;
        printf(" x_size = %d, original_delta = %p, original_input = %p, net.learning_rate = %f \n",
            x_size, original_delta, original_input, net.learning_rate);
        axpy_ongpu(x_size, net.learning_rate, original_delta, 1, original_input, 1);
        constrain_min_max_ongpu(x_size, 0, 1, original_input, 1);
    }

    if (net.benchmark_layers) {
        printf("\n\nSorted by time (backward):\n");
        qsort(sorted_avg_time_per_layer, net.n, sizeof(time_benchmark_layers), time_comparator);
        for (i = 0; i < net.n; ++i) {
            //printf("layer %d - type: %d - avg_time %lf ms \n", avg_time_per_layer[i].layer_id, avg_time_per_layer[i].layer_type, avg_time_per_layer[i].time);
            printf("%d - bw-sort-layer %d - type: %d - avg_time %lf ms \n", i, sorted_avg_time_per_layer[i].layer_id, sorted_avg_time_per_layer[i].layer_type, sorted_avg_time_per_layer[i].time);
        }
    }
}

void update_network_gpu(network net)
{
    cuda_set_device(net.gpu_index);
    const int iteration_num = (*net.seen) / (net.batch * net.subdivisions);
    int i;
    int update_batch = net.batch*net.subdivisions * get_sequence_value(net);
    float rate = get_current_rate(net);
    for(i = 0; i < net.n; ++i){
        layer l = net.layers[i];
        if (l.train == 0) continue;

        l.t = get_current_batch(net);
        if (iteration_num > (net.max_batches * 1 / 2)) l.deform = 0;
        if (l.burnin_update && (l.burnin_update*net.burn_in > iteration_num)) continue;
        if (l.train_only_bn) continue;

        if(l.update_gpu && l.dont_update < iteration_num){
            l.update_gpu(l, update_batch, rate, net.momentum, net.decay, net.loss_scale);
        }
    }
}

void forward_backward_network_gpu(network net, float *x, float *y)
{
    network_state state;
    state.index = 0;
    state.net = net;
    int x_size = get_network_input_size(net)*net.batch;
    int y_size = get_network_output_size(net)*net.batch;
    if(net.layers[net.n-1].truths) y_size = net.layers[net.n-1].truths*net.batch;
    if(!*net.input_gpu){
        *net.input_gpu = cuda_make_array(x, x_size);
        *net.truth_gpu = cuda_make_array(y, y_size);
    }else{
        cuda_push_array(*net.input_gpu, x, x_size);
        cuda_push_array(*net.truth_gpu, y, y_size);
    }
    state.input = *net.input_gpu;
    state.delta = 0;
    if (net.adversarial) {
        state.delta = cuda_make_array(NULL, x_size);
    }
    state.truth = *net.truth_gpu;
    state.train = 1;
#if defined(CUDNN_HALF) && defined(CUDNN)
    int i;
    for (i = 0; i < net.n; ++i) {
        layer l = net.layers[i];
        if (net.cudnn_half){
            if (l.type == CONVOLUTIONAL && l.weights_gpu && l.weights_gpu16) {
                assert((l.nweights) > 0);
                cuda_convert_f32_to_f16(l.weights_gpu, l.nweights, l.weights_gpu16);
            }
            else if (l.type == CRNN && l.input_layer->weights_gpu && l.input_layer->weights_gpu16) {
                assert((l.input_layer->c*l.input_layer->n*l.input_layer->size*l.input_layer->size) > 0);
                cuda_convert_f32_to_f16(l.input_layer->weights_gpu, l.input_layer->nweights, l.input_layer->weights_gpu16);
                cuda_convert_f32_to_f16(l.self_layer->weights_gpu, l.self_layer->nweights, l.self_layer->weights_gpu16);
                cuda_convert_f32_to_f16(l.output_layer->weights_gpu, l.output_layer->nweights, l.output_layer->weights_gpu16);
            }
            else if (l.type == CONV_LSTM && l.wf->weights_gpu && l.wf->weights_gpu16) {
                assert((l.wf->c * l.wf->n * l.wf->size * l.wf->size) > 0);
                if (l.peephole) {
                    cuda_convert_f32_to_f16(l.vf->weights_gpu, l.vf->nweights, l.vf->weights_gpu16);
                    cuda_convert_f32_to_f16(l.vi->weights_gpu, l.vi->nweights, l.vi->weights_gpu16);
                    cuda_convert_f32_to_f16(l.vo->weights_gpu, l.vo->nweights, l.vo->weights_gpu16);
                }
                cuda_convert_f32_to_f16(l.wf->weights_gpu, l.wf->nweights, l.wf->weights_gpu16);
                if (!l.bottleneck) {
                    cuda_convert_f32_to_f16(l.wi->weights_gpu, l.wi->nweights, l.wi->weights_gpu16);
                    cuda_convert_f32_to_f16(l.wg->weights_gpu, l.wg->nweights, l.wg->weights_gpu16);
                    cuda_convert_f32_to_f16(l.wo->weights_gpu, l.wo->nweights, l.wo->weights_gpu16);
                }
                cuda_convert_f32_to_f16(l.uf->weights_gpu, l.uf->nweights, l.uf->weights_gpu16);
                cuda_convert_f32_to_f16(l.ui->weights_gpu, l.ui->nweights, l.ui->weights_gpu16);
                cuda_convert_f32_to_f16(l.ug->weights_gpu, l.ug->nweights, l.ug->weights_gpu16);
                cuda_convert_f32_to_f16(l.uo->weights_gpu, l.uo->nweights, l.uo->weights_gpu16);
            }
        }
    }
#endif
    forward_network_gpu(net, state);
    //hipStreamSynchronize(get_cuda_stream());
    backward_network_gpu(net, state);

    if (net.adversarial) {
        cuda_free(state.delta);
        cuda_pull_array(*net.input_gpu, x, x_size);
    }
    if(*(state.net.total_bbox) > 0)
        fprintf(stderr, " total_bbox = %d, rewritten_bbox = %f %% \n", *(state.net.total_bbox), 100 * (float)*(state.net.rewritten_bbox) / *(state.net.total_bbox));
}

float train_network_datum_gpu(network net, float *x, float *y)
{
    *net.seen += net.batch;
    if (net.adversarial_lr && rand_int(0, 1) == 1 && get_current_iteration(net) > net.burn_in) {
        net.adversarial = 1;
        float lr_old = net.learning_rate;
        float scale = (get_current_iteration(net) / ((float)net.max_batches));
        //scale = sin(scale * M_PI);
        net.learning_rate = net.adversarial_lr * scale;
        layer l = net.layers[net.n - 1];
        int y_size = get_network_output_size(net)*net.batch;
        if (net.layers[net.n - 1].truths) y_size = net.layers[net.n - 1].truths*net.batch;
        float *truth_cpu = (float *)xcalloc(y_size, sizeof(float));

        const int img_size = net.w*net.h*net.c;
        float *old_input = (float *)xcalloc(img_size*net.batch, sizeof(float));
        memcpy(old_input, x, img_size*net.batch * sizeof(float));

        printf("\n adversarial training, adversarial_lr = %f \n", net.adversarial_lr * scale);

        forward_backward_network_gpu(net, x, truth_cpu);

        int b;
        for (b = 0; b < net.batch; ++b) {
            if (b % 2 == 1 && net.contrastive) {
                //printf(" b = %d old img, ", b);
                memcpy(x + img_size*b, old_input + img_size*b, img_size * sizeof(float));
            }
        }

        image im;
        im.w = net.w;
        im.h = net.h;
        im.c = net.c;
        im.data = x;
        show_image(im, "adversarial data augmentation");
        resize_window_cv("adversarial data augmentation", 500, 500);
        wait_key_cv(1);

        free(old_input);
        free(truth_cpu);
        net.learning_rate = lr_old;
        net.adversarial = 0;
    }
    forward_backward_network_gpu(net, x, y);
    float error = get_network_cost(net);
    //if (((*net.seen) / net.batch) % net.subdivisions == 0) update_network_gpu(net);
    const int sequence = get_sequence_value(net);
    //if (((*net.seen) / net.batch) % (net.subdivisions*sequence) == 0) update_network_gpu(net);

    return error;
}

typedef struct {
    network net;
    data d;
    float *err;
} train_args;

void *train_thread(void *ptr)
{
    train_args args = *(train_args*)ptr;
    free(ptr);
    cuda_set_device(args.net.gpu_index);
    *args.err = train_network(args.net, args.d);
    return 0;
}

pthread_t train_network_in_thread(network net, data d, float *err)
{
    pthread_t thread;
    train_args *ptr = (train_args *)calloc(1, sizeof(train_args));
    ptr->net = net;
    ptr->d = d;
    ptr->err = err;
    if(pthread_create(&thread, 0, train_thread, ptr)) error("Thread creation failed", DARKNET_LOC);
    return thread;
}

void pull_updates(layer l)
{
    if(l.type == CONVOLUTIONAL){
        cuda_pull_array(l.bias_updates_gpu, l.bias_updates, l.n);
        cuda_pull_array(l.weight_updates_gpu, l.weight_updates, l.nweights);
        if(l.scale_updates) cuda_pull_array(l.scale_updates_gpu, l.scale_updates, l.n);
    } else if(l.type == CONNECTED){
        cuda_pull_array(l.bias_updates_gpu, l.bias_updates, l.outputs);
        cuda_pull_array(l.weight_updates_gpu, l.weight_updates, l.outputs*l.inputs);
    }
}

void push_updates(layer l)
{
    if(l.type == CONVOLUTIONAL){
        cuda_push_array(l.bias_updates_gpu, l.bias_updates, l.n);
        cuda_push_array(l.weight_updates_gpu, l.weight_updates, l.nweights);
        if(l.scale_updates) cuda_push_array(l.scale_updates_gpu, l.scale_updates, l.n);
    } else if(l.type == CONNECTED){
        cuda_push_array(l.bias_updates_gpu, l.bias_updates, l.outputs);
        cuda_push_array(l.weight_updates_gpu, l.weight_updates, l.outputs*l.inputs);
    }
}

void update_layer(layer l, network net)
{
    int update_batch = net.batch*net.subdivisions;
    float rate = get_current_rate(net);
    l.t = get_current_batch(net);
    if(l.update_gpu){
        l.update_gpu(l, update_batch, rate, net.momentum, net.decay, net.loss_scale);
    }
}

void merge_weights(layer l, layer base)
{
    if (l.type == CONVOLUTIONAL) {
        axpy_cpu(l.n, 1, l.biases, 1, base.biases, 1);
        axpy_cpu(l.nweights, 1, l.weights, 1, base.weights, 1);
        if (l.scales) {
            axpy_cpu(l.n, 1, l.scales, 1, base.scales, 1);
        }
    } else if(l.type == CONNECTED) {
        axpy_cpu(l.outputs, 1, l.biases, 1, base.biases, 1);
        axpy_cpu(l.outputs*l.inputs, 1, l.weights, 1, base.weights, 1);
    }
}

void scale_weights(layer l, float s)
{
    if (l.type == CONVOLUTIONAL) {
        scal_cpu(l.n, s, l.biases, 1);
        scal_cpu(l.nweights, s, l.weights, 1);
        if (l.scales) {
            scal_cpu(l.n, s, l.scales, 1);
        }
    } else if(l.type == CONNECTED) {
        scal_cpu(l.outputs, s, l.biases, 1);
        scal_cpu(l.outputs*l.inputs, s, l.weights, 1);
    }
}


void pull_weights(layer l)
{
    if(l.type == CONVOLUTIONAL){
        cuda_pull_array(l.biases_gpu, l.biases, l.n);
        cuda_pull_array(l.weights_gpu, l.weights, l.nweights);
        if(l.scales) cuda_pull_array(l.scales_gpu, l.scales, l.n);
    } else if(l.type == CONNECTED){
        cuda_pull_array(l.biases_gpu, l.biases, l.outputs);
        cuda_pull_array(l.weights_gpu, l.weights, l.outputs*l.inputs);
    }
}

void push_weights(layer l)
{
    if(l.type == CONVOLUTIONAL){
        cuda_push_array(l.biases_gpu, l.biases, l.n);
        cuda_push_array(l.weights_gpu, l.weights, l.nweights);
        if(l.scales) cuda_push_array(l.scales_gpu, l.scales, l.n);
    } else if(l.type == CONNECTED){
        cuda_push_array(l.biases_gpu, l.biases, l.outputs);
        cuda_push_array(l.weights_gpu, l.weights, l.outputs*l.inputs);
    }
}

void distribute_weights(layer l, layer base)
{
    if(l.type == CONVOLUTIONAL){
        cuda_push_array(l.biases_gpu, base.biases, l.n);
        cuda_push_array(l.weights_gpu, base.weights, l.nweights);
        if(base.scales) cuda_push_array(l.scales_gpu, base.scales, l.n);
    } else if(l.type == CONNECTED){
        cuda_push_array(l.biases_gpu, base.biases, l.outputs);
        cuda_push_array(l.weights_gpu, base.weights, l.outputs*l.inputs);
    }
}


void merge_updates(layer l, layer base)
{
    if (l.type == CONVOLUTIONAL) {
        axpy_cpu(l.n, 1, l.bias_updates, 1, base.bias_updates, 1);
        axpy_cpu(l.nweights, 1, l.weight_updates, 1, base.weight_updates, 1);
        if (l.scale_updates) {
            axpy_cpu(l.n, 1, l.scale_updates, 1, base.scale_updates, 1);
        }
    } else if(l.type == CONNECTED) {
        axpy_cpu(l.outputs, 1, l.bias_updates, 1, base.bias_updates, 1);
        axpy_cpu(l.outputs*l.inputs, 1, l.weight_updates, 1, base.weight_updates, 1);
    }
}

void distribute_updates(layer l, layer base)
{
    if(l.type == CONVOLUTIONAL){
        cuda_push_array(l.bias_updates_gpu, base.bias_updates, l.n);
        cuda_push_array(l.weight_updates_gpu, base.weight_updates, l.nweights);
        if(base.scale_updates) cuda_push_array(l.scale_updates_gpu, base.scale_updates, l.n);
    } else if(l.type == CONNECTED){
        cuda_push_array(l.bias_updates_gpu, base.bias_updates, l.outputs);
        cuda_push_array(l.weight_updates_gpu, base.weight_updates, l.outputs*l.inputs);
    }
}

void sync_layer(network *nets, int n, int j)
{
    //printf("Syncing layer %d\n", j);
    int i;
    network net = nets[0];
    layer base = net.layers[j];
    cuda_set_device(net.gpu_index);
    pull_weights(base);
    for (i = 1; i < n; ++i) {
        cuda_set_device(nets[i].gpu_index);
        layer l = nets[i].layers[j];
        pull_weights(l);
        merge_weights(l, base);
    }
    scale_weights(base, 1./n);
    for (i = 0; i < n; ++i) {
        cuda_set_device(nets[i].gpu_index);
        layer l = nets[i].layers[j];
        distribute_weights(l, base);
    }
    //printf("Done syncing layer %d\n", j);
}

typedef struct{
    network *nets;
    int n;
    int j;
} sync_args;

void *sync_layer_thread(void *ptr)
{
    sync_args args = *(sync_args*)ptr;
    sync_layer(args.nets, args.n, args.j);
    free(ptr);
    return 0;
}

pthread_t sync_layer_in_thread(network *nets, int n, int j)
{
    pthread_t thread;
    sync_args *ptr = (sync_args *)calloc(1, sizeof(sync_args));
    ptr->nets = nets;
    ptr->n = n;
    ptr->j = j;
    if(pthread_create(&thread, 0, sync_layer_thread, ptr)) error("Thread creation failed", DARKNET_LOC);
    return thread;
}

void sync_nets(network *nets, int n, int interval)
{
    int j;
    int layers = nets[0].n;
    pthread_t *threads = (pthread_t *) calloc(layers, sizeof(pthread_t));

    *nets[0].seen += interval * (n-1) * nets[0].batch * nets[0].subdivisions;
    for (j = 0; j < n; ++j){
        *nets[j].seen = *nets[0].seen;
    }
    for (j = 0; j < layers; ++j) {
        threads[j] = sync_layer_in_thread(nets, n, j);
    }
    for (j = 0; j < layers; ++j) {
        pthread_join(threads[j], 0);
    }
    free(threads);
}

float train_networks(network *nets, int n, data d, int interval)
{
    int i;
#ifdef _DEBUG
    int batch = nets[0].batch;
    int subdivisions = nets[0].subdivisions;
    assert(batch * subdivisions * n == d.X.rows);
#endif
    pthread_t *threads = (pthread_t *) calloc(n, sizeof(pthread_t));
    float *errors = (float *) calloc(n, sizeof(float));

    float sum = 0;
    for(i = 0; i < n; ++i){
        data p = get_data_part(d, i, n);
        threads[i] = train_network_in_thread(nets[i], p, errors + i);
    }
    for(i = 0; i < n; ++i){
        pthread_join(threads[i], 0);
        //printf("%f\n", errors[i]);
        sum += errors[i];
    }
    //hipDeviceSynchronize();
    *nets[0].cur_iteration += (n - 1);
    *nets[0].seen = nets[0].batch * nets[0].subdivisions * get_current_iteration(nets[0]); // remove this line, when you will save to weights-file both: seen & cur_iteration
    if (get_current_iteration(nets[0]) % interval == 0)
    {
        printf("Syncing... ");
        fflush(stdout);
        sync_nets(nets, n, interval);
        printf("Done!\n");
    }
    //hipDeviceSynchronize();
    free(threads);
    free(errors);
    return (float)sum/(n);
}

float *get_network_output_layer_gpu(network net, int i)
{
    layer l = net.layers[i];
    if(l.type != REGION && l.type != YOLO && (*net.cuda_graph_ready) == 0) cuda_pull_array(l.output_gpu, l.output, l.outputs*l.batch);
    return l.output;
}

float *get_network_output_gpu(network net)
{
    int i;
    for(i = net.n-1; i > 0; --i) if(net.layers[i].type != COST) break;
    return get_network_output_layer_gpu(net, i);
}

float *network_predict_gpu(network net, float *input)
{
    if (net.gpu_index != cuda_get_device())
        cuda_set_device(net.gpu_index);
    int size = get_network_input_size(net) * net.batch;
    network_state state;
    state.index = 0;
    state.net = net;
    //state.input = cuda_make_array(input, size);   // memory will be allocated in the parse_network_cfg_custom()
    state.input = net.input_state_gpu;
    memcpy(net.input_pinned_cpu, input, size * sizeof(float));
    state.truth = 0;
    state.train = 0;
    state.delta = 0;

    //hipGraphExec_t instance = (hipGraphExec_t)net.cuda_graph_exec;
    static hipGraphExec_t instance;

    if ((*net.cuda_graph_ready) == 0) {
        static hipGraph_t graph;
        if (net.use_cuda_graph == 1) {
            int i;
            for (i = 0; i < 16; ++i) switch_stream(i);

            hipStream_t stream0 = switch_stream(0);
            CHECK_CUDA(hipDeviceSynchronize());
            printf("Try to capture graph... \n");
            //hipGraph_t graph = (hipGraph_t)net.cuda_graph;
            CHECK_CUDA(hipStreamBeginCapture(stream0, hipStreamCaptureModeGlobal));
        }

        cuda_push_array(state.input, net.input_pinned_cpu, size);
        forward_network_gpu(net, state);

        if (net.use_cuda_graph == 1) {
            hipStream_t stream0 = switch_stream(0);
            CHECK_CUDA(hipStreamEndCapture(stream0, &graph));
            CHECK_CUDA(hipGraphInstantiate(&instance, graph, NULL, NULL, 0));
            (*net.cuda_graph_ready) = 1;
            printf(" graph is captured... \n");
            CHECK_CUDA(hipDeviceSynchronize());
        }
        CHECK_CUDA(hipStreamSynchronize(get_cuda_stream()));
    }
    else {
        hipStream_t stream0 = switch_stream(0);
        //printf(" hipGraphLaunch \n");
        CHECK_CUDA( hipGraphLaunch(instance, stream0) );
        CHECK_CUDA( hipStreamSynchronize(stream0) );
        //printf(" ~hipGraphLaunch \n");
    }

    float *out = get_network_output_gpu(net);
    reset_wait_stream_events();
    //cuda_free(state.input);   // will be freed in the free_network()
    return out;
}
